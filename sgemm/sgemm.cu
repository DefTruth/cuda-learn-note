#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

// -------------------------------------- FP32 -------------------------------------- 
// SGEMM naive: compute one c[i,j] element per threads, all row major
__global__ void sgemm_naive_f32_kernel(float* a, float* b, float* c, int M, int N, int K) {

  int n = blockIdx.x * blockDim.x + threadIdx.x;
  int m = blockIdx.y * blockDim.y + threadIdx.y;

  if (m < M && n < N) {
    float psum = 0.0;
    #pragma unroll
    for (int k = 0; k < K; k++) {
      // m row in a matrix, n col in b matrix
      psum += a[m * K + k] * b[k * N + n];
    }
    c[m * N + n] = psum; // c[m,n]
  }
}

// SGEMM: Block Tile + K Tile, with smem
// Block Tile (BM, BN) + K Tile (BK=32)
// grid((N + BN - 1) / BN, (M + BM - 1) / BM), block(BN, BM)
// a: MxK, b: KxN, c: MxN, compute: c = a * b, all row major  
template<const int BM=32, const int BN=32, const int BK=32>
__global__ void sgemm_sliced_k_f32_kernel(float* a, float* b, float* c, int M, int N, int K) {
  // [1] Block Tile: 32x32的block处理c上一块32x32的元素计算
  // [2]     K Tile: 使用共享内存，并将K分块为BK大小的块
  __shared__ float s_a[BM][BK], s_b[BK][BN]; 

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tid = threadIdx.y * blockDim.x + tx; // tid within the block
  // load values to shared memory, 32x32 threads working together 
  // to fetch data along the row direction of a and b both for s_a 
  // and s_b 32x32x4x2=8KB, we use 32x32 threads within block to 
  // load 32x32 elements from global memory to shared memory, namely, 
  // each thread will load 1 element.
  int load_smem_a_m = tid / 32; // 0~31, tid / 32, tid / BM, threadIdx.y
  int load_smem_a_k = tid % 32; // 0~31, tid % 32, tid % BK, threadIdx.x
  int load_smem_b_k = tid / 32; // 0~31, tid / 32, tid / BK, threadIdx.y
  int load_smem_b_n = tid % 32; // 0~31, tid % 32, tid % BN, threadIdx.x
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c
  // if (load_gmem_a_m >= M || load_gmem_b_n >= N) return;
  
  float sum = 0.f;
  for (int bk = 0; bk < (K + BK - 1) / BK; ++bk) {
    int load_gmem_a_k = bk * BK + load_smem_a_k;
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    s_a[load_smem_a_m][load_smem_a_k] = a[load_gmem_a_addr];
    int load_gmem_b_k = bk * BK + load_smem_b_k;
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;
    s_b[load_smem_b_k][load_smem_b_n] = b[load_gmem_b_addr];
    __syncthreads();
    #pragma unroll
    for (int k = 0; k < BK; ++k) {
      int comp_smem_a_m = load_smem_a_m;
      int comp_smem_b_n = load_smem_b_n;
      sum += s_a[comp_smem_a_m][k] * s_b[k][comp_smem_b_n];
    }
    __syncthreads();
  }
  int store_gmem_c_m = load_gmem_a_m;
  int store_gmem_c_n = load_gmem_b_n;
  int store_gmem_c_addr = store_gmem_c_m * N + store_gmem_c_n;
  c[store_gmem_c_addr] = sum;
}

// SGEMM: Block Tile + Thread Tile + K Tile + Vec4, with smem
// BK:TILE_K=8 BM=BN=128
// TM=TN=8 增加计算密度 BM/TM=16 BN/TN=16
// dim3 blockDim(BN/TN, BM/TM);
// dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM)
template<const int BM=128, const int BN=128, const int BK=8, const int TM=8, const int TN=8>
__global__ void sgemm_t_8x8_sliced_k_f32x4_kernel(float* a, float* b, float* c, int M, int N, int K) {
  // [1]  Block Tile: 一个16x16的block处理C上大小为128X128的一个目标块
  // [2] Thread Tile: 每个thread负责计算TM*TN(8*8)个元素，增加计算密度
  // [3]      K Tile: 将K分块，每块BK大小，迭代(K+BK-1/BK)次，
  //                  每次计算TM*TN个元素各自的部分乘累加
  // [4]   Vectorize: 减少load和store指令，使用float4
  
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tid = threadIdx.y * blockDim.x + tx; // tid within the block
  __shared__ float s_a[BM][BK], s_b[BK][BN]; // 2*128*8*4=8KB
  
  // 0. 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=8 按行读取 A行主序
  // 对于s_a每行8个数据，每个线程读取4个，需要2个线程；总共128行，需要128x2刚好256线程
  int load_smem_a_m = tid / 2; // tid/2 (128/8)*(128/8)=256 threads per block, tid/2->[0,128), BM=128 0~127
  int load_smem_a_k = (tid % 2 == 0) ? 0 : 4;  // (tid%2 == 0) ? 0 : 4, col of s_a 0,4
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=8 BN=128 按行读取 B行主序
  // 对于s_b每行128个数据，每个线程读4个数据，需要32个线程；总共8行，需要32x8=256个线程
  int load_smem_b_k = tid / 32; // tid/32, row of s_b 256/32=8 行 0~7
  int load_smem_b_n = (tid % 32) * 4;  // (tid % 32) * 4, col of s_b 0,4,...,124
  // 1. 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c
  
  float r_c[TM][TN] = {0.0}; // 8x8
  // 2. 先对K进行分块，每块BK大小
  for (int bk = 0; bk < (K + BK - 1) / BK; ++bk) {
    // 加载数据到共享内存smem s_a BM*BK 128*8 vectorize float4
    int load_gmem_a_k = bk * BK + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    FLOAT4(s_a[load_smem_a_m][load_smem_a_k]) = FLOAT4(a[load_gmem_a_addr]);
    // 加载数据到共享内存smem s_b BK*BN 8*128 vectorize float4
    int load_gmem_b_k = bk * BK + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 
    FLOAT4(s_b[load_smem_b_k][load_smem_b_n]) = FLOAT4(b[load_gmem_b_addr]); 
    __syncthreads();
    #pragma unroll
    for (int k = 0; k < BK; k++) {
      // 3. 每个线程负责计算BM*BN(12x128)中的TM*TN(8x8)个元素
      #pragma unroll
      for (int m = 0; m < TM; m++) {
        #pragma unroll
        for (int n = 0; n < TN; n++) {
          // k from 0~7，0 ~ BK, ty and tx range from 0 to 15, 16x8=128
          int comp_smem_a_m = ty * TM + m;  // 128*8 128/TM(8)=16 M方向 16线程
          int comp_smem_b_n = tx * TN + n;  // 8*128 128/TN(8)=16 N方向 16线程
          r_c[m][n] += s_a[comp_smem_a_m][k] * s_b[k][comp_smem_b_n];
        }
      }
    }
    __syncthreads();
  }

  #pragma unroll
  for (int m = 0; m < TM; ++m) {
    int store_gmem_c_m = by * BM + ty * TM + m;
    #pragma unroll
    for (int n = 0; n < TN; n += 4) {
      int store_gmem_c_n = bx * BN + tx * TN + n;
      int store_gmem_c_addr = store_gmem_c_m * N + store_gmem_c_n;
      FLOAT4(c[store_gmem_c_addr]) = FLOAT4(r_c[m][n]);
    }
  }
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)           \
if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) { \
  throw std::runtime_error("Tensor size mismatch!");  \
}

// SGEMM naive: compute one c[i,j] element per threads, all row major
void sgemm_naive_f32(torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int BM = 32;
  constexpr int BN = 32;

  dim3 block(BN, BM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  sgemm_naive_f32_kernel<<<grid, block>>>(
    reinterpret_cast<float*>(a.data_ptr()),
    reinterpret_cast<float*>(b.data_ptr()),
    reinterpret_cast<float*>(c.data_ptr()),
    M, N, K
  );
}

// SGEMM: Block Tile + K Tile, with smem
// Block Tile (BM, BN) + K Tile (BK=32)
// grid((N + BN - 1) / BN, (M + BM - 1) / BM), block(BN, BM)
// a: MxK, b: KxN, c: MxN, compute: c = a * b, all row major  
void sgemm_sliced_k_f32(torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int BM = 32;
  constexpr int BN = 32;
  constexpr int BK = 32;

  dim3 block(BN, BM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  sgemm_sliced_k_f32_kernel<BM, BN, BK><<<grid, block>>>(
    reinterpret_cast<float*>(a.data_ptr()),
    reinterpret_cast<float*>(b.data_ptr()),
    reinterpret_cast<float*>(c.data_ptr()),
    M, N, K
  );
}

// SGEMM: Block Tile + Thread Tile + K Tile + Vec4, with smem
// BK:TILE_K=8 BM=BN=128
// TM=TN=8 增加计算密度 BM/TM=16 BN/TN=16
// dim3 blockDim(BN/TN, BM/TM);
// dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM)
void sgemm_t_8x8_sliced_k_f32x4(torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int BM = 128;
  constexpr int BN = 128;
  constexpr int BK = 8; 
  constexpr int TM = 8;
  constexpr int TN = 8;

  dim3 block(BN/TN, BM/TM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  sgemm_t_8x8_sliced_k_f32x4_kernel<BM, BN, BK, TM, TN><<<grid, block>>>(
    reinterpret_cast<float*>(a.data_ptr()),
    reinterpret_cast<float*>(b.data_ptr()),
    reinterpret_cast<float*>(c.data_ptr()),
    M, N, K
  );
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(sgemm_naive_f32)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_sliced_k_f32)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x8_sliced_k_f32x4)
}
