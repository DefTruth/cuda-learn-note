#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])

// -------------------------------------- FP32 -------------------------------------- 
// Warp Reduce Sum
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

// Block reduce sum/max/min device helper for Layer/RMS Norm/Softmax etc.
// grid 1D block 1D, grid(N/256), block(256)
template<const int NUM_THREADS=256>
__device__ __forceinline__ float block_reduce_sum_f32(float val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  
  val = warp_reduce_sum_f32<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
  val = warp_reduce_sum_f32<NUM_WARPS>(val);
  return val;
}

// RMS Norm: x: NxK(K=256<1024), y': NxK, y'=x/rms(x) each row
// 1/rms(x) = rsqrtf( sum(x^2)/K ) each row
// grid(N*K/K), block(K<1024) N=batch_size*seq_len, K=hidden_size
// y=y'*g (g: scale)
template<const int NUM_THREADS=256>
__global__ void rms_norm_f32_kernel(float* x, float* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const float epsilon = 1e-5f;

  __shared__ float s_variance; // shared within block
  float value = (idx < N * K) ? x[idx] : 0.0f; // load once only
  float variance = value * value;
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / (float) K + epsilon);
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  if (idx < N * K) y[idx] = (value * s_variance) * g;
}

// RMS Norm Vec4: x: NxK(K=256<1024), y': NxK, y'=x/rms(x) each row
// 1/rms(x) = rsqrtf( sum(x^2)/K ) each row
// grid(N*K/K), block(K/4<1024) N=batch_size*seq_len, K=hidden_size
// y=y'*g (g: scale)
template<const int NUM_THREADS=256/4>
__global__ void rms_norm_f32x4_kernel(float* x, float* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 4;
  const float epsilon = 1e-5f;

  __shared__ float s_variance; // shared within block
  float4 reg_x = FLOAT4(x[idx]);
  float variance = (idx < N * K) ? (reg_x.x * reg_x.x + reg_x.y * reg_x.y 
                                  + reg_x.z * reg_x.z + reg_x.w * reg_x.w) : 0.0f;
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / (float) K + epsilon);
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  float4 reg_y;
  reg_y.x = reg_x.x * s_variance * g;
  reg_y.y = reg_x.y * s_variance * g;
  reg_y.z = reg_x.z * s_variance * g;
  reg_y.w = reg_x.w * s_variance * g;
  if (idx < N * K) FLOAT4(y[idx]) = reg_y;
}

// -------------------------------------- FP16 -------------------------------------- 
// Warp Reduce Sum: Half
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ half warp_reduce_sum_f16_f16(half val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    // val = __hadd(val, __shfl_xor_sync(0xffffffff, val, mask));
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f16_f32(half val) {
  float val_f32 = __half2float(val);
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val_f32 += __shfl_xor_sync(0xffffffff, val_f32, mask);
  }
  return val_f32;
}

template<const int NUM_THREADS=256>
__device__ half block_reduce_sum_f16_f16(half val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ half shared[NUM_WARPS];
  // reduce using half dtype within warps
  val = warp_reduce_sum_f16_f16<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val;
  __syncthreads();
  val = (lane < NUM_WARPS) ? shared[lane] : __float2half(0.0f);
  val = warp_reduce_sum_f16_f16<NUM_WARPS>(val);
  return val; // half
}

template<const int NUM_THREADS=256>
__device__ float block_reduce_sum_f16_f32(half val) {
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
  static __shared__ float shared[NUM_WARPS];
  // reduce using float dtype within warps
  float val_f32 = warp_reduce_sum_f16_f32<WARP_SIZE>(val);
  if (lane == 0) shared[warp] = val_f32;
  __syncthreads();
  val_f32 = (lane < NUM_WARPS) ? shared[lane] : 0.0f;
  val_f32 = warp_reduce_sum_f32<NUM_WARPS>(val_f32);
  return val_f32; // float
}

template<const int NUM_THREADS=256>
__global__ void rms_norm_f16_f16_kernel(half* x, half* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const half epsilon = __float2half(1e-5f);
  const half g_      = __float2half(g);
  const half K_      = __int2half_rn(K);

  __shared__ half s_variance; // shared within block
  half value = (idx < N * K) ? x[idx] : __float2half(0.0f); // load once only
  half variance = value * value;
  variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
  if (tid == 0) s_variance = hrsqrt(variance / (K_ + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  if (idx < N * K) y[idx] = (value * s_variance) * g_;
}

template<const int NUM_THREADS=256>
__global__ void rms_norm_f16x2_f16_kernel(half* x, half* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 2;
  const half epsilon = __float2half(1e-5f);
  const half g_      = __float2half(g);
  const half K_      = __int2half_rn(K);

  __shared__ half s_variance; // shared within block
  half2 reg_x = HALF2(x[idx]);
  half variance = (idx < N * K) ? (reg_x.x * reg_x.x 
                                 + reg_x.y * reg_x.y): __float2half(0.0f);
  variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
  if (tid == 0) s_variance = hrsqrt(variance / (K_ + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  half2 reg_y;
  reg_y.x = reg_x.x * s_variance * g_;
  reg_y.y = reg_x.y * s_variance * g_;
  if (idx < N * K) HALF2(y[idx]) = reg_y;
}

template<const int NUM_THREADS=256>
__global__ void rms_norm_f16x8_f16_kernel(half* x, half* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = (bid * blockDim.x + threadIdx.x) * 2;
  const half epsilon = __float2half(1e-5f);
  const half g_      = __float2half(g);
  const half K_      = __int2half_rn(K);

  __shared__ half s_variance; // shared within block
  // manual unroll, load 128 bits in 1 memory issue.
  half2 reg_x_0 = HALF2(x[idx + 0]);
  half2 reg_x_1 = HALF2(x[idx + 2]);
  half2 reg_x_2 = HALF2(x[idx + 4]);
  half2 reg_x_3 = HALF2(x[idx + 6]);
  half variance = (((idx + 0) < N * K) ? (reg_x_0.x * reg_x_0.x 
                                        + reg_x_0.y * reg_x_0.y) 
                                        : __float2half(0.0f));
  variance     += (((idx + 2) < N * K) ? (reg_x_1.x * reg_x_1.x 
                                        + reg_x_1.y * reg_x_1.y) 
                                       : __float2half(0.0f));
  variance     += (((idx + 4) < N * K) ? (reg_x_2.x * reg_x_2.x 
                                        + reg_x_2.y * reg_x_2.y) 
                                        : __float2half(0.0f));
  variance     += (((idx + 6) < N * K) ? (reg_x_3.x * reg_x_3.x 
                                        + reg_x_3.y * reg_x_3.y) 
                                        : __float2half(0.0f));
  variance = block_reduce_sum_f16_f16<NUM_THREADS>(variance);
  if (tid == 0) s_variance = hrsqrt(variance / (K_ + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  // manual unroll
  half2 reg_y_0, reg_y_1, reg_y_2, reg_y_3;
  reg_y_0.x = reg_x_0.x * s_variance * g_;
  reg_y_0.y = reg_x_0.y * s_variance * g_;
  reg_y_1.x = reg_x_1.x * s_variance * g_;
  reg_y_1.y = reg_x_1.y * s_variance * g_;
  reg_y_2.x = reg_x_2.x * s_variance * g_;
  reg_y_2.y = reg_x_2.y * s_variance * g_;
  reg_y_3.x = reg_x_3.x * s_variance * g_;
  reg_y_3.y = reg_x_3.y * s_variance * g_;
  if ((idx + 0) < N * K) { HALF2(y[idx + 0]) = reg_y_0; }
  if ((idx + 2) < N * K) { HALF2(y[idx + 2]) = reg_y_1; }
  if ((idx + 4) < N * K) { HALF2(y[idx + 4]) = reg_y_2; }
  if ((idx + 6) < N * K) { HALF2(y[idx + 6]) = reg_y_3; }
}

template<const int NUM_THREADS=256>
__global__ void rms_norm_f16_f32_kernel(half* x, half* y, float g, int N, int K) {
  int tid = threadIdx.x; // 0..K-1
  int bid = blockIdx.x; // 0..N-1
  int idx = bid * blockDim.x + threadIdx.x;
  const float epsilon = 1e-5f;

  __shared__ float s_variance; // shared within block
  float value = (idx < N * K) ? __half2float(x[idx]) : 0.0f; // load once only
  float variance = value * value;
  variance = block_reduce_sum_f32<NUM_THREADS>(variance);
  if (tid == 0) s_variance = rsqrtf(variance / ((float) K + epsilon));
  // wait for s_variance in shared memory to be ready for all threads
  __syncthreads(); 
  if (idx < N * K) {
    y[idx] = __float2half((value * s_variance) * g);
  }
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T1, T2)                                \
if (((T2).size(0) != (T1).size(0)) || ((T2).size(1) != (T1).size(1))) { \
  throw std::runtime_error("Tensor size mismatch!");                    \
}

#define LANUCH_RMS_NORM_F32_KERNEL(K)      \
rms_norm_f32_kernel<(K)><<<grid, block>>>( \
  reinterpret_cast<float*>(x.data_ptr()),  \
  reinterpret_cast<float*>(y.data_ptr()),  \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F32_KERNEL(N, K)   \
  dim3 block((K));                           \
  dim3 grid((N));                            \     
  switch ((K))                               \
  {                                          \
  case 64:                                   \
    LANUCH_RMS_NORM_F32_KERNEL(64)           \
    break;                                   \
  case 128:                                  \
    LANUCH_RMS_NORM_F32_KERNEL(128)          \
    break;                                   \
  case 256:                                  \
    LANUCH_RMS_NORM_F32_KERNEL(256)          \
    break;                                   \
  case 512:                                  \
    LANUCH_RMS_NORM_F32_KERNEL(512)          \
    break;                                   \
  case 1024:                                 \
    LANUCH_RMS_NORM_F32_KERNEL(1024)         \
    break;                                   \
  default:                                   \
    throw std::runtime_error(                \
      "only support K: 64/128/256/512/1024");\
    break;                                   \
  } 

#define LANUCH_RMS_NORM_F32x4_KERNEL(K)        \
rms_norm_f32x4_kernel<(K)/4><<<grid, block>>>( \
  reinterpret_cast<float*>(x.data_ptr()),      \
  reinterpret_cast<float*>(y.data_ptr()),      \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F32x4_KERNEL(N, K) \
  dim3 block((K)/4);                         \
  dim3 grid((N));                            \     
  switch ((K))                               \
  {                                          \
  case 64:                                   \
    LANUCH_RMS_NORM_F32x4_KERNEL(64)         \
    break;                                   \
  case 128:                                  \
    LANUCH_RMS_NORM_F32x4_KERNEL(128)        \
    break;                                   \
  case 256:                                  \
    LANUCH_RMS_NORM_F32x4_KERNEL(256)        \
    break;                                   \
  case 512:                                  \
    LANUCH_RMS_NORM_F32x4_KERNEL(512)        \
    break;                                   \
  case 1024:                                 \
    LANUCH_RMS_NORM_F32x4_KERNEL(1024)       \
    break;                                   \
  default:                                   \
    throw std::runtime_error(                \
      "only support K: 64/128/256/512/1024");\
    break;                                   \
  } 

void rms_norm_f32(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kFloat32)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F32_KERNEL(N, K)
}

void rms_norm_f32x4(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kFloat32)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F32x4_KERNEL(N, K)
}

// fp16
#define LANUCH_RMS_NORM_F16F16_KERNEL(K)         \
rms_norm_f16_f16_kernel<(K)><<<grid, block>>>(   \
  reinterpret_cast<half*>(x.data_ptr()),         \
  reinterpret_cast<half*>(y.data_ptr()),         \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F16F16_KERNEL(N, K)   \
  dim3 block((K));                              \
  dim3 grid((N));                               \     
  switch ((K))                                  \
  {                                             \
  case 64:                                      \
    LANUCH_RMS_NORM_F16F16_KERNEL(64)           \
    break;                                      \
  case 128:                                     \
    LANUCH_RMS_NORM_F16F16_KERNEL(128)          \
    break;                                      \
  case 256:                                     \
    LANUCH_RMS_NORM_F16F16_KERNEL(256)          \
    break;                                      \
  case 512:                                     \
    LANUCH_RMS_NORM_F16F16_KERNEL(512)          \
    break;                                      \
  case 1024:                                    \
    LANUCH_RMS_NORM_F16F16_KERNEL(1024)         \
    break;                                      \
  default:                                      \
    throw std::runtime_error(                   \
      "only support K: 64/128/256/512/1024");   \
    break;                                      \
  } 

#define LANUCH_RMS_NORM_F16F32_KERNEL(K)         \
rms_norm_f16_f32_kernel<(K)><<<grid, block>>>( \
  reinterpret_cast<half*>(x.data_ptr()),         \
  reinterpret_cast<half*>(y.data_ptr()),         \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F16F32_KERNEL(N, K)   \
  dim3 block((K));                              \
  dim3 grid((N));                               \     
  switch ((K))                                  \
  {                                             \
  case 64:                                      \
    LANUCH_RMS_NORM_F16F32_KERNEL(64)           \
    break;                                      \
  case 128:                                     \
    LANUCH_RMS_NORM_F16F32_KERNEL(128)          \
    break;                                      \
  case 256:                                     \
    LANUCH_RMS_NORM_F16F32_KERNEL(256)          \
    break;                                      \
  case 512:                                     \
    LANUCH_RMS_NORM_F16F32_KERNEL(512)          \
    break;                                      \
  case 1024:                                    \
    LANUCH_RMS_NORM_F16F32_KERNEL(1024)         \
    break;                                      \
  default:                                      \
    throw std::runtime_error(                   \
      "only support K: 64/128/256/512/1024");   \
    break;                                      \
  } 

#define LANUCH_RMS_NORM_F16x2F16_KERNEL(K)           \
rms_norm_f16x2_f16_kernel<(K)/2><<<grid, block>>>( \
  reinterpret_cast<half*>(x.data_ptr()),             \
  reinterpret_cast<half*>(y.data_ptr()),             \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F16x2F16_KERNEL(N, K)   \
  dim3 block((K)/2);                              \
  dim3 grid((N));                                 \     
  switch ((K))                                    \
  {                                               \
  case 64:                                        \
    LANUCH_RMS_NORM_F16x2F16_KERNEL(64)           \
    break;                                        \
  case 128:                                       \
    LANUCH_RMS_NORM_F16x2F16_KERNEL(128)          \
    break;                                        \
  case 256:                                       \
    LANUCH_RMS_NORM_F16x2F16_KERNEL(256)          \
    break;                                        \
  case 512:                                       \
    LANUCH_RMS_NORM_F16x2F16_KERNEL(512)          \
    break;                                        \
  case 1024:                                      \
    LANUCH_RMS_NORM_F16x2F16_KERNEL(1024)         \
    break;                                        \
  default:                                        \
    throw std::runtime_error(                     \
      "only support K: 64/128/256/512/1024");     \
    break;                                        \
  } 

#define LANUCH_RMS_NORM_F16x8F16_KERNEL(K)           \
rms_norm_f16x8_f16_kernel<(K)/8><<<grid, block>>>( \
  reinterpret_cast<half*>(x.data_ptr()),             \
  reinterpret_cast<half*>(y.data_ptr()),             \
  g, N, (K));  

#define DISPATCH_RMS_NORM_F16x8F16_KERNEL(N, K)   \
  dim3 block((K)/8);                              \
  dim3 grid((N));                                 \     
  switch ((K))                                    \
  {                                               \
  case 64:                                        \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(64)           \
    break;                                        \
  case 128:                                       \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(128)          \
    break;                                        \
  case 256:                                       \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(256)          \
    break;                                        \
  case 512:                                       \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(512)          \
    break;                                        \
  case 1024:                                      \
    LANUCH_RMS_NORM_F16x8F16_KERNEL(1024)         \
    break;                                        \
  default:                                        \
    throw std::runtime_error(                     \
      "only support K: 64/128/256/512/1024");     \
    break;                                        \
  } 

void rms_norm_f16_f16(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F16F16_KERNEL(N, K)
}

void rms_norm_f16x2_f16(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F16x2F16_KERNEL(N, K)
}

void rms_norm_f16x8_f16(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F16x8F16_KERNEL(N, K)
}

void rms_norm_f16_f32(torch::Tensor x, torch::Tensor y, float g) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kHalf)       
  CHECK_TORCH_TENSOR_DTYPE(y, torch::kHalf)
  CHECK_TORCH_TENSOR_SHAPE(x, y)
  const int N = x.size(0);
  const int K = x.size(1);
  DISPATCH_RMS_NORM_F16F32_KERNEL(N, K)
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f32)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f32x4)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f16_f16)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f16x2_f16)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f16x8_f16)
  TORCH_BINDING_COMMON_EXTENSION(rms_norm_f16_f32)
}
