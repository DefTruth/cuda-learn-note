#include "hip/hip_runtime.h"
// modified from: https://github.com/Byeong-Chan/flash-attention-minimal/blob/add_matmul_optimize/flash_optimize_matmul.cu
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])

// Load matrix to REGISTER
#define LDMATRIX_X4(R0, R1, R2, R3, addr)                                             \
    asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" \
                 : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3)                             \
                 : "r"(addr))

// half mma 16x8x16 (only support "ARCH >= SM_80")
#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1)                                                    \
    asm volatile("mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n" \
                 : "=r"(RD0), "=r"(RD1)                                                                                \
                 : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0), "r"(RC1))

template<const int Bc, const int Br, const int d>
__global__  void flash_attn_2_fwd_f16_mma_m16n8k16_kernel(
  half* Q, half* K, half* V, const int N,
  const int Tc, const int Tr, const float scale,
  half* O) {
  // batch and head index
  int bx = blockIdx.x; int by = blockIdx.y;

  // warp and lane Id
  int warpId = threadIdx.x / 32;
  int laneId = threadIdx.x % 32;

  // Offset into Q, K, V, O - different for each batch and head
  int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = nh

  // Define SRAM for Q,K,V,O
  extern __shared__ half sram[];
  int tile_size = Br * d;  // size of Qi, Kj, Vj (Br == Bc)
  half* Qi = sram;
  half* Kj = sram + tile_size;
  half* Vj = sram + tile_size; // share with K

  // temporary register
  half reg[32];

  for (int i = 0; i < Tr; i++) {
    // Read Q from global memory to shared memory
    for (int x = threadIdx.x * 8; x < tile_size; x += 1024) {
      int dim_x = x % d;
      int dim_y = x / d;

      // fixed colum length(16) conversion for LDMATRIX
      int new_dim_x = dim_x % 16;
      int new_dim_y = (dim_y / 16 * (d / 16) * 16) + (dim_x / 16 * 16) + (dim_y % 16);

      FLOAT4(Qi[new_dim_y * 16 + new_dim_x]) = FLOAT4(Q[qkv_offset + (i * tile_size) + x]);
    }
    __syncthreads();

    // m_old, l_old
    float thread_max_old[2] = { -INFINITY, -INFINITY }; 
    float thread_sum_old[2] = { 0, 0 };

    // REGISTER for O
    float RO[d / 16][2][2][2] = { 0, };

    for (int j = 0; j < Tc; j++) {
      // m, l
      float thread_max[2] = { -INFINITY, -INFINITY }; 
      float thread_sum[2] = { 0, 0 };

      // REGISTER for mma
      uint32_t RC[Bc / 8][2] = { 0, };
      uint32_t RA[4];
      uint32_t RB[4];
      uint32_t RD[4];

      // Read K from global memory to shared memory
      for (int x = threadIdx.x * 8; x < tile_size; x += 1024) {
        int dim_x = x % d;
        int dim_y = x / d;

        int new_dim_x = dim_x % 16;
        int new_dim_y = (dim_y / 16 * (d / 16) * 16) + (dim_x / 16 * 16) + (dim_y % 16);

         FLOAT4(Kj[new_dim_y * 16 + new_dim_x]) = FLOAT4(K[qkv_offset + (j * tile_size) + x]);
      }
      __syncthreads();

      // Q @ K^T
      for (int k = 0; k < d / 16; k++) {
        // Bc x d to Bc / 4 x d (4 is warp size)
        uint32_t Qi_lane_addr = __cvta_generic_to_shared(&Qi[(warpId * 16 * d) + (laneId % 16) * 16 + (laneId / 16) * 8 + (k * 16 * 16)]);
        LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], Qi_lane_addr);

        for (int len = 0; len < Bc; len += 16) {
          uint32_t Kj_lane_addr = __cvta_generic_to_shared(&Kj[(len * d) + (laneId % 16) * 16 + (laneId / 16) * 8 + (k * 16 * 16)]);
          // be careful "not 0 1 2 3"
          LDMATRIX_X4(RB[0], RB[2], RB[1], RB[3], Kj_lane_addr);

          // 16x16x16 wmma *(16x8x16 mma 0)
          HMMA16816(RC[(len / 16) * 2 + 0][0], RC[(len / 16) * 2 + 0][1],
                    RA[0], RA[1], RA[2], RA[3],
                    RB[0], RB[1],
                    RC[(len / 16) * 2 + 0][0], RC[(len / 16) * 2 + 0][1]);

          // 16x16x16 wmma *(16x8x16 mma 1)
          HMMA16816(RC[(len / 16) * 2 + 1][0], RC[(len / 16) * 2 + 1][1],
                    RA[0], RA[1], RA[2], RA[3],
                    RB[2], RB[3],
                    RC[(len / 16) * 2 + 1][0], RC[(len / 16) * 2 + 1][1]);
        }
      }
      __syncthreads();

      // Read V from global memory to shared memory
      for (int x = threadIdx.x * 8; x < tile_size; x += 1024) {
        FLOAT4(reg[0]) = FLOAT4(V[qkv_offset + (j * tile_size) + x]);

        int dim_x = x % d;
        int dim_y = x / d;

        #pragma unroll
        for (int iter = 0; iter < 8; iter++) {
          int new_dim_y = ((dim_x + iter) / 16 * (Bc / 16) * 16) + (dim_y / 16 * 16) + ((dim_x + iter) % 16);
          int new_dim_x = dim_y % 16;

          Vj[new_dim_y * 16 + new_dim_x] = reg[iter];
        }
      }
      __syncthreads();

      // adapt from https://github.com/jundaf2/INT8-Flash-Attention-FMHA-Quantization/blob/main/inc/fmha_i8.cuh
      // Softmax phase (m, l calculate)
      // FETCHING REGISTER
      FLOAT4(reg[0])  = FLOAT4(RC[0][0]);
      FLOAT4(reg[8])  = FLOAT4(RC[2][0]);
      FLOAT4(reg[16]) = FLOAT4(RC[4][0]);
      FLOAT4(reg[24]) = FLOAT4(RC[6][0]);

      // thread level reduce max
      #pragma unroll
      for (int xi = 0; xi < Bc / 16; xi++) {
        #pragma unroll
        for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
          #pragma unroll
          for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
            float tmp_val1 = __half2float(reg[xi * 8 + tc_xi * 4 + tc_yi * 2 + 0]);
            float tmp_val2 = __half2float(reg[xi * 8 + tc_xi * 4 + tc_yi * 2 + 1]);
            float tmp_max_val = max(tmp_val1, tmp_val2) * scale;
            thread_max[tc_yi] = max(thread_max[tc_yi], tmp_max_val);
          }
        }
      }

      // warp level reduce max
      #pragma unroll
      for (int s = 2; s > 0; s >>= 1) {
        #pragma unroll
        for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
          thread_max[tc_yi] = max(thread_max[tc_yi], __shfl_xor_sync(0xffffffff, thread_max[tc_yi], s, 4));
        }
      }
      
      // thread level reduce sum
      #pragma unroll
      for (int xi = 0; xi < Bc / 16; xi++) {
        #pragma unroll
        for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
          #pragma unroll
          for (int tc_xi = 0; tc_xi < 2; tc_xi++) {
            float tmp_sum_val_0 = __expf(__half2float(reg[xi * 8 + tc_xi * 4 + tc_yi * 2 + 0]) * scale - thread_max[tc_yi]);
            float tmp_sum_val_1 = __expf(__half2float(reg[xi * 8 + tc_xi * 4 + tc_yi * 2 + 1]) * scale - thread_max[tc_yi]);
            reg[xi * 8 + tc_xi * 4 + tc_yi * 2 + 0] = __float2half(tmp_sum_val_0);
            reg[xi * 8 + tc_xi * 4 + tc_yi * 2 + 1] = __float2half(tmp_sum_val_1);
            thread_sum[tc_yi] += (tmp_sum_val_0 + tmp_sum_val_1);
          }
        }
      }

      // warp level reduce sum
      #pragma unroll
      for (int s = 2; s > 0; s >>= 1) {
        #pragma unroll
        for (int tc_yi = 0; tc_yi < 2; tc_yi++) {
          thread_sum[tc_yi] += __shfl_xor_sync(0xffffffff, thread_sum[tc_yi], s, 4);
        }
      }

      // FETCHING REGISTER for P
      FLOAT4(RC[0][0]) = FLOAT4(reg[0]);
      FLOAT4(RC[2][0]) = FLOAT4(reg[8]);
      FLOAT4(RC[4][0]) = FLOAT4(reg[16]);
      FLOAT4(RC[6][0]) = FLOAT4(reg[24]);

      // P @ V
      for (int k = 0; k < d / 16; k++) {
        RD[0] = RD[1] = RD[2] = RD[3] = 0;
        for (int len = 0; len < Bc; len += 16) {
          uint32_t Vj_lane_addr = __cvta_generic_to_shared(&Vj[(k * 16 * Bc) + (len * 16) + (laneId % 16) * 16 + (laneId / 16) * 8]);
          LDMATRIX_X4(RB[0], RB[2], RB[1], RB[3], Vj_lane_addr);

          HMMA16816(RD[0], RD[1],
                    RC[len / 16 * 2 + 0][0], RC[len / 16 * 2 + 0][1], RC[len / 16 * 2 + 1][0], RC[len / 16 * 2 + 1][1],
                    RB[0], RB[1],
                    RD[0], RD[1]);

          HMMA16816(RD[2], RD[3],
                    RC[len / 16 * 2 + 0][0], RC[len / 16 * 2 + 0][1], RC[len / 16 * 2 + 1][0], RC[len / 16 * 2 + 1][1],
                    RB[2], RB[3],
                    RD[2], RD[3]);
        }

        FLOAT4(reg[0]) =  FLOAT4(RD[0]);
        #pragma unroll
        for(int tc_yi = 0; tc_yi < 2; tc_yi++) {
          float thread_max_new = max(thread_max_old[tc_yi], thread_max[tc_yi]);
          float exp_max_old = __expf(thread_max_old[tc_yi] - thread_max_new);
          float exp_max = __expf(thread_max[tc_yi] - thread_max_new);
          float thread_sum_new = exp_max_old * thread_sum_old[tc_yi] + exp_max * thread_sum[tc_yi];
          #pragma unroll
          for(int tc_xi=0; tc_xi < 2; tc_xi++) {
            RO[k][tc_yi][tc_xi][0] =
              __frcp_rn(thread_sum_new) *
              (thread_sum_old[tc_yi] *
               exp_max_old * RO[k][tc_yi][tc_xi][0] +
               exp_max * __half2float(reg[tc_xi * 4 + tc_yi * 2 + 0]));

            RO[k][tc_yi][tc_xi][1] =
              __frcp_rn(thread_sum_new) *
              (thread_sum_old[tc_yi] *
               exp_max_old * RO[k][tc_yi][tc_xi][1] +
               exp_max * __half2float(reg[tc_xi * 4 + tc_yi * 2 + 1]));
          }
        }
      }

      // update m, l
      for(int tc_yi = 0; tc_yi < 2; tc_yi++) {
        float thread_max_new = max(thread_max_old[tc_yi], thread_max[tc_yi]);
        float exp_max_old = __expf(thread_max_old[tc_yi] - thread_max_new);
        float exp_max = __expf(thread_max[tc_yi] - thread_max_new);
        float thread_sum_new = exp_max_old * thread_sum_old[tc_yi] + exp_max * thread_sum[tc_yi];
        thread_sum_old[tc_yi] = thread_sum_new;
        thread_max_old[tc_yi] = thread_max_new;
      }
      __syncthreads();
    }

    // update O
    for (int k = 0; k < d / 16; k++) {
      #pragma unroll
      for(int tc_yi = 0; tc_yi < 2; tc_yi++) {
        #pragma unroll
        for(int tc_xi=0; tc_xi < 2; tc_xi++) {
          int lane_pos = qkv_offset + i * Br * d + (warpId * 16 * d) + (laneId / 4 + tc_yi * 8) * d + tc_xi * 8 + laneId % 4 * 2 + (k * 16);
          O[lane_pos + 0] = __float2half(RO[k][tc_yi][tc_xi][0]);
          O[lane_pos + 1] = __float2half(RO[k][tc_yi][tc_xi][1]);
        }
      }
    }
    __syncthreads();
  }
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T1, T2)             \
if (((T2).size(0) != (T1).size(0)) ||                \
    ((T2).size(1) != (T1).size(1)) ||                \
    ((T2).size(2) != (T1).size(2)) ||                \
    ((T2).size(3) != (T1).size(3))) {                \
  throw std::runtime_error("Tensor size mismatch!"); \
}

torch::Tensor flash_attn_2_fwd_f16_mma_m16n8k16(
  torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
  // TODO: determine Bc, Br dynamically
  CHECK_TORCH_TENSOR_DTYPE(Q, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(K, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(V, torch::kHalf)
  const int Bc = 64; 
  const int Br = 64;

  const int B = Q.size(0); 
  const int nh = Q.size(1);
  const int N = Q.size(2); 
  const int d = Q.size(3);
  CHECK_TORCH_TENSOR_SHAPE(K, Q)
  CHECK_TORCH_TENSOR_SHAPE(V, Q)

  const int Tc = ceil((float) N / Bc); 
  const int Tr = ceil((float) N / Br);
  const float scale = 1.0 / sqrt(d);

  // Initialize O, l, m to HBM
  auto options = torch::TensorOptions().device(torch::kCUDA).dtype(torch::kHalf);
  auto O = torch::zeros_like(Q, options);

  // Calculate SRAM size needed per block
  const int sram_size = (2 * Br * d * sizeof(half));
  int max_sram_size;
  hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

  dim3 grid(B, nh);  // batch_size x num_heads
  dim3 block(128);   // 4 Warps per block

  // hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (d == 64) {
    flash_attn_2_fwd_f16_mma_m16n8k16_kernel<Bc, Br, 64><<<
    grid, block, sram_size>>>(
      reinterpret_cast<half*>(Q.data_ptr()),
      reinterpret_cast<half*>(K.data_ptr()),
      reinterpret_cast<half*>(V.data_ptr()),
      N, Tc, Tr, scale,
      reinterpret_cast<half*>(O.data_ptr())
    );
  }
  if (d == 128) {
    flash_attn_2_fwd_f16_mma_m16n8k16_kernel<Bc, Br, 128><<<
    grid, block, sram_size>>>(
      reinterpret_cast<half*>(Q.data_ptr()),
      reinterpret_cast<half*>(K.data_ptr()),
      reinterpret_cast<half*>(V.data_ptr()),
      N, Tc, Tr, scale,
      reinterpret_cast<half*>(O.data_ptr())
    );
  }
  return O;
}

void flash_attn_2_fwd_f16_mma_m16n8k16_v2(
  torch::Tensor Q, torch::Tensor K, torch::Tensor V, torch::Tensor O) {
  // TODO: determine Bc, Br dynamically
  CHECK_TORCH_TENSOR_DTYPE(Q, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(K, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(V, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(O, torch::kHalf)
  const int Bc = 64; 
  const int Br = 64;

  const int B = Q.size(0); 
  const int nh = Q.size(1);
  const int N = Q.size(2); 
  const int d = Q.size(3);
  CHECK_TORCH_TENSOR_SHAPE(K, Q)
  CHECK_TORCH_TENSOR_SHAPE(V, Q)
  CHECK_TORCH_TENSOR_SHAPE(O, Q)

  const int Tc = ceil((float) N / Bc); 
  const int Tr = ceil((float) N / Br);
  const float scale = 1.0 / sqrt(d);

  // Calculate SRAM size needed per block
  const int sram_size = (2 * Br * d * sizeof(half));
  int max_sram_size;
  hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

  dim3 grid(B, nh);  // batch_size x num_heads
  dim3 block(128);   // 4 Warps per block

  // hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (d == 64) {
    flash_attn_2_fwd_f16_mma_m16n8k16_kernel<Bc, Br, 64><<<
    grid, block, sram_size>>>(
      reinterpret_cast<half*>(Q.data_ptr()),
      reinterpret_cast<half*>(K.data_ptr()),
      reinterpret_cast<half*>(V.data_ptr()),
      N, Tc, Tr, scale,
      reinterpret_cast<half*>(O.data_ptr())
    );
  }
  if (d == 128) {
    flash_attn_2_fwd_f16_mma_m16n8k16_kernel<Bc, Br, 128><<<
    grid, block, sram_size>>>(
      reinterpret_cast<half*>(Q.data_ptr()),
      reinterpret_cast<half*>(K.data_ptr()),
      reinterpret_cast<half*>(V.data_ptr()),
      N, Tc, Tr, scale,
      reinterpret_cast<half*>(O.data_ptr())
    );
  }
}
