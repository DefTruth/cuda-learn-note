#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>

#include <torch/types.h>
#include <torch/extension.h>

#include "hipblas.h"

// NN: A/B/C All row major
void cublas_tensor_op_nn(half *A, half *B, half *C,  size_t M, size_t N, size_t K) {

  static hipblasHandle_t handle = nullptr;
  hipblasCreate(&handle);
  hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

  static half alpha = 1.0;
  static half beta = 0.0;

  hipblasGemmEx(handle, 
               HIPBLAS_OP_N, 
               HIPBLAS_OP_N, 
               N, M, K, 
               &alpha, 
               B, HIP_R_16F, N, 
               A, HIP_R_16F, K, 
               &beta,  
               C, HIP_R_16F, N, 
               HIPBLAS_COMPUTE_16F,
               CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  // hipblasDestroy(handle);
}

// TN: A row major MxK, B col major NxK, C row major MxN
void cublas_tensor_op_tn(half *A, half *B, half *C,  size_t M, size_t N, size_t K) {

  static hipblasHandle_t handle = nullptr;
  hipblasCreate(&handle);
  hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

  static half alpha = 1.0;
  static half beta = 0.0;

  hipblasGemmEx(handle, 
               HIPBLAS_OP_T, 
               HIPBLAS_OP_N, 
               N, M, K, 
               &alpha, 
               B, HIP_R_16F, K, 
               A, HIP_R_16F, K, 
               &beta,  
               C, HIP_R_16F, N, 
               HIPBLAS_COMPUTE_16F,
               CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  // hipblasDestroy(handle);
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)           \
if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) { \
  throw std::runtime_error("Tensor size mismatch!");  \
}

// NN: A/B/C All row major
void hgemm_cublas_tensor_op_nn(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)

  cublas_tensor_op_nn(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// TN: A row major MxK, B col major NxK, C row major MxN
void hgemm_cublas_tensor_op_tn(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)

  cublas_tensor_op_tn(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}
