#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>

#include <torch/types.h>
#include <torch/extension.h>

#include "hipblas.h"


void cublas_tensor_op(half *A, half *B, half *C,  size_t M, 
                      size_t N, size_t K) {

  hipblasHandle_t handle = nullptr;
  hipblasCreate(&handle);
  hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

  static half alpha = 1.0;
  static half beta = 0.0;

  hipblasGemmEx(handle, 
               HIPBLAS_OP_N, 
               HIPBLAS_OP_N, 
               N, M, K, 
               &alpha, 
               B, HIP_R_16F, N, 
               A, HIP_R_16F, K, 
               &beta,  
               C, HIP_R_16F, N, 
               HIPBLAS_COMPUTE_16F,
               CUBLAS_GEMM_DEFAULT_TENSOR_OP);

  // why this line will make cublas slow down?  
  // hipblasDestroy(handle);
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)           \
if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) { \
  throw std::runtime_error("Tensor size mismatch!");  \
}

// cublas tensor op
void hgemm_cublas_tensor_op(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)

  cublas_tensor_op(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}
