#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>
#include <torch/types.h>
#include <torch/extension.h>
using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2*>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n) asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes) asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes) asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define LDMATRIX_X1(R, addr) asm volatile("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
#define LDMATRIX_X1_T(R, addr) asm volatile("ldmatrix.sync.aligned.x1.trans.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2_T(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4_T(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1) asm volatile("mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n" : "=r"(RD0), "=r"(RD1) : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0), "r"(RC1))

HOST_DEVICE_INLINE 
int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

// only 1 warp per block(32 threads), m16n8k16. A, B, C: all row_major.
template<const int MMA_M=16, const int MMA_N=8, const int MMA_K=16>
__global__ void hgemm_mma_m16n8k16_naive_kernel(half* A, half* B, half* C, 
                                                int M, int N, int K) {
  const int bx = blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, MMA_K);
  constexpr int BM = MMA_M; // 16
  constexpr int BN = MMA_N; // 8
  constexpr int BK = MMA_K; // 16

  __shared__ half s_a[MMA_M][MMA_K]; // 16x16
  __shared__ half s_b[MMA_K][MMA_N]; // 16x8
  __shared__ half s_c[MMA_M][MMA_N]; // 16x8

  const int tid = threadIdx.y * blockDim.x + threadIdx.x; // within block
  const int warp_id = tid / WARP_SIZE; // 0
  const int lane_id = tid % WARP_SIZE; // 0~31

  // s_a[16][16], 每行16，每线程load 8，需要2线程，共16行，需2x16=32线程
  const int load_smem_a_m = tid / 2; // row 0~15
  const int load_smem_a_k = (tid % 2) * 8; // col 0,8
  // s_b[16][8], 每行8，每线程load 8，需要1线程，共16行，需16线程，只需一半线程加载
  const int load_smem_b_k = tid; // row 0~31, but only use 0~15
  const int load_smem_b_n = 0; // col 0
  const int load_gmem_a_m = by * BM + load_smem_a_m; // global m
  const int load_gmem_b_n = bx * BN + load_smem_b_n; // global n
  if (load_gmem_a_m >= M && load_gmem_b_n >= N) return;

  uint32_t RC[2] = {0, 0};

  #pragma unroll
  for (int k = 0; k < NUM_K_TILES; ++k) {
    // gmem_a -> smem_a
    int load_gmem_a_k = k * MMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    LDST128BITS(s_a[load_smem_a_m][load_smem_a_k]) = (
      LDST128BITS(A[load_gmem_a_addr]));

    // gmem_b -> smem_b
    if (lane_id < MMA_K) {
      int load_gmem_b_k = k * MMA_K + load_smem_b_k; // global row of b
      int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 
      LDST128BITS(s_b[load_smem_b_k][load_smem_b_n]) = (
        LDST128BITS(B[load_gmem_b_addr]));
    }
    __syncthreads(); 

    uint32_t RA[4];
    uint32_t RB[2];
    
    // ldmatrix for s_a, ldmatrix.trans for s_b.
    uint32_t load_smem_a_addr = __cvta_generic_to_shared(
      &s_a[lane_id % 16][(lane_id / 16) * 8]); // (0,1)*8 -> 0,8
    LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], load_smem_a_addr);
    uint32_t load_smem_b_addr = __cvta_generic_to_shared(
      &s_b[lane_id % 16][0]);
    LDMATRIX_X2_T(RB[0], RB[1], load_smem_b_addr);

    HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0], RC[1]);

    __syncthreads();
  }
  
  // s_c[16][8], https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
  // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
  // [0~7][0~3 u32 -> 0~7 f16], [8~15][0~3 u32 -> 0~7 f16]
  LDST32BITS(s_c[lane_id / 4    ][(lane_id % 4) * 2]) = LDST32BITS(RC[0]); 
  LDST32BITS(s_c[lane_id / 4 + 8][(lane_id % 4) * 2]) = LDST32BITS(RC[1]);

  __syncthreads();

  // store s_c[16][8]
  if (lane_id < MMA_M) {
    // store 128 bits per memory issue.
    int store_gmem_c_m = by * BM + lane_id;
    int store_gmem_c_n = bx * BN;
    int store_gmem_c_addr = store_gmem_c_m * N + store_gmem_c_n;
    LDST128BITS(C[store_gmem_c_addr]) = (LDST128BITS(s_c[lane_id][0]));
  }
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)           \
if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) { \
  throw std::runtime_error("Tensor size mismatch!");  \
}

// only 1 warp per block(32 threads), m16n8k16. A, B, C: all row_major.
void hgemm_mma_m16n8k16_naive(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int MMA_M = 16;
  constexpr int MMA_N = 8;
  constexpr int MMA_K = 16; 

  dim3 block(WARP_SIZE);
  dim3 grid(div_ceil(N, MMA_N), div_ceil(M, MMA_M));
 
  hgemm_mma_m16n8k16_naive_kernel<
    MMA_M, MMA_N, MMA_K><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}
