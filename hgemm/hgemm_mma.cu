#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>
#include <torch/types.h>
#include <torch/extension.h>
using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2*>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n) asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes) asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes) asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
// Support A and B matrix with row-major inorder to compare with the kernels using CUDA Cores in
// hgemm.cu and hgemm_async.cu. 


HOST_DEVICE_INLINE 
int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

// only 1 warp per block(32 threads), m16n16k16. A, B, C: all row_major.
template<const int WMMA_M=16, const int WMMA_N=16, const int WMMA_K=16>
__global__ void hgemm_mma_m16n16k16_naive_kernel(half* A, half* B, half* C, 
                                                int M, int N, int K) {

}
