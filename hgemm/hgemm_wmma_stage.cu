#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>
#include <torch/types.h>
#include <torch/extension.h>
using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2*>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n) asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes) asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes) asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
// Support A and B matrix with row-major inorder to compare with the kernels using CUDA Cores in
// hgemm.cu and hgemm_async.cu. 


HOST_DEVICE_INLINE 
int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }


// stage2/3/4 (stage2=double buffers+copy async)
template<const int WMMA_M=16, const int WMMA_N=16, const int WMMA_K=16, 
         const int WMMA_TILE_M=4, const int WMMA_TILE_N=2, 
         const int WARP_TILE_M=2, const int WARP_TILE_N=4,
         const int K_STAGE=3, const int OFFSET=0>
__global__ void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel(
  half* A, half* B, half* C, int M, int N, int K) {
  // 256 threads(8 warps) per block.
  const int bx = blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, WMMA_K);
  constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M; // 16x4*2=128
  constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N; // 16x2*4=128
  constexpr int BK = WMMA_K; // 16
  // 16x128x2=4KB, 4xK_STAGE=4x(2,3,4)=(8,12,16)KB, (8,12,16)KBx2=16,24,32KB
  // padding to reduce bank conflicts.16,24,32KB+8KB=24,32,40KB
  __shared__ half s_a[K_STAGE][BM][BK+OFFSET], s_b[K_STAGE][BK][BN+OFFSET]; 
 
  // 要保证相同的warp下thread执行相同的指令
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int warp_id = tid / WARP_SIZE; // 0~7 warp_id within block
  const int lane_id = tid % WARP_SIZE; // 0~31
  const int warp_m = warp_id / 2; // 0,1,2,3
  const int warp_n = warp_id % 2; // 0,1
  
  // 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=8 按行读取 A行主序
  // 对于s_a每行16个数据，每个线程读取8个，需要2个线程；总共128行，需要128x2刚好256线程
  int load_smem_a_m = tid / 2; // row 0~127
  int load_smem_a_k = (tid % 2 == 0) ? 0 : 8; // col 0,8
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=16 BN=128 按行读取 B行主序
  // 对于s_b每行128个数据，每个线程读8个数据，需要16个线程；总共16行，需要16x16=256个线程
  int load_smem_b_k = tid / 16; // row 0~15
  int load_smem_b_n = (tid % 16) * 8; // col 0,8,...,120
  // 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c

  wmma::fragment<wmma::accumulator, 
                 WMMA_M, WMMA_N, WMMA_K, 
                 half> C_frag[WARP_TILE_M][WARP_TILE_N];
  
  // fragment reuse.
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, 
                 wmma::row_major> A_frag[WARP_TILE_M];
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, 
                 wmma::row_major> B_frag[WARP_TILE_N];

  #pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      wmma::fill_fragment(C_frag[i][j], 0.0);
    }
  }

  #pragma unroll
  for (int k = 0; k < (K_STAGE - 1); ++k) { // 0, 1
    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 

    uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
      &s_a[k][load_smem_a_m][load_smem_a_k]);
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
      &s_b[k][load_smem_b_k][load_smem_b_n]);
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);

    CP_ASYNC_COMMIT_GROUP();
  }

  CP_ASYNC_WAIT_GROUP(K_STAGE-2); // s2->0, s3->1, s4->2
  __syncthreads(); 

  #pragma unroll
  for (int k = (K_STAGE - 1); k < NUM_K_TILES; k++) { // start from 2
    int smem_sel = (k + 1) % K_STAGE; // s3 k 2->0, k 3->1, k 4->2...
    int smem_sel_next = k % K_STAGE;  // s3 k 2->2, k 3->0, k 4->1...

    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 

    // load stage 2, k start from 2
    uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
      &s_a[smem_sel_next][load_smem_a_m][load_smem_a_k]);
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
      &s_b[smem_sel_next][load_smem_b_k][load_smem_b_n]);
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);
    CP_ASYNC_COMMIT_GROUP();
    
    // compute stage 0
    #pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
      const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      wmma::load_matrix_sync(A_frag[i], &s_a[smem_sel][warp_smem_a_m][0], BK+OFFSET); 
    }

    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
      const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_M) + j * WMMA_N;
      wmma::load_matrix_sync(B_frag[j], &s_b[smem_sel][0][warp_smem_b_n], BN+OFFSET);
    }

    #pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      #pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
      }
    }

    // make sure all memory issues ready, for final k iters.
    if (k == (NUM_K_TILES - 1)) {
      CP_ASYNC_WAIT_GROUP(0);
    } else {
      CP_ASYNC_WAIT_GROUP(K_STAGE-2); // s2->0, s3->1, s4->2
    }
    __syncthreads(); 
  }
  
  // processing last (K_STAGE-1) stage(k)
  {
    #pragma unroll
    for (int k = 0; k < (K_STAGE - 1); k++) {
      int stage_sel = ((NUM_K_TILES - (K_STAGE - 1) + k) % K_STAGE);
      #pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
        const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
        wmma::load_matrix_sync(A_frag[i], &s_a[stage_sel][warp_smem_a_m][0], BK+OFFSET); 
      }

      #pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
        const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_M) + j * WMMA_N;
        wmma::load_matrix_sync(B_frag[j], &s_b[stage_sel][0][warp_smem_b_n], BN+OFFSET);
      }
      
      #pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        #pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j) {
          wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
        }
      }
    }
  }

  // finally, store back to C matrix.
  #pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      const int store_gmem_a_m = by * BM + warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      const int store_gmem_a_n = bx * BN + warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
      wmma::store_matrix_sync(C + store_gmem_a_m * N + store_gmem_a_n, C_frag[i][j], N, 
                              wmma::mem_row_major);
    }
  }
}

// TODO: stage with 256x128 block

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)           \
if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) { \
  throw std::runtime_error("Tensor size mismatch!");  \
}

// stage2
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage2(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 2;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 4 * 2 * 32 = 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_M * WMMA_TILE_M * WARP_TILE_M), 
            div_ceil(M, WMMA_N * WMMA_TILE_N * WARP_TILE_N));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 0><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// stage2 + padding
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage2_offset(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 2;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 4 * 2 * 32 = 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_M * WMMA_TILE_M * WARP_TILE_M), 
            div_ceil(M, WMMA_N * WMMA_TILE_N * WARP_TILE_N));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 8><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// stage3
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage3(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 3;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 4 * 2 * 32 = 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_M * WMMA_TILE_M * WARP_TILE_M), 
            div_ceil(M, WMMA_N * WMMA_TILE_N * WARP_TILE_N));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 0><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// stage3 + padding
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage3_offset(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 3;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 4 * 2 * 32 = 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_M * WMMA_TILE_M * WARP_TILE_M), 
            div_ceil(M, WMMA_N * WMMA_TILE_N * WARP_TILE_N));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 8><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// stage4
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage4(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 4;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 4 * 2 * 32 = 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_M * WMMA_TILE_M * WARP_TILE_M), 
            div_ceil(M, WMMA_N * WMMA_TILE_N * WARP_TILE_N));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 0><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// stage4 + padding
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage4_offset(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 4;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 4 * 2 * 32 = 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_M * WMMA_TILE_M * WARP_TILE_M), 
            div_ceil(M, WMMA_N * WMMA_TILE_N * WARP_TILE_N));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 8><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}
