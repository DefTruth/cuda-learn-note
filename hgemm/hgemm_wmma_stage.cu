#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>
#include <torch/types.h>
#include <torch/extension.h>
using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2*>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n) asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes) asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes) asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
// Support A and B matrix with row-major inorder to compare with the kernels using CUDA Cores in
// hgemm.cu and hgemm_async.cu. 


HOST_DEVICE_INLINE 
int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

// stage2/3/4 (stage2=double buffers+copy async)
template<const int WMMA_M=16, const int WMMA_N=16, const int WMMA_K=16, 
         const int WMMA_TILE_M=4, const int WMMA_TILE_N=2, 
         const int WARP_TILE_M=2, const int WARP_TILE_N=4,
         const int K_STAGE=3, const int OFFSET=0>
__global__ void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel(
  half* A, half* B, half* C, int M, int N, int K) {
  // 256 threads(8 warps) per block.
  const int bx = blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, WMMA_K);
  constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M; // 16x4*2=128
  constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N; // 16x2*4=128
  constexpr int BK = WMMA_K; // 16
  // s2: 2*128*(16+8)*2=12KB, 2*16*(128+8)*2=8.50KB,  ~21KB
  // s3: 3*128*(16+8)*2=18KB, 3*16*(128+8)*2=12.75KB, ~31KB
  // s4: 4*128*(16+8)*2=24KB, 4*16*(128+8)*2=17KB,    ~41KB
  __shared__ half s_a[K_STAGE][BM][BK+OFFSET], s_b[K_STAGE][BK][BN+OFFSET]; 
 
  // 要保证相同的warp下thread执行相同的指令
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int warp_id = tid / WARP_SIZE; // 0~7 warp_id within block
  const int warp_m = warp_id / 2; // 0,1,2,3
  const int warp_n = warp_id % 2; // 0,1
  
  // 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=16 按行读取 A行主序
  // 对于s_a每行16个数据，每个线程读取8个，需要2个线程；总共128行，需要128x2刚好256线程
  int load_smem_a_m = tid / 2; // row 0~127
  int load_smem_a_k = (tid % 2 == 0) ? 0 : 8; // col 0,8
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=16 BN=128 按行读取 B行主序
  // 对于s_b每行128个数据，每个线程读8个数据，需要16个线程；总共16行，需要16x16=256个线程
  int load_smem_b_k = tid / 16; // row 0~15
  int load_smem_b_n = (tid % 16) * 8; // col 0,8,...,120
  // 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c

  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> 
  C_frag[WARP_TILE_M][WARP_TILE_N];
  
  #pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      wmma::fill_fragment(C_frag[i][j], 0.0);
    }
  }

  #pragma unroll
  for (int k = 0; k < (K_STAGE - 1); ++k) { // 0, 1
    // k * WMMA_K, WMMA_K=16 -> (k << 4)
    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 

    uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
      &s_a[k][load_smem_a_m][load_smem_a_k]);
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
      &s_b[k][load_smem_b_k][load_smem_b_n]);
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);

    CP_ASYNC_COMMIT_GROUP();
  }

  CP_ASYNC_WAIT_GROUP(K_STAGE-2); // s2->0, s3->1, s4->2
  __syncthreads(); 

  #pragma unroll
  for (int k = (K_STAGE - 1); k < NUM_K_TILES; k++) { 
    // s2/4 can use bitwise ops but s3 can not, so, we use mod
    // ops for all stages kernel. s2: (k + 1)&1, s4: (k + 1)&3
    // s3: (k + 1) % 3
    int smem_sel = (k + 1) % K_STAGE; // s3 k 2->0, k 3->1, k 4->2...
    int smem_sel_next = k % K_STAGE;  // s3 k 2->2, k 3->0, k 4->1...

    // k * WMMA_K, WMMA_K=16 -> (k << 4)
    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 

    // load stage 2, k start from 2
    uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
      &s_a[smem_sel_next][load_smem_a_m][load_smem_a_k]);
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
      &s_b[smem_sel_next][load_smem_b_k][load_smem_b_n]);
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);
    CP_ASYNC_COMMIT_GROUP();

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, 
                   wmma::row_major> A_frag[WARP_TILE_M];
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, 
                   wmma::row_major> B_frag[WARP_TILE_N];
    
    // compute stage 0
    #pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
      const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      wmma::load_matrix_sync(A_frag[i], &s_a[smem_sel][warp_smem_a_m][0], BK+OFFSET); 
    }

    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
      const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
      wmma::load_matrix_sync(B_frag[j], &s_b[smem_sel][0][warp_smem_b_n], BN+OFFSET);
    }

    #pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      #pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
      }
    }

    CP_ASYNC_WAIT_GROUP(K_STAGE-2);
    __syncthreads(); 
  }
  
  // make sure all memory issues ready.
  CP_ASYNC_WAIT_GROUP(0);
  __syncthreads(); 
  // processing last (K_STAGE-1) k iters.
  {
    #pragma unroll
    for (int k = 0; k < (K_STAGE - 1); k++) {
      const int stage_sel = ((NUM_K_TILES - (K_STAGE - 1) + k) % K_STAGE);
      wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, 
                     wmma::row_major> A_frag[WARP_TILE_M];
      wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, 
                     wmma::row_major> B_frag[WARP_TILE_N];
    
      #pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
        const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
        wmma::load_matrix_sync(A_frag[i], &s_a[stage_sel][warp_smem_a_m][0], BK+OFFSET); 
      }

      #pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
        const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
        wmma::load_matrix_sync(B_frag[j], &s_b[stage_sel][0][warp_smem_b_n], BN+OFFSET);
      }
      
      #pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        #pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j) {
          wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
        }
      }
    }
  }

  // finally, store back to C matrix.
  #pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      const int store_gmem_a_m = by * BM + warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      const int store_gmem_a_n = bx * BN + warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
      wmma::store_matrix_sync(C + store_gmem_a_m * N + store_gmem_a_n, C_frag[i][j], N, 
                              wmma::mem_row_major);
    }
  }
}

// stage with 256x128 block
// stage2/3/4 (stage2=double buffers+copy async)
template<const int WMMA_M=16, const int WMMA_N=16, const int WMMA_K=16, 
         const int WMMA_TILE_M=4, const int WMMA_TILE_N=2, 
         const int WARP_TILE_M=4, const int WARP_TILE_N=4,
         const int K_STAGE=3, const int OFFSET=0>
__global__ void hgemm_wmma_m16n16k16_mma4x2_warp4x4_stages_kernel(
  half* A, half* B, half* C, int M, int N, int K) {
  // 256 threads(8 warps) per block.
  const int bx = blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, WMMA_K);
  constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M; // 16x4*4=256
  constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N; // 16x2*4=128
  constexpr int BK = WMMA_K; // 16
  // s2: 2*256*(16+8)*2=24KB, 2*16*(128+8)*2=8.50KB,  ~33KB
  // s3: 3*256*(16+8)*2=36KB, 3*16*(128+8)*2=12.75KB, ~50KB
  // s4: 4*256*(16+8)*2=48KB, 4*16*(128+8)*2=17KB,    ~65KB
  __shared__ half s_a[K_STAGE][BM][BK+OFFSET], s_b[K_STAGE][BK][BN+OFFSET]; 
 
  // 要保证相同的warp下thread执行相同的指令
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int warp_id = tid / WARP_SIZE; // 0~7 warp_id within block
  const int warp_m = warp_id / 2; // 0,1,2,3
  const int warp_n = warp_id % 2; // 0,1
  
  // 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=256 BK=16 按行读取 A行主序
  // 对于s_a每行16个数据，每个线程读取16个，需要1个线程；总共256行，256x1刚好256线程
  int load_smem_a_m = tid; // row 0~255
  int load_smem_a_k = 0; // col 0,16
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=16 BN=128 按行读取 B行主序
  // 对于s_b每行128个数据，每个线程读8个数据，需要16个线程；总共16行，需要16x16=256个线程
  int load_smem_b_k = tid / 16; // row 0~15
  int load_smem_b_n = (tid % 16) * 8; // col 0,8,...,120
  // 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c

  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> 
  C_frag[WARP_TILE_M][WARP_TILE_N];
  
  #pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      wmma::fill_fragment(C_frag[i][j], 0.0);
    }
  }

  #pragma unroll
  for (int k = 0; k < (K_STAGE - 1); ++k) { // 0, 1
    // k * WMMA_K, WMMA_K=16 -> (k << 4)
    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 

    uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
      &s_a[k][load_smem_a_m][load_smem_a_k]);
    CP_ASYNC_CG(load_smem_a_ptr,      &A[load_gmem_a_addr    ], 16);
    CP_ASYNC_CG(load_smem_a_ptr + 16, &A[load_gmem_a_addr + 8], 16);

    uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
      &s_b[k][load_smem_b_k][load_smem_b_n]);
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);

    CP_ASYNC_COMMIT_GROUP();
  }

  CP_ASYNC_WAIT_GROUP(K_STAGE-2); // s2->0, s3->1, s4->2
  __syncthreads(); 

  #pragma unroll
  for (int k = (K_STAGE - 1); k < NUM_K_TILES; k++) { 
    // s2/4 can use bitwise ops but s3 can not, so, we use mod
    // ops for all stages kernel. s2: (k + 1)&1, s4: (k + 1)&3
    // s3: (k + 1) % 3
    int smem_sel = (k + 1) % K_STAGE; // s3 k 2->0, k 3->1, k 4->2...
    int smem_sel_next = k % K_STAGE;  // s3 k 2->2, k 3->0, k 4->1...

    // k * WMMA_K, WMMA_K=16 -> (k << 4)
    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 

    // load stage 2, k start from 2
    uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
      &s_a[smem_sel_next][load_smem_a_m][load_smem_a_k]);
    CP_ASYNC_CG(load_smem_a_ptr,      &A[load_gmem_a_addr    ], 16);
    CP_ASYNC_CG(load_smem_a_ptr + 16, &A[load_gmem_a_addr + 8], 16);

    uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
      &s_b[smem_sel_next][load_smem_b_k][load_smem_b_n]);
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);
    CP_ASYNC_COMMIT_GROUP();

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, 
                   wmma::row_major> A_frag[WARP_TILE_M];
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, 
                   wmma::row_major> B_frag[WARP_TILE_N];
    
    // compute stage 0
    #pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
      const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      wmma::load_matrix_sync(A_frag[i], &s_a[smem_sel][warp_smem_a_m][0], BK+OFFSET); 
    }

    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
      const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
      wmma::load_matrix_sync(B_frag[j], &s_b[smem_sel][0][warp_smem_b_n], BN+OFFSET);
    }

    #pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      #pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
      }
    }

    CP_ASYNC_WAIT_GROUP(K_STAGE-2);
    __syncthreads(); 
  }
  
  // make sure all memory issues ready.
  CP_ASYNC_WAIT_GROUP(0);
  __syncthreads(); 
  // processing last (K_STAGE-1) k iters.
  {
    #pragma unroll
    for (int k = 0; k < (K_STAGE - 1); k++) {
      const int stage_sel = ((NUM_K_TILES - (K_STAGE - 1) + k) % K_STAGE);
      wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, 
                     wmma::row_major> A_frag[WARP_TILE_M];
      wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, 
                     wmma::row_major> B_frag[WARP_TILE_N];
    
      #pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
        const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
        wmma::load_matrix_sync(A_frag[i], &s_a[stage_sel][warp_smem_a_m][0], BK+OFFSET); 
      }

      #pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
        const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
        wmma::load_matrix_sync(B_frag[j], &s_b[stage_sel][0][warp_smem_b_n], BN+OFFSET);
      }
      
      #pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        #pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j) {
          wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
        }
      }
    }
  }

  // finally, store back to C matrix.
  #pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      const int store_gmem_a_m = by * BM + warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      const int store_gmem_a_n = bx * BN + warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
      wmma::store_matrix_sync(C + store_gmem_a_m * N + store_gmem_a_n, C_frag[i][j], N, 
                              wmma::mem_row_major);
    }
  }
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)           \
if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) { \
  throw std::runtime_error("Tensor size mismatch!");  \
}

// stage2
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage2(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 2;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 2 * 4 * 32 = 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_N * WMMA_TILE_N * WARP_TILE_N), 
            div_ceil(M, WMMA_M * WMMA_TILE_M * WARP_TILE_M));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 0><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// stage2 + padding
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage2_offset(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 2;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 2 * 4 * 32 = 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_N * WMMA_TILE_N * WARP_TILE_N), 
            div_ceil(M, WMMA_M * WMMA_TILE_M * WARP_TILE_M));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 8><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// stage3
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage3(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 3;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_N * WMMA_TILE_N * WARP_TILE_N), 
            div_ceil(M, WMMA_M * WMMA_TILE_M * WARP_TILE_M));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 0><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// stage3 + padding
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage3_offset(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 3;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_N * WMMA_TILE_N * WARP_TILE_N), 
            div_ceil(M, WMMA_M * WMMA_TILE_M * WARP_TILE_M));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 8><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// stage4
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage4(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 4;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_N * WMMA_TILE_N * WARP_TILE_N), 
            div_ceil(M, WMMA_M * WMMA_TILE_M * WARP_TILE_M));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 0><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// stage4 + padding
void hgemm_wmma_m16n16k16_mma4x2_warp2x4_stage4_offset(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 4;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_N * WMMA_TILE_N * WARP_TILE_N), 
            div_ceil(M, WMMA_M * WMMA_TILE_M * WARP_TILE_M));
 
  hgemm_wmma_m16n16k16_mma4x2_warp2x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 8><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// 256x128
// stage2
void hgemm_wmma_m16n16k16_mma4x2_warp4x4_stage2(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 4;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 2;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE);

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_N * WMMA_TILE_N * WARP_TILE_N), 
            div_ceil(M, WMMA_M * WMMA_TILE_M * WARP_TILE_M));
 
  hgemm_wmma_m16n16k16_mma4x2_warp4x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 0><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}

// stage2 + padding
void hgemm_wmma_m16n16k16_mma4x2_warp4x4_stage2_offset(
  torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 16;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2; 
  constexpr int WARP_TILE_M = 4;
  constexpr int WARP_TILE_N = 4;
  constexpr int K_STAGE = 2;
  constexpr int NUM_THREADS= (
    WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE);

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, WMMA_N * WMMA_TILE_N * WARP_TILE_N), 
            div_ceil(M, WMMA_M * WMMA_TILE_M * WARP_TILE_M));
 
  hgemm_wmma_m16n16k16_mma4x2_warp4x4_stages_kernel<
    WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, 
    WARP_TILE_M, WARP_TILE_N, K_STAGE, 8><<<grid, block>>>(
    reinterpret_cast<half*>(a.data_ptr()),
    reinterpret_cast<half*>(b.data_ptr()),
    reinterpret_cast<half*>(c.data_ptr()),
    M, N, K
  );
}
