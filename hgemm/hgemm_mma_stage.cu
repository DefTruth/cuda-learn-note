#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>
#include <torch/types.h>
#include <torch/extension.h>
using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2*>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n) asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes) asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes) asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define LDMATRIX_X1(R, addr) asm volatile("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
#define LDMATRIX_X1_T(R, addr) asm volatile("ldmatrix.sync.aligned.x1.trans.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2_T(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4_T(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1) asm volatile("mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n" : "=r"(RD0), "=r"(RD1) : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0), "r"(RC1))

HOST_DEVICE_INLINE 
int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

// 128x128, mma2x4, warp4x4(64,32,16), stages, block swizzle
template<const int MMA_M=16, 
         const int MMA_N=8, 
         const int MMA_K=16,
         const int MMA_TILE_M=2,
         const int MMA_TILE_N=4,
         const int WARP_TILE_M=4,
         const int WARP_TILE_N=4,
         const int A_PAD=0, 
         const int B_PAD=0,
         const int K_STAGE=2, 
         const bool BLOCK_SWIZZLE=false>
__global__ void  __launch_bounds__(256) 
hgemm_mma_m16n8k16_mma2x4_warp4x4_stages_kernel(
  half* A, half* B, half* C, int M, int N, int K) {
  // BLOCK_SWIZZLE 0/1 control use block swizzle or not.
  const int bx = ((int) BLOCK_SWIZZLE) * blockIdx.z * gridDim.x + blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, MMA_K);
  constexpr int BM = MMA_M * MMA_TILE_M * WARP_TILE_M; // 16*2*4=128
  constexpr int BN = MMA_N * MMA_TILE_N * WARP_TILE_N; // 8*4*4=128
  constexpr int BK = MMA_K; // 16

  __shared__ half s_a[K_STAGE][BM][BK+A_PAD]; // 128*16*2=4KB
  __shared__ half s_b[K_STAGE][BK][BN+B_PAD]; // 16*128*2=4KB, 16*(128+16)*2=4.5KB
  constexpr int s_a_stage_offset = BM * (BK + A_PAD);
  constexpr int s_b_stage_offset = BK * (BN + B_PAD);

  const int tid = threadIdx.y * blockDim.x + threadIdx.x; // within block
  const int warp_id = tid / WARP_SIZE; // 0~7 warp_id within block
  const int lane_id = tid % WARP_SIZE; // 0~31
  const int warp_m = warp_id % 2; // 0,1
  const int warp_n = warp_id / 2; // 0,1,2,3

  // 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=16 按行读取 A行主序
  // 对于s_a每行16个数据，每个线程读取8个，需要2个线程；总共128行，需要128x2刚好256线程
  int load_smem_a_m = tid / 2; // row 0~127
  int load_smem_a_k = (tid % 2 == 0) ? 0 : 8; // col 0,8
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=16 BN=128 按行读取 B行主序
  // 对于s_b每行128个数据，每个线程读8个数据，需要16个线程；总共16行，需要16x16=256个线程
  int load_smem_b_k = tid / 16; // row 0~15
  int load_smem_b_n = (tid % 16) * 8; // col 0,8,...,120
  // 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c

  uint32_t RC[WARP_TILE_M][WARP_TILE_N][2];
  #pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      RC[i][j][0] = 0;
      RC[i][j][1] = 0;
    }
  }

  // may avoid cvta overhead ? only cvta smem base ptr once for cp.async.
  uint32_t smem_a_base_ptr = __cvta_generic_to_shared(s_a);
  uint32_t smem_b_base_ptr = __cvta_generic_to_shared(s_b);

  #pragma unroll
  for (int k = 0; k < (K_STAGE - 1); ++k) { // 0, 1
    // k * WMMA_K, WMMA_K=16 -> (k << 4)
    int load_gmem_a_k = k * BK + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * BK + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 

    uint32_t load_smem_a_ptr = (
      smem_a_base_ptr + (k * s_a_stage_offset + 
                         load_smem_a_m * (BK + A_PAD) + 
                         load_smem_a_k) * sizeof(half)
    );
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr = (
      smem_b_base_ptr + (k * s_b_stage_offset + 
                         load_smem_b_k * (BN + B_PAD) + 
                         load_smem_b_n) * sizeof(half)
    );
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);

    CP_ASYNC_COMMIT_GROUP();
  }

  CP_ASYNC_WAIT_GROUP(K_STAGE-2); // s2->0, s3->1, s4->2
  __syncthreads(); 

  #pragma unroll
  for (int k = (K_STAGE - 1); k < NUM_K_TILES; ++k) {
    // gmem -> smem
    // s2/4 can use bitwise ops but s3 can not, so, we use mod
    // ops for all stages kernel. s2: (k + 1)&1, s4: (k + 1)&3
    // s3: (k + 1) % 3
    int smem_sel = (k + 1) % K_STAGE; // s3 k 2->0, k 3->1, k 4->2...
    int smem_sel_next = k % K_STAGE;  // s3 k 2->2, k 3->0, k 4->1...

    int load_gmem_a_k = k * BK + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * BK + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 

    uint32_t load_smem_a_ptr = (
      smem_a_base_ptr + (smem_sel_next * s_a_stage_offset + 
                         load_smem_a_m * (BK + A_PAD) + 
                         load_smem_a_k) * sizeof(half)
    );
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr = (
      smem_b_base_ptr + (smem_sel_next * s_b_stage_offset + 
                         load_smem_b_k * (BN + B_PAD) + 
                         load_smem_b_n) * sizeof(half)
    );
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);
    CP_ASYNC_COMMIT_GROUP();
    
    // ldmatrix for s_a, ldmatrix.trans for s_b.
    uint32_t RA[WARP_TILE_M][4];
    uint32_t RB[WARP_TILE_N][2];

    // smem -> reg
    #pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      int warp_smem_a_m = warp_m * (MMA_M * WARP_TILE_M) + i * MMA_M;
      int lane_smem_a_m = warp_smem_a_m + lane_id % 16; // 0~15
      int lane_smem_a_k = (lane_id / 16) * 8; // 0,8
      uint32_t lane_smem_a_ptr = __cvta_generic_to_shared(
        &s_a[smem_sel][lane_smem_a_m][lane_smem_a_k]);
      LDMATRIX_X4(RA[i][0], RA[i][1], RA[i][2], RA[i][3], lane_smem_a_ptr);
    }

    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      int warp_smem_b_n = warp_n * (MMA_N * WARP_TILE_N) + j * MMA_N;
      int lane_smem_b_k = lane_id % 16;  // 0~15
      int lane_smem_b_n = warp_smem_b_n; // 0, MMA_N=8
      uint32_t lane_smem_b_ptr = __cvta_generic_to_shared(
        &s_b[smem_sel][lane_smem_b_k][lane_smem_b_n]);
      LDMATRIX_X2_T(RB[j][0], RB[j][1], lane_smem_b_ptr);
    }
    
    // MMA compute
    #pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      #pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        HMMA16816(RC[i][j][0], RC[i][j][1], 
                  RA[i][0], RA[i][1], RA[i][2], RA[i][3], 
                  RB[j][0], RB[j][1], 
                  RC[i][j][0], RC[i][j][1]);
      }
    }

    CP_ASYNC_WAIT_GROUP(K_STAGE-2);
    __syncthreads(); 
  }

  // make sure all memory issues ready.
  if ((K_STAGE - 2) > 0) {
    CP_ASYNC_WAIT_GROUP(0);
    __syncthreads(); 
  }

  // processing last (K_STAGE-1) k iters.
  {
    #pragma unroll
    for (int k = 0; k < (K_STAGE - 1); k++) {
      int stage_sel = ((NUM_K_TILES - (K_STAGE - 1) + k) % K_STAGE);
      // ldmatrix for s_a, ldmatrix.trans for s_b.
      uint32_t RA[WARP_TILE_M][4];
      uint32_t RB[WARP_TILE_N][2];

      // smem -> reg
      #pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        int warp_smem_a_m = warp_m * (MMA_M * WARP_TILE_M) + i * MMA_M;
        int lane_smem_a_m = warp_smem_a_m + lane_id % 16; // 0~15
        int lane_smem_a_k = (lane_id / 16) * 8; // 0,8
        uint32_t lane_smem_a_ptr = __cvta_generic_to_shared(
          &s_a[stage_sel][lane_smem_a_m][lane_smem_a_k]);
        LDMATRIX_X4(RA[i][0], RA[i][1], RA[i][2], RA[i][3], lane_smem_a_ptr);
      }

      #pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        int warp_smem_b_n = warp_n * (MMA_N * WARP_TILE_N) + j * MMA_N;
        int lane_smem_b_k = lane_id % 16;  // 0~15
        int lane_smem_b_n = warp_smem_b_n; // 0, MMA_N=8
        uint32_t lane_smem_b_ptr = __cvta_generic_to_shared(
          &s_b[stage_sel][lane_smem_b_k][lane_smem_b_n]);
        LDMATRIX_X2_T(RB[j][0], RB[j][1], lane_smem_b_ptr);
      }

      // MMA compute
      #pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        #pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j) {
          HMMA16816(RC[i][j][0], RC[i][j][1], 
                    RA[i][0], RA[i][1], RA[i][2], RA[i][3], 
                    RB[j][0], RB[j][1], 
                    RC[i][j][0], RC[i][j][1]);
        }
      }
    }
  }

  // reg -> gmem, MMA_MxMMA_N=16x8
  #pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      int store_warp_smem_c_m = warp_m * (MMA_M * WARP_TILE_M) + i * MMA_M;
      int store_warp_smem_c_n = warp_n * (MMA_N * WARP_TILE_N) + j * MMA_N;
      // mapping lane smem index -> global index.
      // [16][8], https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
      // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
      // [0~7][0~3 u32 -> 0~7 f16], [8~15][0~3 u32 -> 0~7 f16]
      int store_lane_gmem_c_m = by * BM + store_warp_smem_c_m + lane_id / 4;
      int store_lane_gmem_c_n = bx * BN + store_warp_smem_c_n + (lane_id % 4) * 2;
      int store_gmem_c_addr_0 = store_lane_gmem_c_m * N + store_lane_gmem_c_n;
      int store_gmem_c_addr_1 = (store_lane_gmem_c_m + 8) * N + store_lane_gmem_c_n;
      // TODO: how to use LDST128BITS here ?
      LDST32BITS(C[store_gmem_c_addr_0]) = LDST32BITS(RC[i][j][0]); 
      LDST32BITS(C[store_gmem_c_addr_1]) = LDST32BITS(RC[i][j][1]); 
    }
  }
}

// 128x128, mma2x4, warp4x4(64,32,16), stages, block swizzle, dsmem
template<const int MMA_M=16, 
         const int MMA_N=8, 
         const int MMA_K=16,
         const int MMA_TILE_M=2,
         const int MMA_TILE_N=4,
         const int WARP_TILE_M=4,
         const int WARP_TILE_N=4,
         const int A_PAD=0, 
         const int B_PAD=0,
         const int K_STAGE=2, 
         const bool BLOCK_SWIZZLE=false>
__global__ void  __launch_bounds__(256) 
hgemm_mma_m16n8k16_mma2x4_warp4x4_stages_dsmem_kernel(
  half* A, half* B, half* C, int M, int N, int K) {
  // BLOCK_SWIZZLE 0/1 control use block swizzle or not.
  const int bx = ((int) BLOCK_SWIZZLE) * blockIdx.z * gridDim.x + blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, MMA_K);
  constexpr int BM = MMA_M * MMA_TILE_M * WARP_TILE_M; // 16*2*4=128
  constexpr int BN = MMA_N * MMA_TILE_N * WARP_TILE_N; // 8*4*4=128
  constexpr int BK = MMA_K; // 16

  extern __shared__ half smem[]; 
  half* s_a = smem;
  half* s_b = smem + K_STAGE * BM * (BK + A_PAD);
  constexpr int s_a_stage_offset = BM * (BK + A_PAD);
  constexpr int s_b_stage_offset = BK * (BN + B_PAD);

  const int tid = threadIdx.y * blockDim.x + threadIdx.x; // within block
  const int warp_id = tid / WARP_SIZE; // 0~7 warp_id within block
  const int lane_id = tid % WARP_SIZE; // 0~31
  const int warp_m = warp_id % 2; // 0,1
  const int warp_n = warp_id / 2; // 0,1,2,3

  // 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=16 按行读取 A行主序
  // 对于s_a每行16个数据，每个线程读取8个，需要2个线程；总共128行，需要128x2刚好256线程
  int load_smem_a_m = tid / 2; // row 0~127
  int load_smem_a_k = (tid % 2 == 0) ? 0 : 8; // col 0,8
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=16 BN=128 按行读取 B行主序
  // 对于s_b每行128个数据，每个线程读8个数据，需要16个线程；总共16行，需要16x16=256个线程
  int load_smem_b_k = tid / 16; // row 0~15
  int load_smem_b_n = (tid % 16) * 8; // col 0,8,...,120
  // 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c

  uint32_t RC[WARP_TILE_M][WARP_TILE_N][2];
  #pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      RC[i][j][0] = 0;
      RC[i][j][1] = 0;
    }
  }

  // may avoid cvta overhead ? only cvta smem base ptr once for cp.async.
  uint32_t smem_a_base_ptr = __cvta_generic_to_shared(s_a);
  uint32_t smem_b_base_ptr = __cvta_generic_to_shared(s_b);

  #pragma unroll
  for (int k = 0; k < (K_STAGE - 1); ++k) { // 0, 1
    // k * WMMA_K, WMMA_K=16 -> (k << 4)
    int load_gmem_a_k = k * BK + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * BK + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 

    uint32_t load_smem_a_ptr = (
      smem_a_base_ptr + (k * s_a_stage_offset + 
                         load_smem_a_m * (BK + A_PAD) + 
                         load_smem_a_k) * sizeof(half)
    );
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr = (
      smem_b_base_ptr + (k * s_b_stage_offset + 
                         load_smem_b_k * (BN + B_PAD) + 
                         load_smem_b_n) * sizeof(half)
    );
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);

    CP_ASYNC_COMMIT_GROUP();
  }

  CP_ASYNC_WAIT_GROUP(K_STAGE-2); // s2->0, s3->1, s4->2
  __syncthreads(); 

  #pragma unroll
  for (int k = (K_STAGE - 1); k < NUM_K_TILES; ++k) {
    // gmem -> smem
    // s2/4 can use bitwise ops but s3 can not, so, we use mod
    // ops for all stages kernel. s2: (k + 1)&1, s4: (k + 1)&3
    // s3: (k + 1) % 3
    int smem_sel = (k + 1) % K_STAGE; // s3 k 2->0, k 3->1, k 4->2...
    int smem_sel_next = k % K_STAGE;  // s3 k 2->2, k 3->0, k 4->1...

    int load_gmem_a_k = k * BK + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * BK + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n; 

    uint32_t load_smem_a_ptr = (
      smem_a_base_ptr + (smem_sel_next * s_a_stage_offset + 
                         load_smem_a_m * (BK + A_PAD) + 
                         load_smem_a_k) * sizeof(half)
    );
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr = (
      smem_b_base_ptr + (smem_sel_next * s_b_stage_offset + 
                         load_smem_b_k * (BN + B_PAD) + 
                         load_smem_b_n) * sizeof(half)
    );
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);
    CP_ASYNC_COMMIT_GROUP();
    
    // ldmatrix for s_a, ldmatrix.trans for s_b.
    uint32_t RA[WARP_TILE_M][4];
    uint32_t RB[WARP_TILE_N][2];

    // smem -> reg
    #pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      int warp_smem_a_m = warp_m * (MMA_M * WARP_TILE_M) + i * MMA_M;
      int lane_smem_a_m = warp_smem_a_m + lane_id % 16; // 0~15
      int lane_smem_a_k = (lane_id / 16) * 8; // 0,8
      uint32_t lane_smem_a_ptr = (
        smem_a_base_ptr + (smem_sel * s_a_stage_offset + 
                           lane_smem_a_m * (BK + A_PAD) + 
                           lane_smem_a_k) * sizeof(half)
      );
      LDMATRIX_X4(RA[i][0], RA[i][1], RA[i][2], RA[i][3], lane_smem_a_ptr);
    }

    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      int warp_smem_b_n = warp_n * (MMA_N * WARP_TILE_N) + j * MMA_N;
      int lane_smem_b_k = lane_id % 16;  // 0~15
      int lane_smem_b_n = warp_smem_b_n; // 0, MMA_N=8
      uint32_t lane_smem_b_ptr = (
        smem_b_base_ptr + (smem_sel * s_b_stage_offset + 
                           lane_smem_b_k * (BN + B_PAD) + 
                           lane_smem_b_n) * sizeof(half)
      );
      LDMATRIX_X2_T(RB[j][0], RB[j][1], lane_smem_b_ptr);
    }
    
    // MMA compute
    #pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      #pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        HMMA16816(RC[i][j][0], RC[i][j][1], 
                  RA[i][0], RA[i][1], RA[i][2], RA[i][3], 
                  RB[j][0], RB[j][1], 
                  RC[i][j][0], RC[i][j][1]);
      }
    }

    CP_ASYNC_WAIT_GROUP(K_STAGE-2);
    __syncthreads(); 
  }

  // make sure all memory issues ready.
  if ((K_STAGE - 2) > 0) {
    CP_ASYNC_WAIT_GROUP(0);
    __syncthreads(); 
  }

  // processing last (K_STAGE-1) k iters.
  {
    #pragma unroll
    for (int k = 0; k < (K_STAGE - 1); k++) {
      int stage_sel = ((NUM_K_TILES - (K_STAGE - 1) + k) % K_STAGE);
      // ldmatrix for s_a, ldmatrix.trans for s_b.
      uint32_t RA[WARP_TILE_M][4];
      uint32_t RB[WARP_TILE_N][2];

      // smem -> reg
      #pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        int warp_smem_a_m = warp_m * (MMA_M * WARP_TILE_M) + i * MMA_M;
        int lane_smem_a_m = warp_smem_a_m + lane_id % 16; // 0~15
        int lane_smem_a_k = (lane_id / 16) * 8; // 0,8
        uint32_t lane_smem_a_ptr = (
          smem_a_base_ptr + (stage_sel * s_a_stage_offset + 
                             lane_smem_a_m * (BK + A_PAD) + 
                             lane_smem_a_k) * sizeof(half)
        );
        LDMATRIX_X4(RA[i][0], RA[i][1], RA[i][2], RA[i][3], lane_smem_a_ptr);
      }

      #pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        int warp_smem_b_n = warp_n * (MMA_N * WARP_TILE_N) + j * MMA_N;
        int lane_smem_b_k = lane_id % 16;  // 0~15
        int lane_smem_b_n = warp_smem_b_n; // 0, MMA_N=8
        uint32_t lane_smem_b_ptr = (
          smem_b_base_ptr + (stage_sel * s_b_stage_offset + 
                             lane_smem_b_k * (BN + B_PAD) + 
                             lane_smem_b_n) * sizeof(half)
        );
        LDMATRIX_X2_T(RB[j][0], RB[j][1], lane_smem_b_ptr);
      }

      // MMA compute
      #pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        #pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j) {
          HMMA16816(RC[i][j][0], RC[i][j][1], 
                    RA[i][0], RA[i][1], RA[i][2], RA[i][3], 
                    RB[j][0], RB[j][1], 
                    RC[i][j][0], RC[i][j][1]);
        }
      }
    }
  }

  // reg -> gmem, MMA_MxMMA_N=16x8
  #pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
    #pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      int store_warp_smem_c_m = warp_m * (MMA_M * WARP_TILE_M) + i * MMA_M;
      int store_warp_smem_c_n = warp_n * (MMA_N * WARP_TILE_N) + j * MMA_N;
      // mapping lane smem index -> global index.
      // [16][8], https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
      // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
      // [0~7][0~3 u32 -> 0~7 f16], [8~15][0~3 u32 -> 0~7 f16]
      int store_lane_gmem_c_m = by * BM + store_warp_smem_c_m + lane_id / 4;
      int store_lane_gmem_c_n = bx * BN + store_warp_smem_c_n + (lane_id % 4) * 2;
      int store_gmem_c_addr_0 = store_lane_gmem_c_m * N + store_lane_gmem_c_n;
      int store_gmem_c_addr_1 = (store_lane_gmem_c_m + 8) * N + store_lane_gmem_c_n;
      // TODO: how to use LDST128BITS here ?
      LDST32BITS(C[store_gmem_c_addr_0]) = LDST32BITS(RC[i][j][0]); 
      LDST32BITS(C[store_gmem_c_addr_1]) = LDST32BITS(RC[i][j][1]); 
    }
  }
}

// TODO: Warp swizzle/permute support ? (MMA, not WMMA)

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)           \
if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) { \
  throw std::runtime_error("Tensor size mismatch!");  \
}

// 128x128, mma2x4, warp4x4(64,32,16), stages, block swizzle
#define LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_KERNEL(stages, stride)    \
{                                                                           \
  const int N_SWIZZLE = (N + (stride) - 1) / (stride);                      \
  dim3 block(NUM_THREADS);                                                  \
  dim3 grid((div_ceil(N, BN) + N_SWIZZLE - 1) / N_SWIZZLE,                  \
             div_ceil(M, BM),                                               \
             N_SWIZZLE);                                                    \
  hgemm_mma_m16n8k16_mma2x4_warp4x4_stages_kernel<                          \
    MMA_M, MMA_N, MMA_K, MMA_TILE_M, MMA_TILE_N,                            \
    WARP_TILE_M, WARP_TILE_N, A_PAD, B_PAD,                                 \
    (stages), true><<<grid, block>>>(                                       \
    reinterpret_cast<half*>(a.data_ptr()),                                  \
    reinterpret_cast<half*>(b.data_ptr()),                                  \
    reinterpret_cast<half*>(c.data_ptr()),                                  \
    M, N, K                                                                 \
  );                                                                        \
}

#define LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_KERNEL(stages)         \
{                                                                           \
  dim3 block(NUM_THREADS);                                                  \
  dim3 grid(div_ceil(N, BN), div_ceil(M, BM));                              \
  hgemm_mma_m16n8k16_mma2x4_warp4x4_stages_kernel<                          \
    MMA_M, MMA_N, MMA_K, MMA_TILE_M, MMA_TILE_N,                            \
    WARP_TILE_M, WARP_TILE_N, A_PAD, B_PAD,                                 \
    (stages), false><<<grid, block>>>(                                      \
    reinterpret_cast<half*>(a.data_ptr()),                                  \
    reinterpret_cast<half*>(b.data_ptr()),                                  \
    reinterpret_cast<half*>(c.data_ptr()),                                  \
    M, N, K                                                                 \
  );                                                                        \
}

// 128x128, mma2x4, warp4x4(64,32,16), stages, block swizzle
void hgemm_mma_m16n8k16_mma2x4_warp4x4_stages(
  torch::Tensor a, torch::Tensor b, torch::Tensor c, 
  int stages, bool swizzle, int swizzle_stride) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int MMA_M = 16;
  constexpr int MMA_N = 8;
  constexpr int MMA_K = 16;
  constexpr int MMA_TILE_M = 2;
  constexpr int MMA_TILE_N = 4; 
  constexpr int WARP_TILE_M = 4;
  constexpr int WARP_TILE_N = 4;
  // s_a 4  ways bank conflicts within warp, after pad 8  -> 4 ways bank conflicts.
  // s_b 16 ways bank conflicts within warp, after pad 8  -> 8 ways bank conflicts.
  // s_b 16 ways bank conflicts within warp, after pad 16 -> 4 ways bank conflicts.
  // so, the best padding policy for s_a and s_b is A_PAD=0/8, B_PAD=16. Thus, 
  // improve B_PAD consume 8x~ less smem than A_PAD, 16xB_PAD vs 128xA_PAD.
  constexpr int A_PAD = 0;  // 0,8,16
  constexpr int B_PAD = 16; // 0,8,16
  constexpr int NUM_THREADS= (
    MMA_TILE_M * MMA_TILE_N * WARP_SIZE); // 2 * 4 * 32 = 256
  constexpr int BM = MMA_M * MMA_TILE_M * WARP_TILE_M;    
  constexpr int BN = MMA_N * MMA_TILE_N * WARP_TILE_N;    
  constexpr int BK = MMA_K;   
  // s2: 2*128*(16)*2=8KB,  2*16*(128+16)*2=9KB,    ~17KB
  // s3: 3*128*(16)*2=12KB, 3*16*(128+16)*2=13.5KB, ~26KB
  // s4: 4*128*(16)*2=16KB, 4*16*(128+16)*2=18KB,   ~34KB                            
  // s5: 5*128*(16)*2=20KB, 5*16*(128+16)*2=22.5KB, ~43KB    
  if (swizzle) {
    assert(swizzle_stride % 256 == 0);
    switch (stages)
    {
    case 2: // ~17KB
      LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_KERNEL(2, swizzle_stride);
      break;
    case 3: // ~26KB
      LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_KERNEL(3, swizzle_stride);
      break;
    case 4: // ~34KB
      LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_KERNEL(4, swizzle_stride);
      break;
    case 5: // ~43KB
      LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_KERNEL(5, swizzle_stride);
      break;
    default:
      LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_KERNEL(2, swizzle_stride);
      break;
    }
  } else {
    switch (stages)
    {
    case 2:
      LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_KERNEL(2);
      break;
    case 3:
      LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_KERNEL(3);
      break;
    case 4:
      LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_KERNEL(4);
      break;
    case 5:
      LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_KERNEL(5);
      break;
    default:
      LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_KERNEL(2);
      break;
    }
  }
}

// 128x128, mma2x4, warp4x4(64,32,16), stages, block swizzle, dsmem
#define LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(stages, stride)   \
{                                                                                \
  const int smem_max_size = (                                                    \
    (stages) * BM * (BK + A_PAD) * sizeof(half) +                                \
    (stages) * BK * (BN + B_PAD) * sizeof(half));                                \
  hipFuncSetAttribute(reinterpret_cast<const void*>(                                                          \
    hgemm_mma_m16n8k16_mma2x4_warp4x4_stages_dsmem_kernel<                       \
      MMA_M), MMA_N, MMA_K, MMA_TILE_M, MMA_TILE_N,                               \
      WARP_TILE_M, WARP_TILE_N, A_PAD, B_PAD, (stages), true>,                   \
    hipFuncAttributeMaxDynamicSharedMemorySize,                                 \
    98304);                                                                      \
  const int N_SWIZZLE = (N + (stride) - 1) / (stride);                           \
  dim3 block(NUM_THREADS);                                                       \
  dim3 grid((div_ceil(N, BN) + N_SWIZZLE - 1) / N_SWIZZLE,                       \
             div_ceil(M, BM),                                                    \
             N_SWIZZLE);                                                         \
  hgemm_mma_m16n8k16_mma2x4_warp4x4_stages_dsmem_kernel<                         \
    MMA_M, MMA_N, MMA_K, MMA_TILE_M, MMA_TILE_N,                                 \
    WARP_TILE_M, WARP_TILE_N, A_PAD, B_PAD, (stages), true><<<                   \
    grid, block, smem_max_size>>>(                                               \
    reinterpret_cast<half*>(a.data_ptr()),                                       \
    reinterpret_cast<half*>(b.data_ptr()),                                       \
    reinterpret_cast<half*>(c.data_ptr()),                                       \
    M, N, K                                                                      \
  );                                                                             \
}

#define LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(stages)     \
{                                                                             \
  const int smem_max_size = (                                                 \
    (stages) * BM * (BK + A_PAD) * sizeof(half) +                             \
    (stages) * BK * (BN + B_PAD) * sizeof(half));                             \
  hipFuncSetAttribute(reinterpret_cast<const void*>(                                                       \
    hgemm_mma_m16n8k16_mma2x4_warp4x4_stages_dsmem_kernel<                    \
      MMA_M), MMA_N, MMA_K, MMA_TILE_M, MMA_TILE_N,                            \
      WARP_TILE_M, WARP_TILE_N, A_PAD, B_PAD, (stages), false>,               \
    hipFuncAttributeMaxDynamicSharedMemorySize,                              \
    98304);                                                                   \
  dim3 block(NUM_THREADS);                                                    \
  dim3 grid(div_ceil(N, BN), div_ceil(M, BM));                                \
  hgemm_mma_m16n8k16_mma2x4_warp4x4_stages_dsmem_kernel<                      \
    MMA_M, MMA_N, MMA_K, MMA_TILE_M, MMA_TILE_N,                              \
    WARP_TILE_M, WARP_TILE_N, A_PAD, B_PAD, (stages), false><<<               \
    grid, block, smem_max_size>>>(                                            \
    reinterpret_cast<half*>(a.data_ptr()),                                    \
    reinterpret_cast<half*>(b.data_ptr()),                                    \
    reinterpret_cast<half*>(c.data_ptr()),                                    \
    M, N, K                                                                   \
  );                                                                          \
}

// 128x128, mma2x4, warp4x4(64,32,16), stages, block swizzle, dsmem
void hgemm_mma_m16n8k16_mma2x4_warp4x4_stages_dsmem(
  torch::Tensor a, torch::Tensor b, torch::Tensor c, 
  int stages, bool swizzle, int swizzle_stride) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1); 
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int MMA_M = 16;
  constexpr int MMA_N = 8;
  constexpr int MMA_K = 16;
  constexpr int MMA_TILE_M = 2;
  constexpr int MMA_TILE_N = 4; 
  constexpr int WARP_TILE_M = 4;
  constexpr int WARP_TILE_N = 4;
  // s_a 4  ways bank conflicts within warp, after pad 8  -> 4 ways bank conflicts.
  // s_b 16 ways bank conflicts within warp, after pad 8  -> 8 ways bank conflicts.
  // s_b 16 ways bank conflicts within warp, after pad 16 -> 4 ways bank conflicts.
  // so, the best padding policy for s_a and s_b is A_PAD=0/8, B_PAD=16. Thus, 
  // improve B_PAD consume 8x~ less smem than A_PAD, 16xB_PAD vs 128xA_PAD.
  constexpr int A_PAD = 0;  // 0,8,16
  constexpr int B_PAD = 16; // 0,8,16
  constexpr int NUM_THREADS= (
    MMA_TILE_M * MMA_TILE_N * WARP_SIZE); // 2 * 4 * 32 = 256
  constexpr int BM = MMA_M * MMA_TILE_M * WARP_TILE_M;    
  constexpr int BN = MMA_N * MMA_TILE_N * WARP_TILE_N;    
  constexpr int BK = MMA_K;   
  // s2: 2*128*(16)*2=8KB,  2*16*(128+16)*2=9KB,    ~17KB
  // s3: 3*128*(16)*2=12KB, 3*16*(128+16)*2=13.5KB, ~26KB
  // s4: 4*128*(16)*2=16KB, 4*16*(128+16)*2=18KB,   ~34KB                            
  // s5: 5*128*(16)*2=20KB, 5*16*(128+16)*2=22.5KB, ~43KB    
  if (swizzle) {
    assert(swizzle_stride % 256 == 0);
    switch (stages)
    {
    case 2: // ~17KB
      LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(2, swizzle_stride);
      break;
    case 3: // ~26KB
      LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(3, swizzle_stride);
      break;
    case 4: // ~34KB
      LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(4, swizzle_stride);
      break;
    case 5: // ~43KB
      LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(5, swizzle_stride);
      break;
    default:
      LAUNCH_16816_STAGE_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(2, swizzle_stride);
      break;
    }
  } else {
    switch (stages)
    {
    case 2:
      LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(2);
      break;
    case 3:
      LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(3);
      break;
    case 4:
      LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(4);
      break;
    case 5:
      LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(5);
      break;
    default:
      LAUNCH_16816_STAGE_NO_SWIZZLE_MMA2x4_WARP4x4_DSMEM_KERNEL(2);
      break;
    }
  }
}
