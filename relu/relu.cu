#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <torch/types.h>
#include <torch/extension.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])

// -------------------------------------- FP32 -------------------------------------- 
// Relu x: N, y: N y=max(0,x)
// grid(N/256), block(K=256) 
__global__ void relu_f32_kernel(float* x, float* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = fmaxf(0.0f, x[idx]);
}

// Relu x: N, y: N y=max(0,x) Vec4
// grid(N/256/4), block(256/4) 
__global__ void relu_f32x4_kernel(float* x, float* y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  if (idx < N) {
    float4 reg_x = FLOAT4(x[idx]);
    float4 reg_y;
    reg_y.x = fmaxf(0.0f, reg_x.x);
    reg_y.y = fmaxf(0.0f, reg_x.y);
    reg_y.z = fmaxf(0.0f, reg_x.z);
    reg_y.w = fmaxf(0.0f, reg_x.w);
    FLOAT4(y[idx]) = reg_y;
  }
}

// -------------------------------------- FP16 -------------------------------------- 
__global__ void relu_f16_kernel(half* x, half* y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) y[idx] = __hmax(__float2half(0.0f), x[idx]);
}

__global__ void relu_f16x2_kernel(half* x, half* y, int N) {
  int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
  if (idx < N) {
    half2 reg_x = HALF2(x[idx]);
    half2 reg_y = HALF2(y[idx]);
    reg_y.x = __hmax(__float2half(0.0f), reg_x.x);
    reg_y.y = __hmax(__float2half(0.0f), reg_x.y);
    HALF2(y[idx]) = reg_y;
  }
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T, S0) \
if (((T).size(0) != (S0))) { throw std::runtime_error("Tensor size mismatch!"); }

#define TORCH_BINDING_RELU(packed_type, th_type, element_type, n_elements)       \
torch::Tensor relu_##packed_type(torch::Tensor x) {                              \
  CHECK_TORCH_TENSOR_DTYPE(x, (th_type))                                         \
  auto options = torch::TensorOptions().dtype((th_type)).device(                 \
    torch::kCUDA, 0);                                                            \
  const int N = x.size(0);                                                       \
  auto y = torch::zeros({N}, options);                                           \
  static const int NUM_THREADS_PER_BLOCK = 256 / (n_elements);                   \
  const int NUM_BLOCKS = (N + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;\
  dim3 block(NUM_THREADS_PER_BLOCK);                                             \
  dim3 grid(NUM_BLOCKS);                                                         \
  relu_##packed_type##_kernel<<<grid, block>>>(                                  \
      reinterpret_cast<element_type*>(x.data_ptr()),                             \
      reinterpret_cast<element_type*>(y.data_ptr()), N);                         \
  return y;                                                                      \
}

#define TORCH_BINDING_RELU_V2(packed_type, th_type, element_type, n_elements)    \
void relu_##packed_type##_v2(torch::Tensor x, torch::Tensor y) {                 \
  CHECK_TORCH_TENSOR_DTYPE(x, (th_type))                                         \
  CHECK_TORCH_TENSOR_DTYPE(y, (th_type))                                         \
  const int N = x.size(0);                                                       \
  CHECK_TORCH_TENSOR_SHAPE(y, N)                                                 \
  static const int NUM_THREADS_PER_BLOCK = 256 / (n_elements);                   \
  const int NUM_BLOCKS = (N + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;\
  dim3 block(NUM_THREADS_PER_BLOCK);                                             \
  dim3 grid(NUM_BLOCKS);                                                         \
  relu_##packed_type##_kernel<<<grid, block>>>(                                  \
      reinterpret_cast<element_type*>(x.data_ptr()),                             \
      reinterpret_cast<element_type*>(y.data_ptr()), N);                         \
}

TORCH_BINDING_RELU(f32,    torch::kFloat32,    float,    1)
TORCH_BINDING_RELU(f32x4,  torch::kFloat32,    float,    4)
TORCH_BINDING_RELU(f16,    torch::kHalf,       half,     1)
TORCH_BINDING_RELU(f16x2,  torch::kHalf,       half,     2)
TORCH_BINDING_RELU_V2(f32,    torch::kFloat32,    float,    1)
TORCH_BINDING_RELU_V2(f32x4,  torch::kFloat32,    float,    4)
TORCH_BINDING_RELU_V2(f16,    torch::kHalf,       half,     1)
TORCH_BINDING_RELU_V2(f16x2,  torch::kHalf,       half,     2)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(relu_f32)
  TORCH_BINDING_COMMON_EXTENSION(relu_f32x4)
  TORCH_BINDING_COMMON_EXTENSION(relu_f16)
  TORCH_BINDING_COMMON_EXTENSION(relu_f16x2)
  TORCH_BINDING_COMMON_EXTENSION(relu_f32_v2)
  TORCH_BINDING_COMMON_EXTENSION(relu_f32x4_v2)
  TORCH_BINDING_COMMON_EXTENSION(relu_f16_v2)
  TORCH_BINDING_COMMON_EXTENSION(relu_f16x2_v2)
}
