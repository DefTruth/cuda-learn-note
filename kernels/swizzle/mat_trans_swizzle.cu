#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>

// reference: https://zhuanlan.zhihu.com/p/4746910252
// 转置前的矩阵存储在dev_A中，矩阵大小为M*N，转置后的数据存储在dev_B中
__global__ void mat_trans_smem_naive_kernel(int* dev_A, int M, int N, int* dev_B) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  
  // 每个block处理32*32的矩阵块
  __shared__ int s_data[32][32];

  if (row < M && col < N) {
    // 从全局内存中加载数据，转置后写到共享内存中
    s_data[threadIdx.x][threadIdx.y] = dev_A[row * N + col];
    __syncthreads();
    int n_col = blockIdx.y * blockDim.y + threadIdx.x;
    int n_row = blockIdx.x * blockDim.x + threadIdx.y;
    if (n_col < M && n_row < N) {
      // 从转置后的共享内存按行写到全局内存结果中
      dev_B[n_row * M + n_col] = s_data[threadIdx.y][threadIdx.x];
    }
  }
}

// reference: https://zhuanlan.zhihu.com/p/4746910252
__global__ void mat_trans_smem_padding_kernel(int* dev_A, int M, int N, int* dev_B) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // 每个block处理32*32的矩阵块，尾部padding来避免bank conflict
  __shared__ int s_data[32][33];

  if (row < M && col < N) {
    s_data[threadIdx.x][threadIdx.y] = dev_A[row * N + col];
    __syncthreads();
    int n_col = blockIdx.y * blockDim.y + threadIdx.x;
    int n_row = blockIdx.x * blockDim.x + threadIdx.y;
    if (n_col < M && n_row < N) {
      dev_B[n_row * M + n_col] = s_data[threadIdx.y][threadIdx.x];
    }
  }
}

// reference: https://zhuanlan.zhihu.com/p/4746910252
__global__ void mat_trans_smem_swizzle_kernel(int* dev_A, int M, int N, int* dev_B) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ int s_data[32][32];

  if (row < M && col < N) {
    // 从全局内存读取数据写入共享内存的逻辑坐标(row=x,col=y)
    // 其映射的物理存储位置位置(row=x,col=x^y)
    s_data[threadIdx.x][threadIdx.x ^ threadIdx.y] = dev_A[row * N + col];
    __syncthreads();
    int n_col = blockIdx.y * blockDim.y + threadIdx.x;
    int n_row = blockIdx.x * blockDim.x + threadIdx.y;
    if (n_row < N && n_col < M) {
      // 从共享内存的逻辑坐标(row=y,col=x)读取数据
      // 其映射的物理存储位置(row=y,col=x^y)
      dev_B[n_row * M + n_col] = s_data[threadIdx.y][threadIdx.x ^ threadIdx.y];
    }
  }
}
