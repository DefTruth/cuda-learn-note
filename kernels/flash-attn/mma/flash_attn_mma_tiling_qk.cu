#include "hip/hip_runtime.h"
#include "utils.h"

// Write FlashAttention-2 from scratch using Tensor Cores with MMA PTX instruction.
// The input is Q,K,V, 4D tensor with shape [batch_size, num_heads, seq_len, head_dim].
// The output is O, a 4D tensor with shape [batch_size, num_heads, seq_len, head_dim].

// The FlashAttention-2 algorithm is described in the following paper:
// https://arxiv.org/pdf/2307.08691

// Q,K,V,O: [batch_size, num_heads, seq_len, head_dim], [B,H,N,d]
// each block processes Q_tile with shape [Br,d] and full K,V with shape [N,d]

// Split Q across MMA(Warps) and keep access KV for all MMA(Warps),
// in order to reduce the comm between warps via smem and warp shuffle.

// MMA = m16n8k16, Br=16x4=64, Bc=8x8=64, layout: 4 warps
// |   64x64   |      warp_KV 0       |
// | warp_QP 0 | MMA 0 ... MMA 0 (x8) |
// | warp_QP 1 | MMA 1 ... MMA 1 (x8) |
// | warp_QP 2 | MMA 2 ... MMA 2 (x8) |
// | warp_QP 3 | MMA 3 ... MMA 3 (x8) |

// MMA = m16n8k16, Br=16x8=128, Bc=8x16=128, layout: 8 warps
// |  128x128  |      warp_KV 0        |
// | warp_QP 0 | MMA 0 ... MMA 0 (x16) |
// | warp_QP 1 | MMA 1 ... MMA 1 (x16) |
// | warp_QP 2 | MMA 2 ... MMA 2 (x16) |
// | warp_QP 3 | MMA 3 ... MMA 3 (x16) |
// | warp_QP 4 | MMA 4 ... MMA 4 (x16) |
// | warp_QP 5 | MMA 5 ... MMA 5 (x16) |
// | warp_QP 6 | MMA 6 ... MMA 6 (x16) |
// | warp_QP 7 | MMA 7 ... MMA 7 (x16) |

// MMA = m16n8k16, Br=16x8=128, Bc=8x8=64, layout: 8 warps
// |  128x64  |      warp_KV 0        |
// | warp_QP 0 | MMA 0 ... MMA 0 (x8) |
// | warp_QP 1 | MMA 1 ... MMA 1 (x8) |
// | warp_QP 2 | MMA 2 ... MMA 2 (x8) |
// | warp_QP 3 | MMA 3 ... MMA 3 (x8) |
// | warp_QP 4 | MMA 4 ... MMA 4 (x8) |
// | warp_QP 5 | MMA 5 ... MMA 5 (x8) |
// | warp_QP 6 | MMA 6 ... MMA 6 (x8) |
// | warp_QP 7 | MMA 7 ... MMA 7 (x8) |

// Fine-grained tiling at the MMA level for Q and K results in a constant SRAM usage of
// 64 * kMmaAtomK for Q and K. For V, the SRAM complexity is O(kMmaAtomK * d), leading to
// an overall SRAM complexity of O(kMmaAtomK * d). Consequently, this approach allows us to
// extend D (head dimension) up to 1024. Performance optimizations are ongoing. 
// Stay tuned for updates ~

template<
         const int kHeadDim,          // Headdim, 32,64,128     
         const int kMmaAtomM,         // MMA Atom M, 16
         const int kMmaAtomN,         // MMA Atom N, 8
         const int kMmaAtomK,         // MMA Atom K, 16
         const int kMmaTileSeqLenQ,   // 4, more MMA(warp), M=16*4=64, Q@K^T=[Br(M), d(K)]@[d(K),  Bc(N)]  
         const int kMmaTileSeqLenK,   // 1, more MMA(warp), N=8*1 =8,  Q@K^T=[Br(M), d(K)]@[d(K),  Bc(N)]    
         const int kMmaTileSeqLenP,   // 4, more MMA(warp), M=16*4=64, P@V  =[Br(M),Bc(K)]@[Bc(K), d(N) ]
         const int kMmaTileHeadDimV,  // 1, more MMA(warp), N=8*1 =8,  P@V  =[Br(M),Bc(K)]@[Bc(K), d(N) ]       
         const int kWarpTileSeqLenQ,  // 1, more values, M, Br=64*1=64, matmul M 
         const int kWarpTileSeqLenK,  // 8, more values, N, Bc=8*8 =64, matmul N
         const int kWarpTileSeqLenP,  // 1, more values, M, Br=64*1=64, matmul M
         const int kWarpTileHeadDimV, // 8, more values, N, d=8*(1|2|3|4|...)=8|...|32|64|96|128|...
         const int kStage, 
         const int kPad
         >
__global__ void __launch_bounds__(
  WARP_SIZE * kMmaTileSeqLenQ * kMmaTileSeqLenK) 
flash_attn_mma_stages_split_q_tiling_qk_kernel(half* Q, 
                                               half* K, 
                                               half* V, 
                                               half* O, 
                                               int QKV_seqlen,
                                               int QKV_head) {
  // Matmul Layout: Q[Br,d]@K^T[d,Bc] NT, P[Br,Bc]@V[Bc,d] NN.
  // NOTE: K[Bc,d] with row major means K^T[d,Bc] in col major.
  static_assert(kMmaAtomM == 16 && kMmaAtomN == 8 && kMmaAtomK == 16); // m16n8k16
  static_assert(kMmaTileSeqLenQ  <= 8 && kMmaTileSeqLenK  == 1);  // Q@K^T
  static_assert(kMmaTileSeqLenP  <= 8 && kMmaTileHeadDimV == 1);  // P@V
  static_assert(kWarpTileSeqLenQ == 1 && kWarpTileSeqLenK <= 16); // Q@K^T
  // kWarpTileHeadDimV: d=8*(1|2|3|4|...) = 8|...|32|64|96|128|..., etc.
  // e.g, kWarpTileHeadDimV = 8 -> d = 8*8 = 64; 16 -> d = 8*16 = 128.
  static_assert(kWarpTileSeqLenP == 1 && kWarpTileHeadDimV == (
    kHeadDim / (kMmaAtomN * kMmaTileHeadDimV))); // P@V
  static_assert(kStage < 3 && kStage > 0); 
  static_assert(kPad >= 0 && kPad % 8 == 0); // 0,8,16
  constexpr int Br = kMmaAtomM * kMmaTileSeqLenQ * kWarpTileSeqLenQ; // 16*4*1=64
  constexpr int Bc = kMmaAtomN * kMmaTileSeqLenK * kWarpTileSeqLenK; //  8*1*8=64
  static_assert(Br >= Bc); // for shared memory reuse.
  constexpr int kNumThreads = WARP_SIZE * kMmaTileSeqLenQ * kMmaTileSeqLenK; // 32*4*1=128, num threads
  // Now, N must be mutliples of Bc(32/64) for KV tiling across seqlen.
  const int Tc = div_ceil(QKV_seqlen, Bc); // Tc K_tile[Bc,d]
  const float scale = 1.0f / sqrt((float) kHeadDim);
  
  // grid(div_ceil(QKV_seqlen, Br), QKV_batch * QKV_head), (x,y,z)
  const int QKV_batch_id = blockIdx.y / QKV_head; // Batch size
  const int QKV_head_id  = blockIdx.y % QKV_head; // Head num
  const int Q_tile_id    = blockIdx.x;            // Q tile_id, range [0, Tr]
  const int O_tile_id    = Q_tile_id;             // O tile_id, same as Q.
  const int tid          = threadIdx.x;           // within block
  const int warp_id      = tid / WARP_SIZE;       // 0~7 warp_id within block
  const int lane_id      = tid % WARP_SIZE;       // 0~31
  const int warp_QP      = warp_id;               // 0,1,2,3 or 0~7
  const int warp_KV      = 0;                     // 0
  // MMA Layout [Br,Bc]=[64,64], MMA = m16n8k16, Br=16x4=64, Bc=8x8=64, layout: 4 warps
  // |   64x64   |      warp_KV 0       |
  // | warp_QP 0 | MMA 0 ... MMA 0 (x8) |
  // | warp_QP 1 | MMA 1 ... MMA 1 (x8) |
  // | warp_QP 2 | MMA 2 ... MMA 2 (x8) |
  // | warp_QP 3 | MMA 3 ... MMA 3 (x8) |
  // MMA Layout [Br,Bc]=[128,128], MMA = m16n8k16, Br=16x8=128, Bc=8x16=128, layout: 8 warps
  // |  128x128  |      warp_KV 0        |
  // | warp_QP 0 | MMA 0 ... MMA 0 (x16) |
  // | warp_QP 1 | MMA 1 ... MMA 1 (x16) |
  // | warp_QP 2 | MMA 2 ... MMA 2 (x16) |
  // | warp_QP 3 | MMA 3 ... MMA 3 (x16) |
  // | warp_QP 4 | MMA 4 ... MMA 4 (x16) |
  // | warp_QP 5 | MMA 5 ... MMA 5 (x16) |
  // | warp_QP 6 | MMA 6 ... MMA 6 (x16) |
  // | warp_QP 7 | MMA 7 ... MMA 7 (x16) |
  const int Q_gmem_offset = ((QKV_batch_id * QKV_head * QKV_seqlen * kHeadDim) + 
                             (QKV_head_id * QKV_seqlen * kHeadDim)); // Q [seqlen,d]
  const int K_gmem_offset = ((QKV_batch_id * QKV_head * QKV_seqlen * kHeadDim) + 
                             (QKV_head_id * QKV_seqlen * kHeadDim)); // K [seqlen,d]                           
  const int V_gmem_offset = Q_gmem_offset; // V [seqlen,d]
  const int O_gmem_offset = Q_gmem_offset; // O [seqlen,d]

  // Mapping Q gmem -> tid -> smem, Q[Br,kMmaAtomK]=[64/128,16], 128/256 threads.
  int load_smem_Q_Br = (tid / (kNumThreads / Br)); // Br 64, tid / 2, row 0~64
  int load_smem_Q_d  = (tid % (kNumThreads / Br)) * (kMmaAtomK / (kNumThreads / Br)); // (tid % 2) * 8, 0,8,...
  // Mapping K gmem -> tid -> smem, K[Bc,kMmaAtomK]=[64/128,16], 128 threads.
  int load_smem_K_Bc = (tid / (kNumThreads / Bc)); // Bc 64, tid / 2, row 0~64
  int load_smem_K_d  = (tid % (kNumThreads / Bc)) * (kMmaAtomK / (kNumThreads / Bc)); // (tid % 2) * 8, 0,8,...
  // TODO: Mapping V gmem -> tid -> smem, V[kMmaAtomK,kMmaAtomN]=[16,64/128], 128 threads.
  // Mapping V gmem -> tid -> smem, V[kMmaAtomK,d]=[16,64/128], 128 threads.
  int load_smem_V_Bc = (tid / (kNumThreads / kMmaAtomK)); // kMmaAtomK 16, tid / 8, row 0~15
  int load_smem_V_d  = (tid % (kNumThreads / kMmaAtomK)) * (kHeadDim / (kNumThreads / kMmaAtomK)); // (tid % 8) * 8, 0,8,56...
  // global Q row of current head for tile [Br,d] per block.
  int load_gmem_Q_Br = Q_tile_id * Br + load_smem_Q_Br; 
  if (load_gmem_Q_Br >= QKV_seqlen) return;
  constexpr bool kIsVCanLoadIn128b = (kHeadDim / (kNumThreads / kMmaAtomK)) % 8 == 0;
  constexpr bool kIsVCanLoadIn64b  = (kHeadDim / (kNumThreads / kMmaAtomK)) % 4 == 0;
  static_assert(kIsVCanLoadIn128b || kIsVCanLoadIn64b, "V can't load in 128b or 64b."); // 32,64,128,192,256,...

  // Shared memory for Q,K,V, we don not need additional smem for O 
  // collective store which perform via registers reuse and warp shuffle.
  extern __shared__ half smem[];
  // Split Q + Shared KV SMEM + Fine grain tiling, only need O(1) SRAM complexity.
  constexpr int Q_tile_size = Br * (kMmaAtomK + kPad); // Q[Br,16], 64*16*2=2048 bytes, 2M
  constexpr int K_tile_size = Bc * (kMmaAtomK + kPad); // K[Bc,16], 2M
  constexpr int V_tile_size = kMmaAtomK * (kHeadDim + kPad); // V[16,d], 2M
  // TODO: optimize QKV kStage smem store layout as in HGEMM.
  half* Q_tile_smem = smem; // 8M/16M
  half* K_tile_smem = Q_tile_smem + kStage * Q_tile_size; // 8M/16M
  half* V_tile_smem = Q_tile_smem; // V may reuse all Q+K smem after Q@K^T.
  // stage 1, Q/K smem = 64*16*2/1024=2M, V smem =16*d(64|128|...)*2/1024=2M/4M/..
  // stage 1, total smem = max(QK_smem, V_smem) = 4M if d <= 64 else V_smem.
  // stage 1, V shared QK smem, Br=Bc=64,  d=64:  2M+(2M) =4M,  +Pad(2M)  = 6M
  // stage 1, V shared QK smem, Br=Bc=128, d=64:  4M+4M   =8M,  +Pad(2M)  = 10M
  // stage 2, V shared QK smem, Br=Bc=64,  d=64:  4M+(4M) =8M,  +Pad(2M)  = 10M
  // stage 2, V shared QK smem, Br=Bc=128, d=64:  8M+8M   =16M,  +Pad(2M) = 18M
  uint32_t smem_Q_base_ptr = __cvta_generic_to_shared(Q_tile_smem);
  uint32_t smem_K_base_ptr = __cvta_generic_to_shared(K_tile_smem);
  uint32_t smem_V_base_ptr = __cvta_generic_to_shared(V_tile_smem);

  // --------------------- Registers/SMEM for thread block -------------------------
  // block m_old, l_old, store in lane, use float to keep precision.
  float lane_block_row_max_old[kWarpTileSeqLenQ][2]; // [1][2]
  float lane_block_row_sum_old[kWarpTileSeqLenQ][2]; // [1][2]
  fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_block_row_max_old, -INFINITY);
  fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_block_row_sum_old, 0.0f);

  // ---------------------- Registers for S=Q@K^T/O=P@V ----------------------------
  // registers for QKV, S=Q[Br,d]@K[Bc,d]=[Br,Bc] and O=P[Br,Bc]@V[Bc,d]=[Br,d].
  uint32_t R_Q[kWarpTileSeqLenQ][ 4]; // [1][4]
  uint32_t R_K[kWarpTileSeqLenK][ 2]; // [8][2]
  uint32_t R_V[kWarpTileHeadDimV][2]; // [8][2]
  // NOTE: For R_V[kWarpTileHeadDimV][2], kWarpTileHeadDimV will increase with d.
  // so, for large d, R_V will need more registers and cause performance down.
  // We have to find a way to apply MMA level tiling for V(R_V) for large d.
  // registers for current tile_K_seqlen within, [64,64] = S_tile[Br,Bc]
  // = Q_tile[Br,d] * K[Bc,d], each thread hold 2x32 bits regs.
  uint32_t R_S[kWarpTileSeqLenQ][kWarpTileSeqLenK][ 2]; // [1][8][2]
  // registers for tile_K_seqlen O=PV[Br,d]=P@V, [2][2/4][2], 8 or 16 regs.
  uint32_t R_O[kWarpTileSeqLenP][kWarpTileHeadDimV][2]; // [1][8][2]
  // registers final Output [D]=final rescale(R_O), [2][2/4][2], 8 or 16 regs.
  uint32_t R_D[kWarpTileSeqLenP][kWarpTileHeadDimV][2]; // [1][8][2]
  fill_3D_regs<uint32_t, kWarpTileSeqLenQ, kWarpTileSeqLenK,  2>(R_S, 0);
  fill_3D_regs<uint32_t, kWarpTileSeqLenP, kWarpTileHeadDimV, 2>(R_D, 0);
  fill_3D_regs<uint32_t, kWarpTileSeqLenP, kWarpTileHeadDimV, 2>(R_O, 0);
  
  // <loop over K seqlen>: for K^T[d,seqlen] with K^T_tile[d,Bc]
  // tile_K_seqlen: compute S_tile[Br,Bc] = Q@K^T = Q_tile[Br,d] * K^T[d,Bc]
  #pragma unroll 1
  for (int tile_K_seqlen = 0; tile_K_seqlen < Tc; ++tile_K_seqlen) { 
    // TODO: process last tile_K_seqlen ? pad to multiple of 8.
    
    // Q/K g2s
    if constexpr (kStage > 1) {
      #pragma unroll
      for (int stage = 0; stage < (kStage - 1); ++stage) {
        // Q g2s
        int load_gmem_Q_d = (stage * kMmaAtomK) + load_smem_Q_d; // 0,8
        int load_gmem_Q_addr = (
          Q_gmem_offset + load_gmem_Q_Br * kHeadDim + load_gmem_Q_d);
        uint32_t load_smem_Q_ptr = (
          smem_Q_base_ptr + (stage * Q_tile_size + 
                             load_smem_Q_Br * (kMmaAtomK + kPad) + 
                             load_smem_Q_d) * sizeof(half));
        #pragma unroll
        for (int i = 0; i < (kMmaAtomK / (kNumThreads / Br)); i += 8) {
          CP_ASYNC_CG(load_smem_Q_ptr + i * 2, &Q[load_gmem_Q_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();
        
        // K g2s
        int load_gmem_K_Bc = (tile_K_seqlen * Bc) + load_smem_K_Bc; // < seqlen
        int load_gmem_K_d  = (stage * kMmaAtomK) + load_smem_K_d; // K [Bc,16] from [seqlen,d]
        int load_gmem_K_addr = (
          K_gmem_offset + load_gmem_K_Bc * kHeadDim + load_gmem_K_d);
        uint32_t load_smem_K_ptr = (
          smem_K_base_ptr + (stage * K_tile_size + 
                             load_smem_K_Bc * (kMmaAtomK + kPad) + 
                             load_smem_K_d) * sizeof(half)
        );
        #pragma unroll
        for (int i = 0; i < (kMmaAtomK / (kNumThreads / Bc)); i += 8) {
          CP_ASYNC_CG(load_smem_K_ptr + i * 2, &K[load_gmem_K_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();
      } // end for stage

      CP_ASYNC_WAIT_GROUP(kStage - 2); // s2->0, s3->1, s4->2
      __syncthreads(); 
    } // end if kStage > 1

    // <loop over K d>: tile_K_d, kMmaAtomK = 16, K_tile_d[kMmaAtomK,Bc]
    // Matmul with NT layout, Q row major, K^T col major. 
    // NOTE: K[Bc,d] with row major means K^T[d,Bc] in col major.
    // S_tile[Br,Bc]=Q_tile[Br,d]@K[Bc,d]
    // <HGEMM in shared memory>
    fill_3D_regs<uint32_t, kWarpTileSeqLenQ, kWarpTileSeqLenK, 2>(R_S, 0);
    #pragma unroll
    for (int tile_K_d = 0; tile_K_d < (kHeadDim / kMmaAtomK); ++tile_K_d) {
      // s2 tn 0->0, 1->1, 2->0; s3 tn 0->0, 1->1, 2->2, 3->0;
      int smem_sel      = (tile_K_d) % kStage;   
      // s2 tn 0->1, 1->0, 2->1; s3 tn 0->2, 1->0, 2->1, 3->2;  
      int smem_sel_next = (tile_K_d + (kStage - 1)) % kStage;

      // stages for Q, K
      if constexpr (kStage > 1) {
        if ((tile_K_d + 1) < (kHeadDim / kMmaAtomK)) {
          // next Q tile g2s
          int load_gmem_Q_d = ((tile_K_d + 1) * kMmaAtomK) + load_smem_Q_d;
          int load_gmem_Q_addr = (
            Q_gmem_offset + load_gmem_Q_Br * kHeadDim + load_gmem_Q_d);
          uint32_t load_smem_Q_ptr = (
            smem_Q_base_ptr + (smem_sel_next * Q_tile_size + 
                               load_smem_Q_Br * (kMmaAtomK + kPad) + 
                               load_smem_Q_d) * sizeof(half));
          #pragma unroll
          for (int i = 0; i < (kMmaAtomK / (kNumThreads / Br)); i += 8) {
            CP_ASYNC_CG(load_smem_Q_ptr + i * 2, &Q[load_gmem_Q_addr + i], 16);
          }
          CP_ASYNC_COMMIT_GROUP();

          // next K tile g2s
          int load_gmem_K_Bc = tile_K_seqlen * Bc + load_smem_K_Bc; // < seqlen
          int load_gmem_K_d  = ((tile_K_d + 1) * kMmaAtomK) + load_smem_K_d; // K [Bc,16] from [seqlen,d]
          int load_gmem_K_addr = (
            K_gmem_offset + load_gmem_K_Bc * kHeadDim + load_gmem_K_d);
          uint32_t load_smem_K_ptr = (
            smem_K_base_ptr + (smem_sel_next * K_tile_size + 
                               load_smem_K_Bc * (kMmaAtomK + kPad) + 
                               load_smem_K_d) * sizeof(half)
          );
          #pragma unroll
          for (int i = 0; i < (kMmaAtomK / (kNumThreads / Bc)); i += 8) {
            CP_ASYNC_CG(load_smem_K_ptr + i * 2, &K[load_gmem_K_addr + i], 16);
          }
          CP_ASYNC_COMMIT_GROUP();
        } 
      } else {
        // sync load curr Q, K g2s
        // curr Q tile g2s
        int load_gmem_Q_d = (tile_K_d * kMmaAtomK) + load_smem_Q_d;
        int load_gmem_Q_addr = (
          Q_gmem_offset + load_gmem_Q_Br * kHeadDim + load_gmem_Q_d);
        uint32_t load_smem_Q_ptr = (
          smem_Q_base_ptr + (smem_sel * Q_tile_size + 
                             load_smem_Q_Br * (kMmaAtomK + kPad) + 
                             load_smem_Q_d) * sizeof(half));
        #pragma unroll
        for (int i = 0; i < (kMmaAtomK / (kNumThreads / Br)); i += 8) {
          CP_ASYNC_CG(load_smem_Q_ptr + i * 2, &Q[load_gmem_Q_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();

        // curr K tile g2s
        int load_gmem_K_Bc = (tile_K_seqlen * Bc) + load_smem_K_Bc; // < seqlen
        int load_gmem_K_d  = (tile_K_d * kMmaAtomK) + load_smem_K_d; // K [Bc,16] from [seqlen,d]
        int load_gmem_K_addr = (
          K_gmem_offset + load_gmem_K_Bc * kHeadDim + load_gmem_K_d);
        uint32_t load_smem_K_ptr = (
          smem_K_base_ptr + (smem_sel * K_tile_size + 
                             load_smem_K_Bc * (kMmaAtomK + kPad) + 
                             load_smem_K_d) * sizeof(half)
        );
        #pragma unroll
        for (int i = 0; i < (kMmaAtomK / (kNumThreads / Bc)); i += 8) {
          CP_ASYNC_CG(load_smem_K_ptr + i * 2, &K[load_gmem_K_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();
        // Wait curr Q, K tile ready.
        CP_ASYNC_WAIT_GROUP(0); 
        __syncthreads(); 
      } // end if kStage > 1

      // Q s2r
      #pragma unroll
      for (int i = 0; i < kWarpTileSeqLenQ; ++i) { // Q[Br,d]=[M,K]
        int warp_smem_Q_Br = warp_QP * (kMmaAtomM * kWarpTileSeqLenQ) + i * kMmaAtomM;
        int lane_smem_Q_Br = warp_smem_Q_Br + lane_id % 16; // 0~15
        int lane_smem_Q_d  = (lane_id / 16) * 8; // 0,8
        uint32_t lane_smem_Q_ptr = (
            smem_Q_base_ptr + (smem_sel * Q_tile_size + 
                               lane_smem_Q_Br * (kMmaAtomK + kPad) + 
                               lane_smem_Q_d) * sizeof(half)
        );
        LDMATRIX_X4(R_Q[i][0], R_Q[i][1], R_Q[i][2], R_Q[i][3], 
                    lane_smem_Q_ptr); // now, R_Q[1][4]
      }

      // smem -> reg, load k16n8 from smem K, offset d according tile_K_d.
      // ldmatrix.x2 for K_tile_smem, [Bc,kMmaAtomK] from [Bc,d]=[K,N]
      #pragma unroll
      for (int j = 0; j < kWarpTileSeqLenK; ++j) {
        // load k16n8 via ldmatrix.x2 from K_tile_smem[Bc,d]. 
        // K[Bc,d] with row major means K^T[d,Bc] in col major.
        int warp_smem_K_Bc = warp_KV * (kMmaAtomN * kWarpTileSeqLenK) + j * kMmaAtomN;
        int lane_smem_K_Bc = warp_smem_K_Bc + lane_id % 8; // 0~7
        int lane_smem_K_d  = ((lane_id / 8) % 2) * 8; // 0,8
        uint32_t lane_smem_K_ptr = (
            smem_K_base_ptr + (smem_sel * K_tile_size + 
                               lane_smem_K_Bc * (kMmaAtomK + kPad) + 
                               lane_smem_K_d) * sizeof(half)
        );
        LDMATRIX_X2(R_K[j][0], R_K[j][1], lane_smem_K_ptr); // R_K
      } // end for kWarpTileSeqLenK
      if constexpr (kStage < 2) {
        // Wait Q, K s2r ready if kStage < 2 in order to avoid 
        // the next Q, K tile g2s overwrite.
        __syncthreads();
      }
      
      // MMA compute
      #pragma unroll
      for (int i = 0; i < kWarpTileSeqLenQ; ++i) {
        #pragma unroll
        for (int j = 0; j < kWarpTileSeqLenK; ++j) {
          HMMA16816(R_S[i][j][0], R_S[i][j][1], 
                    R_Q[i][0], R_Q[i][1], R_Q[i][2], R_Q[i][3], 
                    R_K[j][0], R_K[j][1], 
                    R_S[i][j][0], R_S[i][j][1]);
        }
      }

      if constexpr (kStage > 1) {
        // Wait next Q, K tile g2s ready.
        CP_ASYNC_WAIT_GROUP(kStage - 2);
        __syncthreads(); 
      }

    } // end loop over d, S=Q@K^T
    __syncthreads();

    // V g2s stages. (reuse Q+K smem) load [16,d] from [Bc,d]
    if constexpr (kStage > 1) {
      #pragma unroll
      for (int stage = 0; stage < (kStage - 1); ++stage) {
        // V g2s
        int load_gmem_V_Bc = (
          (tile_K_seqlen * Bc) + (stage * kMmaAtomK) + load_smem_V_Bc); // 0~15
        int load_gmem_V_d  = load_smem_V_d;
        int load_gmem_V_addr = (
          V_gmem_offset + load_gmem_V_Bc * kHeadDim + load_gmem_V_d);
        uint32_t load_smem_V_ptr = (
          smem_V_base_ptr + (stage * V_tile_size + 
                             load_smem_V_Bc * (kHeadDim + kPad) + 
                             load_smem_V_d) * sizeof(half)
        );
        // headdim must be multiple of 32, (kHeadDim/8)%8==0 for 128 bits ld.
        if constexpr (kIsVCanLoadIn128b) {
          // 64,128,192,256,...
          #pragma unroll
          for (int i = 0; i < (kHeadDim / (kNumThreads / kMmaAtomK)); i += 8) {
            CP_ASYNC_CG(load_smem_V_ptr + i * 2, &V[load_gmem_V_addr + i], 16);
          }
        } else {
          // 32,96,160,224
          #pragma unroll
          for (int i = 0; i < (kHeadDim / (kNumThreads / kMmaAtomK)); i += 4) {
            CP_ASYNC_CA(load_smem_V_ptr + i * 2, &V[load_gmem_V_addr + i], 8);
          }
        }
        CP_ASYNC_COMMIT_GROUP();
      } // end for stage
    }
  
    // MMA = m16n8k16, Br=16x4=64, Bc=8x8=64, layout: 4 warps
    // |   64x64   |      warp_KV 0       |
    // | warp_QP 0 | MMA 0 ... MMA 0 (x8) |
    // | warp_QP 1 | MMA 1 ... MMA 1 (x8) |
    // | warp_QP 2 | MMA 2 ... MMA 2 (x8) |
    // | warp_QP 3 | MMA 3 ... MMA 3 (x8) |

    // Online safe softmax, warp/block reduce max/sum, row wise
    float lane_row_max_new[kWarpTileSeqLenQ][2]; // [1][2]
    float lane_row_sum_new[kWarpTileSeqLenQ][2]; // [1][2]
    fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_row_max_new, -INFINITY);
    fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_row_sum_new, 0.0f);

    // Row max for [Br,Bc] tile, Thread -> Warp -> Block.
    #pragma unroll
    for (int i = 0; i < kWarpTileSeqLenQ; ++i) {
      // Thread level reduce max across kWarpTileSeqLenK dim, namely Bc.
      #pragma unroll
      for (int j = 0; j < kWarpTileSeqLenK; ++j) {
        // reference: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
        // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
        // The layout of the fragments held by different threads for C. (m16n8k16)
        // Row\Col  0    1    2    3    4    5    6    7
        // 0        T0: {c0, c1}  T1: {c0, c1}  T2: {c0, c1}  T3: {c0, c1}
        // 1        T4: {c0, c1}  T5: {c0, c1}  T6: {c0, c1}  T7: {c0, c1}
        // 2        ...
        // ...
        // 7        T28: {c0, c1}  T29: {c0, c1}  T30: {c0, c1}  T31: {c0, c1}
        // 8        T0: {c2, c3}   T1: {c2, c3}   T2: {c2, c3}   T3: {c2, c3}
        // 9        T4: {c2, c3}   T5: {c2, c3}   T6: {c2, c3}   T7: {c2, c3}
        // 10       ...
        // ...
        // 15       T28: {c2, c3}  T29: {c2, c3}  T30: {c2, c3}  T31: {c2, c3}
        float2 t_reg_S_0 = __half22float2(HALF2(R_S[i][j][0])); // 0~7  {c0, c1}
        float2 t_reg_S_1 = __half22float2(HALF2(R_S[i][j][1])); // 8~15 {c2, c3}
        // This should be the row max after S = (Q @ K^T) / sqrt(d)
        float tmp_max_0 = max(t_reg_S_0.x, t_reg_S_0.y) * scale;
        float tmp_max_1 = max(t_reg_S_1.x, t_reg_S_1.y) * scale;
        lane_row_max_new[i][0] = max(lane_row_max_new[i][0], tmp_max_0);
        lane_row_max_new[i][1] = max(lane_row_max_new[i][1], tmp_max_1);
      } // end for kWarpTileSeqLenK

      // Warp level reduce max, warp_size = 4
      // Each thread contains the maximum of 2 rows of Br, 
      // and only the values of T0, T4, ..., T28 are used.
      lane_row_max_new[i][0] = warp_reduce_max<float, 4>(lane_row_max_new[i][0]);
      lane_row_max_new[i][1] = warp_reduce_max<float, 4>(lane_row_max_new[i][1]);
    } // end for kWarpTileSeqLenQ
    __syncthreads();

    // Exp sum and mul scale_factor for [Br,Bc] tile, Thread -> Warp -> Block.
    #pragma unroll
    for (int i = 0; i < kWarpTileSeqLenQ; ++i) {
      // Use latest global row max without update.
      // Br 0, row_id, 0~7,  16~23, 32~39, 48~55; 
      float block_row_max_new_0 = lane_row_max_new[i][0]; 
      // Br 1, row_id, 8~15, 24~31, 40~47, 56~63;
      float block_row_max_new_1 = lane_row_max_new[i][1];
  
      float block_row_max_old_0 = lane_block_row_max_old[i][0];
      float block_row_max_old_1 = lane_block_row_max_old[i][1];
      // Apply m_new = max(m_old, m_new) here.
      block_row_max_new_0 = max(block_row_max_old_0, block_row_max_new_0);
      block_row_max_new_1 = max(block_row_max_old_1, block_row_max_new_1);

      #pragma unroll
      for (int j = 0; j < kWarpTileSeqLenK; ++j) {
        float2 t_reg_S_0 = __half22float2(HALF2(R_S[i][j][0])); // 0~7  {c0, c1}
        float2 t_reg_S_1 = __half22float2(HALF2(R_S[i][j][1])); // 8~15 {c2, c3}
        // P = Exp(S - m_new), fmaf(x, y, z) = x * y + z;
        t_reg_S_0.x = __expf(__fmaf_rn(t_reg_S_0.x, scale, - block_row_max_new_0));
        t_reg_S_0.y = __expf(__fmaf_rn(t_reg_S_0.y, scale, - block_row_max_new_0));
        t_reg_S_1.x = __expf(__fmaf_rn(t_reg_S_1.x, scale, - block_row_max_new_1));
        t_reg_S_1.y = __expf(__fmaf_rn(t_reg_S_1.y, scale, - block_row_max_new_1));
        lane_row_sum_new[i][0] += (t_reg_S_0.x + t_reg_S_0.y);
        lane_row_sum_new[i][1] += (t_reg_S_1.x + t_reg_S_1.y);
        // Update R_S for P[Br,Bc] = Exp(S-m), point wise.
        HALF2(R_S[i][j][0]) = __float22half2_rn(t_reg_S_0);
        HALF2(R_S[i][j][1]) = __float22half2_rn(t_reg_S_1);
      } // end for kWarpTileSeqLenK

      // Warp level reduce sum, warp_size = 4
      lane_row_sum_new[i][0] = warp_reduce_sum<float, 4>(lane_row_sum_new[i][0]);
      lane_row_sum_new[i][1] = warp_reduce_sum<float, 4>(lane_row_sum_new[i][1]);
    } // end for kWarpTileSeqLenQ
    __syncthreads();
    
    // <loop over V Bc>: P[Br,Bc]@V[Bc,d]=[Br,d]=[64,64/128], partion Attention.
    // Matmul with NN layout: P[Br,Bc] row major, V[Bc,d] row major.
    // Make sure to clear the states in R_O before MMA for P@V for each step.

    // NOTE: Values for P[Br,Bc] already in R_S registers, can we use these 
    // registers for P(A) matrix directly ? How to do that ?
    // according to the A matrix layout for MMA m16n8k16 instruction. 
    // reference: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
    // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
    // The layout of the fragments held by different threads for A matrix with .f16.
    // R\C  0    1    2    3    4    5    6    7    8    9   10   11   12   13   14   15
    // 0    T0: {a0, a1}  T1: {a0, a1}  T2: {a0, a1}  T3: {a0, a1}  T0: {a4, a5}  T1: {a4, a5}  T2: {a4, a5}  T3: {a4, a5}
    // 1    T4: {a0, a1}  T5: {a0, a1}  T6: {a0, a1}  T7: {a0, a1}  T4: {a4, a5}  T5: {a4, a5}  T6: {a4, a5}  T7: {a4, a5}
    // 2    (dashed arrow pointing right)
    // ...
    // 7    T28: {a0, a1}  T29: {a0, a1}  T30: {a0, a1}  T31: {a0, a1}  T28: {a4, a5}  T29: {a4, a5}  T30: {a4, a5}  T31: {a4, a5}
    // 8    T0: {a2, a3}   T1: {a2, a3}   T2: {a2, a3}   T3: {a2, a3}   T0: {a6, a7}   T1: {a6, a7}   T2: {a6, a7}   T3: {a6, a7}
    // 9    T4: {a2, a3}   T5: {a2, a3}   T6: {a2, a3}   T7: {a2, a3}   T4: {a6, a7}   T5: {a6, a7}   T6: {a6, a7}   T7: {a6, a7}
    // 10   (dashed arrow pointing right)
    // ...
    // 15   T28: {a2, a3}  T29: {a2, a3}  T30: {a2, a3}  T31: {a2, a3}  T28: {a6, a7}  T29: {a6, a7}  T30: {a6, a7}  T31: {a6, a7}

    // Wait V g2s stages ready.
    if constexpr (kStage > 1) {
      CP_ASYNC_WAIT_GROUP(kStage - 2); // s2->0, s3->1, s4->2
      __syncthreads(); 
    }
    
    // <HGEMM in registers> P@V=[Br,Bc]@[Bc,d]
    fill_3D_regs<uint32_t, kWarpTileSeqLenP, kWarpTileHeadDimV, 2>(R_O, 0);
    #pragma unroll
    for (int tile_V_Bc = 0; tile_V_Bc < (Bc / kMmaAtomK); ++tile_V_Bc) {
      // s2 tn 0->0, 1->1, 2->0; s3 tn 0->0, 1->1, 2->2, 3->0;
      int smem_sel      = (tile_V_Bc) % kStage;   
      // s2 tn 0->1, 1->0, 2->1; s3 tn 0->2, 1->0, 2->1, 3->2;  
      int smem_sel_next = (tile_V_Bc + (kStage - 1)) % kStage;

      // stages for V
      if constexpr (kStage > 1) {
        if ((tile_V_Bc + 1) < (Bc / kMmaAtomK)) {
          //  next V tile g2s
          int load_gmem_V_Bc = (
            (tile_K_seqlen * Bc) + (tile_V_Bc + 1) * kMmaAtomK + load_smem_V_Bc); // 0~15
          int load_gmem_V_d  = load_smem_V_d;
          int load_gmem_V_addr = (
            V_gmem_offset + load_gmem_V_Bc * kHeadDim + load_gmem_V_d);
          uint32_t load_smem_V_ptr = (
            smem_V_base_ptr + (smem_sel_next * V_tile_size + 
                               load_smem_V_Bc * (kHeadDim + kPad) + 
                               load_smem_V_d) * sizeof(half)
          );
          // headdim must be multiple of 32, (kHeadDim/8)%8==0 for 128 bits ld.
          if constexpr (kIsVCanLoadIn128b) {
            // 64,128,192,256,...
            #pragma unroll
            for (int i = 0; i < (kHeadDim / (kNumThreads / kMmaAtomK)); i += 8) {
              CP_ASYNC_CG(load_smem_V_ptr + i * 2, &V[load_gmem_V_addr + i], 16);
            }
          } else {
            // 32,96,160,224
            #pragma unroll
            for (int i = 0; i < (kHeadDim / (kNumThreads / kMmaAtomK)); i += 4) {
              CP_ASYNC_CA(load_smem_V_ptr + i * 2, &V[load_gmem_V_addr + i], 8);
            }
          }
          CP_ASYNC_COMMIT_GROUP();
        }
      } else {
        // sync load curr V g2s
        int load_gmem_V_Bc = (
          (tile_K_seqlen * Bc) + (tile_V_Bc * kMmaAtomK) + load_smem_V_Bc); // 0~15
        int load_gmem_V_d = load_smem_V_d;
        int load_gmem_V_addr = (
          V_gmem_offset + load_gmem_V_Bc * kHeadDim + load_gmem_V_d);
        uint32_t load_smem_V_ptr = (
          smem_V_base_ptr + (smem_sel * V_tile_size + 
                             load_smem_V_Bc * (kHeadDim + kPad) + 
                             load_smem_V_d) * sizeof(half)
        );
        // headdim must be multiple of 32, (kHeadDim/8)%8==0 for 128 bits ld.
        if constexpr (kIsVCanLoadIn128b) {
          // 64,128,192,256,...
          #pragma unroll
          for (int i = 0; i < (kHeadDim / (kNumThreads / kMmaAtomK)); i += 8) {
            CP_ASYNC_CG(load_smem_V_ptr + i * 2, &V[load_gmem_V_addr + i], 16);
          }
        } else {
          // 32,96,160,224
          #pragma unroll
          for (int i = 0; i < (kHeadDim / (kNumThreads / kMmaAtomK)); i += 4) {
            CP_ASYNC_CA(load_smem_V_ptr + i * 2, &V[load_gmem_V_addr + i], 8);
          }
        }
        CP_ASYNC_COMMIT_GROUP();
        // Wait curr V tile ready.
        CP_ASYNC_WAIT_GROUP(0); 
        __syncthreads(); 
      }

      // Load k16n8 V from smem -> regs, R_KV, ldmatrix.x2.trans.
      #pragma unroll
      for (int j = 0; j < kWarpTileHeadDimV; ++j) { 
        int warp_smem_V_d  = warp_KV * (kMmaAtomN * kWarpTileHeadDimV) + j * kMmaAtomN; // d, matmaul N
        int lane_smem_V_Bc = lane_id % 16; // 0~15; Bc, matmul K
        int lane_smem_V_d  = warp_smem_V_d; // 0
        uint32_t lane_smem_V_ptr = (
          smem_V_base_ptr + (smem_sel * V_tile_size + 
                             lane_smem_V_Bc * (kHeadDim + kPad) + 
                             lane_smem_V_d) * sizeof(half)
        );
        LDMATRIX_X2_T(R_V[j][0], R_V[j][1], lane_smem_V_ptr); // R_V
      }
      if constexpr (kStage < 2) {
        // Wait V s2r ready if kStage < 2 in order to avoid 
        // the next V tile g2s overwrite.
        __syncthreads();
      }
      
      // For R_S[1][8][2], mapping the layout below of P matrix.
      // MMA = m16n8k16, Br=16x4=64, Bc=8x8=64, layout: 4 warps
      // |   64x64   |      warp_KV 0       |
      // | warp_QP 0 | MMA 0 ... MMA 0 (x8) |
      // | warp_QP 1 | MMA 1 ... MMA 1 (x8) |
      // | warp_QP 2 | MMA 2 ... MMA 2 (x8) |
      // | warp_QP 3 | MMA 3 ... MMA 3 (x8) |
      // tile_V_Bc = 0, all curr MMAs(0~4) need slice P[:,  0:16], 0, 1; stored in all MMAs.
      // tile_V_Bc = 1, all curr MMAs(0~4) need slice P[:, 16:32], 2, 3; stored in all MMAs.
      // tile_V_Bc = 2, all curr MMAs(0~4) need slice P[:, 32:48], 4, 5; stored in all MMAs. 
      // tile_V_Bc = 3, all curr MMAs(0~4) need slice P[:, 48:64], 6, 7; stored in all MMAs. 
      int w = tile_V_Bc * 2; // MMA(Warp) selected, 0, 2, 4, 6
      #pragma unroll
      for (int i = 0; i < kWarpTileSeqLenP; ++i) { // 1
        #pragma unroll
        for (int j = 0; j < kWarpTileHeadDimV; ++j) { // 8, 16, 32, ...
          HMMA16816(R_O[i][j][0], R_O[i][j][1], 
                    R_S[i][w][0], R_S[i][w][1], R_S[i][w + 1][0],  R_S[i][w + 1][1], 
                    R_V[j][0],    R_V[j][1],
                    R_O[i][j][0], R_O[i][j][1]);
        }
      }

      if constexpr (kStage > 1) {
        // Wait next V tile g2s ready.
        CP_ASYNC_WAIT_GROUP(kStage - 2); 
        __syncthreads();
      }

    } // end for V Bc.
    __syncthreads(); 

    // Rescale O -> Update row sum Exp -> then, Update row max.
    #pragma unroll
    for (int i = 0; i < kWarpTileSeqLenP; ++i) { // kWarpTileSeqLenQ=kWarpTileSeqLenP=1
      // m = max(m_old, m_new), l = exp(m_old - m) * l_old + l_new (FA2 paper)
      // Br 0, row_id, 0~7,  16~23, 32~39, 48~55; Br 1, row_id, 8~15, 24~31, 40~47, 56~63
      float block_row_max_new_0 = lane_row_max_new[i][0]; 
      float block_row_max_new_1 = lane_row_max_new[i][1];
      float block_row_sum_new_0 = lane_row_sum_new[i][0];
      float block_row_sum_new_1 = lane_row_sum_new[i][1];
      
      float block_row_max_old_0 = lane_block_row_max_old[i][0];
      float block_row_max_old_1 = lane_block_row_max_old[i][1];
      // NOTE: max(-inf, val) = val.
      block_row_max_new_0 = max(block_row_max_old_0, block_row_max_new_0);
      block_row_max_new_1 = max(block_row_max_old_1, block_row_max_new_1);   
      // Avoid inf value while using m_old for rescaling O.
      block_row_max_old_0 = (tile_K_seqlen > 0 ? block_row_max_old_0 : 
                                                 block_row_max_new_0);                                       
      block_row_max_old_1 = (tile_K_seqlen > 0 ? block_row_max_old_1 : 
                                                 block_row_max_new_1);  

      // rescale factor for O and l, exp(m_old - m)
      float rescale_o_factor_0 = __expf(block_row_max_old_0 - block_row_max_new_0);
      float rescale_o_factor_1 = __expf(block_row_max_old_1 - block_row_max_new_1);
      // 0. Rescale O: Online rescaling O each tile_K_seqlen step, need m_new, m_old.
      // m = max(m_old, m_new), O_new[Br,d] = exp(m_old - m) * O_old + P@V
      #pragma unroll
      for (int j = 0; j < kWarpTileHeadDimV; ++j) { // 8, 16, 32, ...
        float2 t_reg_O_0 = __half22float2(HALF2(R_O[i][j][0])); // 0~7  {c0, c1}
        float2 t_reg_O_1 = __half22float2(HALF2(R_O[i][j][1])); // 8~15 {c2, c3}
        float2 t_reg_D_0 = __half22float2(HALF2(R_D[i][j][0])); // 0~7  {c0, c1}
        float2 t_reg_D_1 = __half22float2(HALF2(R_D[i][j][1])); // 8~15 {c2, c3}
        // Note that the formula in the FA2 paper is incorrect; here, 
        // the inverse of the exp function should not be taken, as it 
        // would result in an error during rescaling, namely, you have
        // use exp(m_old - m_new), not 1/(m_old - m_new).
        // O_new[Br,d] = exp(m_old - m_new) * O_old + P@V
        t_reg_D_0.x = __fmaf_rn(rescale_o_factor_0, t_reg_D_0.x, t_reg_O_0.x);
        t_reg_D_0.y = __fmaf_rn(rescale_o_factor_0, t_reg_D_0.y, t_reg_O_0.y);
        t_reg_D_1.x = __fmaf_rn(rescale_o_factor_1, t_reg_D_1.x, t_reg_O_1.x);
        t_reg_D_1.y = __fmaf_rn(rescale_o_factor_1, t_reg_D_1.y, t_reg_O_1.y);
        HALF2(R_D[i][j][0]) = __float22half2_rn(t_reg_D_0);
        HALF2(R_D[i][j][1]) = __float22half2_rn(t_reg_D_1);
      } // end for kWarpTileHeadDimV.

      // Now, we can update m, l after O has been scaled.
      // 1. First, update block row sum Exp for each lane which
      // need both m_new and m_old.
      float block_row_sum_old_0 = lane_block_row_sum_old[i][0];
      float block_row_sum_old_1 = lane_block_row_sum_old[i][1];
      // Update l = exp(m_old - m_new) * l_old + row_sum(P).
      lane_block_row_sum_old[i][0] = (__fmaf_rn(
        rescale_o_factor_0, block_row_sum_old_0, block_row_sum_new_0));
      lane_block_row_sum_old[i][1] = (__fmaf_rn(
        rescale_o_factor_1, block_row_sum_old_1, block_row_sum_new_1));
      // 2. Then, update block row max for each lane.
      lane_block_row_max_old[i][0] = block_row_max_new_0;
      lane_block_row_max_old[i][1] = block_row_max_new_1;
    }
  } // end loop over N
  __syncthreads();

  // Finaly, we still have to rescale O once more.
  // O_output(D) = ( 1/l_final ) * O_final (FA2 paper)
  // NOTE: Here, we choose to reuse R_O as final output 
  // in order to reduce regs usage.
  #pragma unroll
  for (int i = 0; i < kWarpTileSeqLenP; ++i) { // 1
    float rescale_factor_0 = __frcp_rn(lane_block_row_sum_old[i][0]);
    float rescale_factor_1 = __frcp_rn(lane_block_row_sum_old[i][1]);
    #pragma unroll
    for (int j = 0; j < kWarpTileHeadDimV; ++j) { // 8, 16, 32, ...
      float2 t_reg_D_0 = __half22float2(HALF2(R_D[i][j][0])); // 0~7  {c0, c1}
      float2 t_reg_D_1 = __half22float2(HALF2(R_D[i][j][1])); // 8~15 {c2, c3}
      t_reg_D_0.x = rescale_factor_0 * t_reg_D_0.x;
      t_reg_D_0.y = rescale_factor_0 * t_reg_D_0.y;
      t_reg_D_1.x = rescale_factor_1 * t_reg_D_1.x;
      t_reg_D_1.y = rescale_factor_1 * t_reg_D_1.y;
      HALF2(R_D[i][j][0]) = __float22half2_rn(t_reg_D_0);
      HALF2(R_D[i][j][1]) = __float22half2_rn(t_reg_D_1);
    }
  }

  // Store O(D): Write O[Br,d] from regs -> gmem, collective store 
  // with reg reuse & warp shuffle. need R_Z[2][4]. 
  // TODO: reuse Q smem for collective store: regs -> smem -> gmem
  #pragma unroll
  for (int i = 0; i < kWarpTileSeqLenP; ++i) { // 1
    #pragma unroll
    for (int j = 0; j < kWarpTileHeadDimV; ++j) { // 8
      // we have to use new R_Z regs for collective store.
      uint32_t R_Z[2][4];
      R_Z[0][0] = R_D[i][j][0]; R_Z[1][0] = R_D[i][j][1]; // warp_size 4
      R_Z[0][1] = __shfl_sync((0xffffffff), R_D[i][j][0], lane_id + 1, 4);
      R_Z[0][2] = __shfl_sync((0xffffffff), R_D[i][j][0], lane_id + 2, 4);
      R_Z[0][3] = __shfl_sync((0xffffffff), R_D[i][j][0], lane_id + 3, 4);
      R_Z[1][1] = __shfl_sync((0xffffffff), R_D[i][j][1], lane_id + 1, 4);
      R_Z[1][2] = __shfl_sync((0xffffffff), R_D[i][j][1], lane_id + 2, 4);
      R_Z[1][3] = __shfl_sync((0xffffffff), R_D[i][j][1], lane_id + 3, 4);
      // st.global.v4 128 bits. [Br,d]
      if (lane_id % 4 == 0) {
        // (0/1)*32 + (0/1)*16=(0,16,32,48), + 0~7 -> 0~56
        int store_warp_regs_O_Br = warp_QP * (kMmaAtomM * kWarpTileSeqLenP ) + i * kMmaAtomM;
        int store_lane_gmem_O_Br = O_tile_id * Br + store_warp_regs_O_Br + lane_id / 4; // 0~7
        // (0~3)*16 + (0/1)*8=(0,8,16,24,...,48,56)
        int store_warp_regs_O_d = warp_KV * (kMmaAtomN * kWarpTileHeadDimV) + j * kMmaAtomN;
        int store_lane_gmem_O_d = store_warp_regs_O_d; // (0~3)*16+(0/8)
        int store_gmem_O_addr_0 = (
          O_gmem_offset + (store_lane_gmem_O_Br + 0) * kHeadDim + store_lane_gmem_O_d);
        int store_gmem_O_addr_1 = (
          O_gmem_offset + (store_lane_gmem_O_Br + 8) * kHeadDim + store_lane_gmem_O_d);
        LDST128BITS(O[store_gmem_O_addr_0]) = LDST128BITS(R_Z[0][0]);
        LDST128BITS(O[store_gmem_O_addr_1]) = LDST128BITS(R_Z[1][0]);
      }
    } // end for kWarpTileHeadDimV
  } // end for kWarpTileSeqLenQ
}

// Launch kernel for flash_attn_mma_stages_split_q_tiling_qk
template<const int kHeadDim, const int kStage>
void launch_flash_attn_mma_stages_split_q_tiling_qk(
  torch::Tensor Q, torch::Tensor K, torch::Tensor V, torch::Tensor O) {
  // Now: fixed tile BrxBc=128x128 for d>= 128, 64x64 for d<128.
  // TODO: dynamic tile size for Br, Bc according to kHeadDim and shared memory size.
  constexpr int kMmaAtomM = 16;
  constexpr int kMmaAtomN = 8;
  constexpr int kMmaAtomK = 16;
  constexpr int kMmaTileSeqLenQ  = (kHeadDim < 128) ? 4 : 8;
  constexpr int kMmaTileSeqLenK  = 1;
  constexpr int kMmaTileSeqLenP  = (kHeadDim < 128) ? 4 : 8;
  constexpr int kMmaTileHeadDimV = 1;
  constexpr int kWarpTileSeqLenQ = 1;
  constexpr int kWarpTileSeqLenK = (kHeadDim < 128) ? 8 : 16;
  constexpr int kWarpTileSeqLenP = 1;
  constexpr int kWarpTileHeadDimV = (kHeadDim / (kMmaAtomN * kMmaTileHeadDimV)); // (d=64)8,(d=128)16,32,....
  constexpr int Br = kMmaAtomM * kMmaTileSeqLenQ * kWarpTileSeqLenQ; // 16*4*1=64
  constexpr int Bc = kMmaAtomN * kMmaTileSeqLenK * kWarpTileSeqLenK; //  8*1*8=64
  constexpr int kNumThreads = WARP_SIZE * kMmaTileSeqLenQ * kMmaTileSeqLenK; // 32*4*1=128, num threads
  constexpr int kPad = 8; // 0.25~0.5M
  
  // static int kMaxSramPerBlock;
  // hipDeviceGetAttribute(&kMaxSramPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
  // Calculate SRAM size needed per block, Q,K,V smem size, V shared the QK smem.
  constexpr int QK_smem_size = (kStage * (Br * (kMmaAtomK + kPad)) + 
                                kStage * (Bc * (kMmaAtomK + kPad)));
  // Now, for V_smem_size, s=2, d=64, 4M, 16 regs; d=128, 8M, 32 regs; 
  // d=256, 16M, 64 regs; d=512, 32M, 128 regs; d=1024, 64M, 256 regs;
  // TODO: Fully sub-tiling for d while perform P@V, kMmaAtomK * (kMmaAtomN)
  constexpr int V_smem_size  = (kStage * (kMmaAtomK * (kHeadDim + kPad))); 
  // try to let V reuse all Q+K smem after Q@K^T, reduce smem usage.
  const int smem_max_size = max(QK_smem_size, V_smem_size) * sizeof(half);

  const int QKV_batch  = Q.size(0); 
  const int QKV_head   = Q.size(1);
  const int QKV_seqlen = Q.size(2); // QKV_seqlen
  assert(QKV_seqlen % max(Br, Bc) == 0); // multiple of max(Br, Bc)
  
  // TODO: How to apply block swizzle to improve L2 Cache hit rate?
  // NOTE: reorder (B,H,Tr) -> (Tr,B*H) seems can improve L2 Cache hit rate. 
  // This might be because SM schedules blocks starting from the x-dimension. 
  // Placing Tr at the forefront ensures that identical KV pairs are placed 
  // in consecutive scheduling queues, thereby improving L2 Cache hit rates.
  // Tr(=N/Br), batch_size x num_heads
  dim3 grid(div_ceil(QKV_seqlen, Br), QKV_batch * QKV_head); 
  dim3 block(kNumThreads); // 4/8 warps per block

  hipFuncSetAttribute(reinterpret_cast<const void*>(
    flash_attn_mma_stages_split_q_tiling_qk_kernel<
      kHeadDim), 
      kMmaAtomM, 
      kMmaAtomN, 
      kMmaAtomK, 
      kMmaTileSeqLenQ, 
      kMmaTileSeqLenK, 
      kMmaTileSeqLenP, 
      kMmaTileHeadDimV, 
      kWarpTileSeqLenQ, 
      kWarpTileSeqLenK, 
      kWarpTileSeqLenP, 
      kWarpTileHeadDimV, 
      kStage, 
      kPad
    >,
    hipFuncAttributeMaxDynamicSharedMemorySize,
    // kMaxSramPerBlock
    98304
  );

  flash_attn_mma_stages_split_q_tiling_qk_kernel<
    kHeadDim, 
    kMmaAtomM, 
    kMmaAtomN, 
    kMmaAtomK, 
    kMmaTileSeqLenQ,  
    kMmaTileSeqLenK,
    kMmaTileSeqLenP, 
    kMmaTileHeadDimV, 
    kWarpTileSeqLenQ, 
    kWarpTileSeqLenK, 
    kWarpTileSeqLenP, 
    kWarpTileHeadDimV, 
    kStage, 
    kPad
  ><<<grid, block, smem_max_size>>>(
    reinterpret_cast<half*>(Q.data_ptr()),
    reinterpret_cast<half*>(K.data_ptr()),
    reinterpret_cast<half*>(V.data_ptr()),
    reinterpret_cast<half*>(O.data_ptr()),
    QKV_seqlen,
    QKV_head
  );
}

void flash_attn_mma_stages_split_q_tiling_qk(torch::Tensor Q, 
                                             torch::Tensor K, 
                                             torch::Tensor V, 
                                             torch::Tensor O, 
                                             int stages) {
  CHECK_TORCH_TENSOR_DTYPE(Q, torch::kHalf) // Q [B,H,N,D]
  CHECK_TORCH_TENSOR_DTYPE(K, torch::kHalf) // K [B,H,N,D]
  CHECK_TORCH_TENSOR_DTYPE(V, torch::kHalf) // V [B,H,N,D]
  CHECK_TORCH_TENSOR_DTYPE(O, torch::kHalf) // O [B,H,N,D]
  const int d = Q.size(3); // B, H, N, d

  if (stages > 1) {
    switch (d)
    {
    case 32:
      launch_flash_attn_mma_stages_split_q_tiling_qk<32,   2>(Q, K, V, O);
      break;
    case 64:
      launch_flash_attn_mma_stages_split_q_tiling_qk<64,   2>(Q, K, V, O);
      break;
    case 96:
      launch_flash_attn_mma_stages_split_q_tiling_qk<96,   2>(Q, K, V, O);
      break;
    case 128:
      launch_flash_attn_mma_stages_split_q_tiling_qk<128,  2>(Q, K, V, O);
      break;
    case 256:
      launch_flash_attn_mma_stages_split_q_tiling_qk<256,  2>(Q, K, V, O);
      break;
    case 512:
      launch_flash_attn_mma_stages_split_q_tiling_qk<512,  2>(Q, K, V, O);
      break;
    case 1024:
      launch_flash_attn_mma_stages_split_q_tiling_qk<1024, 2>(Q, K, V, O);
      break;
    default:
      throw std::runtime_error("headdim not support!");
      break;
    }
  } else {
    switch (d)
    {
    case 32:
      launch_flash_attn_mma_stages_split_q_tiling_qk<32,   1>(Q, K, V, O);
      break;
    case 64:
      launch_flash_attn_mma_stages_split_q_tiling_qk<64,   1>(Q, K, V, O);
      break;
    case 96:
      launch_flash_attn_mma_stages_split_q_tiling_qk<96,   1>(Q, K, V, O);
      break;
    case 128:
      launch_flash_attn_mma_stages_split_q_tiling_qk<128,  1>(Q, K, V, O);
      break;
    case 256:
      launch_flash_attn_mma_stages_split_q_tiling_qk<256,  1>(Q, K, V, O);
      break;
    case 512:
      launch_flash_attn_mma_stages_split_q_tiling_qk<512,  1>(Q, K, V, O);
      break;
    case 1024:
      launch_flash_attn_mma_stages_split_q_tiling_qk<1024, 1>(Q, K, V, O);
      break;
    default:
      throw std::runtime_error("headdim not support!");
      break;
    }
  }
}
