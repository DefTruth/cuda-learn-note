#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>
using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2*>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
// gmem -> smem
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n) asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes) asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes) asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
// smem -> gmem: requires sm_90 or higher.
#define CP_ASYNC_BULK_COMMIT_GROUP() asm volatile("cp.async.bulk.commit_group;\n" ::)
#define CP_ASYNC_BULK_WAIT_ALL() asm volatile("cp.async.bulk.wait_all;\n" ::)
#define CP_ASYNC_BULK_WAIT_GROUP(n) asm volatile("cp.async.bulk.wait_group %0;\n" ::"n"(n))
#define CP_ASYNC_BULK(dst, src, bytes) asm volatile("cp.async.bulk.global.shared::cta.bulk_group.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
// ldmatrix
#define LDMATRIX_X1(R, addr) asm volatile("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
#define LDMATRIX_X1_T(R, addr) asm volatile("ldmatrix.sync.aligned.x1.trans.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2_T(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4_T(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
// stmatrix: requires sm_90 or higher.
#define STMATRIX_X1(addr, R) asm volatile("stmatrix.sync.aligned.x1.m8n8.shared.b16 [%0], {%1};\n" :: "r"(addr), "r"(R))
#define STMATRIX_X2(addr, R0, R1) asm volatile("stmatrix.sync.aligned.x2.m8n8.shared.b16 [%0], {%1, %2};\n" :: "r"(addr), "r"(R0), "r"(R1))
#define STMATRIX_X4(addr, R0, R1, R2, R3) asm volatile("stmatrix.sync.aligned.x4.m8n8.shared.b16 [%0], {%1, %2, %3, %4};\n" :: "r"(addr), "r"(R0), "r"(R1), "r"(R2), "r"(R3))
#define STMATRIX_X1_T(addr, R) asm volatile("stmatrix.sync.aligned.x1.trans.m8n8.shared.b16 [%0], {%1};\n" :: "r"(addr), "r"(R))
#define STMATRIX_X2_T(addr, R0, R1) asm volatile("stmatrix.sync.aligned.x2.trans.m8n8.shared.b16 [%0], {%1, %2};\n" :: "r"(addr), "r"(R0), "r"(R1))
#define STMATRIX_X4_T(addr, R0, R1, R2, R3) asm volatile("stmatrix.sync.aligned.x4.trans.m8n8.shared.b16 [%0], {%1, %2, %3, %4};\n" :: "r"(addr), "r"(R0), "r"(R1), "r"(R2), "r"(R3))
// mma m16n8k16
#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1) asm volatile("mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n" : "=r"(RD0), "=r"(RD1) : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0), "r"(RC1))

HOST_DEVICE_INLINE 
int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

// Write FlashAttention-2 from scratch using Tensor Cores with MMA PTX instruction.
// The input is Q,K,V, 4D tensor with shape [batch_size, num_heads, seq_len, head_dim].
// The output is O, a 4D tensor with shape [batch_size, num_heads, seq_len, head_dim].

// The FlashAttention-2 algorithm is described in the following paper:
// https://arxiv.org/abs/2110.08210

// m16n8k16_mma2x4_warp4x4
// Q,K,V,O: [batch_size, num_heads, seq_len, head_dim], [Nxd]
// each block processes Q_tile with shape [Br,d] and full K,V with shape [N,d]
// Br or Bc = 64,128,256, etc.
// grid(batch, head_num, N/Br), block(256=8*mma)
template<const int Br, const int Bc, const int d>
__global__  void flash_attn_mma_kernel(
  half* Q, half* K, half* V,  half* O, int N) {
  // step 0: S_tile[Br,N] = Q_tile[Br,d] * K[N,d], slice-k manner matmul
  // across K's N dim, each K_tile/V_tile inner loop has shape [Bc,d].
  // step 1: P_tile[Br,N] = softmax(S_tile[Br,N]), row wise.
  // step 2: O_tile[Br,d] = P_tile[Br,N] * V[N,d], matmul.
  const int Tr = div_ceil(N, Br); // Tr Q_tile[Br,d]
  const int Tc = div_ceil(N, Bc); // Tc K/V_tile[Bc,d]
  const float scale = 1.0 / sqrt((float)d);
  


}