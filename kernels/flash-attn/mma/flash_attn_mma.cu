#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>
using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2*>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
// gmem -> smem
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n) asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes) asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes) asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
// smem -> gmem: requires sm_90 or higher.
#define CP_ASYNC_BULK_COMMIT_GROUP() asm volatile("cp.async.bulk.commit_group;\n" ::)
#define CP_ASYNC_BULK_WAIT_ALL() asm volatile("cp.async.bulk.wait_all;\n" ::)
#define CP_ASYNC_BULK_WAIT_GROUP(n) asm volatile("cp.async.bulk.wait_group %0;\n" ::"n"(n))
#define CP_ASYNC_BULK(dst, src, bytes) asm volatile("cp.async.bulk.global.shared::cta.bulk_group.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
// ldmatrix
#define LDMATRIX_X1(R, addr) asm volatile("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
#define LDMATRIX_X1_T(R, addr) asm volatile("ldmatrix.sync.aligned.x1.trans.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2_T(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4_T(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
// stmatrix: requires sm_90 or higher.
#define STMATRIX_X1(addr, R) asm volatile("stmatrix.sync.aligned.x1.m8n8.shared.b16 [%0], {%1};\n" :: "r"(addr), "r"(R))
#define STMATRIX_X2(addr, R0, R1) asm volatile("stmatrix.sync.aligned.x2.m8n8.shared.b16 [%0], {%1, %2};\n" :: "r"(addr), "r"(R0), "r"(R1))
#define STMATRIX_X4(addr, R0, R1, R2, R3) asm volatile("stmatrix.sync.aligned.x4.m8n8.shared.b16 [%0], {%1, %2, %3, %4};\n" :: "r"(addr), "r"(R0), "r"(R1), "r"(R2), "r"(R3))
#define STMATRIX_X1_T(addr, R) asm volatile("stmatrix.sync.aligned.x1.trans.m8n8.shared.b16 [%0], {%1};\n" :: "r"(addr), "r"(R))
#define STMATRIX_X2_T(addr, R0, R1) asm volatile("stmatrix.sync.aligned.x2.trans.m8n8.shared.b16 [%0], {%1, %2};\n" :: "r"(addr), "r"(R0), "r"(R1))
#define STMATRIX_X4_T(addr, R0, R1, R2, R3) asm volatile("stmatrix.sync.aligned.x4.trans.m8n8.shared.b16 [%0], {%1, %2, %3, %4};\n" :: "r"(addr), "r"(R0), "r"(R1), "r"(R2), "r"(R3))
// mma m16n8k16
#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1) asm volatile("mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n" : "=r"(RD0), "=r"(RD1) : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0), "r"(RC1))

__device__ inline int div_ceil(int a, int b) { 
  return (a % b != 0) ? (a / b + 1) : (a / b); 
}

template<typename T, const int kWarpSize = WARP_SIZE>
__device__ inline T warp_reduce_sum(T val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

template<typename T, const int kWarpSize = WARP_SIZE>
__device__ inline T warp_reduce_max(T val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    T val_compare = __shfl_xor_sync(0xffffffff, val, mask);
    val = val > val_compare ? val : val_compare;
  }
  return val;
}

template<typename T, const int kNumThreads = 256, const int kWarpSize = WARP_SIZE>
__device__ T block_reduce_sum(T val) {
  static_assert(kWarpSize == 32, "only support warp size = 32.");
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int kNumWarps = (kNumThreads + kWarpSize - 1) / kWarpSize;
  int warp = threadIdx.x / kWarpSize;
  int lane = threadIdx.x % kWarpSize;
  static __shared__ T shared[kNumWarps];
  
  T value = warp_reduce_sum<T, kWarpSize>(val);
  if (lane == 0) shared[warp] = value;
  __syncthreads();
  value = (lane < kNumWarps) ? shared[lane] : 0.0f;
  value = warp_reduce_sum<T, kNumWarps>(value);  
  // WRAN: need to broadcast value to all threads within warp
  value = __shfl_sync(0xffffffff, value, 0);
  return value;
}

template<typename T, const int kNumThreads = 256, const int kWarpSize = WARP_SIZE>
__device__ T block_reduce_max(T val) {
  static_assert(kWarpSize == 32, "only support warp size = 32.");
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int kNumWarps = (kNumThreads + kWarpSize - 1) / kWarpSize;
  int warp = threadIdx.x / kWarpSize;
  int lane = threadIdx.x % kWarpSize;
  static __shared__ T shared[kNumWarps];
  
  T value = warp_reduce_max<T, kWarpSize>(val);
  if (lane == 0) shared[warp] = value;
  __syncthreads();
  value = (lane < kNumWarps) ? shared[lane] : -FLT_MAX;
  value = warp_reduce_max<T, kNumWarps>(value);
  // WRAN: need to broadcast value to all threads within warp
  value = __shfl_sync(0xffffffff, value, 0);
  return value;
}

template<const int kWarpTileQP, const int kWarpTileKV>
__device__ inline void fill_SPO_regs(
  uint32_t (&R_SPO)[kWarpTileQP][kWarpTileKV][2], 
  const uint32_t val = 0) {
  #pragma unroll
  for (int i = 0; i < kWarpTileQP; ++i) {
    #pragma unroll
    for (int j = 0; j < kWarpTileKV; ++j) {
      R_SPO[i][j][0] = val;
      R_SPO[i][j][1] = val;
    }
  }
}

#define INFHALF  = __float2half(65536.0f)
#define ZEROHALF = __float2half(0.0f)
// Write FlashAttention-2 from scratch using Tensor Cores with MMA PTX instruction.
// The input is Q,K,V, 4D tensor with shape [batch_size, num_heads, seq_len, head_dim].
// The output is O, a 4D tensor with shape [batch_size, num_heads, seq_len, head_dim].

// The FlashAttention-2 algorithm is described in the following paper:
// https://arxiv.org/abs/2110.08210

// Q,K,V,O: [batch_size, num_heads, seq_len, head_dim], [B,H,N,d]
// each block processes Q_tile with shape [Br,d] and full K,V with shape [N,d]
// Br or Bc = 64,128,256, etc.

// [64,64], m16n8k16, mma2x4, warp2x2(32,16,16)
// (32x2,16x4,16)=(64,64,16), 256 threads, 8 warps.
// default: Br=128|64, Bc=128|64, d=64|128, kStage=2, kPad=0
// tiling: Q_tile[Br,d]=[128,64], K/V_tile[Bc,d]=[128,64]
// outputs: O_tile[Br,d], lse=logsumexp[Br] per thread block.
// iteration: loop over N for K/V with K/V_tile[Bc,d], Tc iters.
// launch: grid(batch, head_num, N/Br=Tr), block(256=8*mma or 128=4*mma)
// TODO: may return lse=logsumexp[Br].
template<
         const int kHeadDim,    // 32,64,128     
         const int kMmaQP,      // M 16
         const int kMmaKV,      // N 8
         const int kMmaHeadDim, // K 16
         const int kMmaTileQP,  // 2    
         const int kMmaTileKV,  // 4 
         const int kWarpTileQP, // 2
         const int kWarpTileKV, // 2
         const int kStage,      // 1,2
         const int kPad,        // 0,8,16
         >
__global__  void flash_attn_mma_kernel(
  half* Q, half* K, half* V,  half* O, int N) {
  // step 0: S_tile[Br,N] = Q_tile[Br,d] * K[N,d], slice-k manner matmul
  // across K's N dim, each K_tile/V_tile inner loop has shape [Bc,d].
  // step 1: P_tile[Br,N] = softmax(S_tile[Br,N]), row wise.
  // step 2: O_tile[Br,d] = P_tile[Br,N] * V[N,d], matmul.
  static_assert(kHeadDim % 32 == 0); // may relax for 16 ?
  static_assert(kMmaQP == 16 && kMmaKV == 8 && kMmaHeadDim == 16); // m16n8k16
  static_assert(kMmaTileQP  == 2 && kMmaTileKV  == 4);
  static_assert(kWarpTileQP == 2 && kWarpTileKV == 2);
  static_assert(kStage > 0 && kStage < 3); // 1,2
  static_assert(kPad >= 0 && kPad % 8 == 0); // 0,8,16
  constexpr int d  = kHeadDim; // alias
  constexpr int Br = kMmaQP * kMmaTileQP * kWarpTileQP; // 16*2*2=64
  constexpr int Bc = kMmaKV * kMmaTileKV * kWarpTileKV; // 8*4*2=64
  constexpr int Bd = kMmaHeadDim; // 16, tile head_dim(d) according MMA
  constexpr int Tn = WARP_SIZE * kMmaTileQP * kMmaTileKV; // 32*2*4=256
  // NOTE: Now, N must be mutliples of Bc(32/64) for KV tiling across N.
  const int Tr = div_ceil(N, Br); // Tr Q_tile[Br,d]
  const int Tc = div_ceil(N, Bc); // Tc K/V_tile[Bc,d]
  const int Td = div_ceil(d, Bd); // Td K_tile_d[Bc,Bd], e.g [64,16]
  const float scale = 1.0 / sqrt((float)d);
  
  // grid(batch, head_num, N/Br=Tr), block(256=8*mma or 128=4*mma)
  const int QKV_batch_id = blockIdx.x; // B, bx
  const int QKV_head_id  = blockIdx.y; // H, by
  const int QO_tile_id   = blockIdx.z; // Q/O_tile_id, range [0, Tr), bz  
  const int tid = threadIdx.y * blockDim.x + threadIdx.x; // within block
  const int warp_id = tid / WARP_SIZE; // 0~7 warp_id within block
  const int lane_id = tid % WARP_SIZE; // 0~31
  const int warp_QP = warp_id % 2; // 0,1
  const int warp_KV = warp_id / 2; // 0,1,2,3
  // The layout of 8 MMA(2x4) [before] kWarpTileQPxkWarpTileKV(2x2) -> 16x2,8x4=32x32:
  // |  [32,32]  | warp_KV 0 | warp_KV 1 | warp_KV 2 | warp_KV 3 |
  // | warp_QP 0 |-- MMA 0 --|-- MMA 2 --|-- MMA 4 --|-- MMA 6 --|
  // | warp_QP 1 |-- MMA 1 --|-- MMA 3 --|-- MMA 5 --|-- MMA 7 --|
  // The layout of 8 MMA(2x4)  [after] kWarpTileQPxkWarpTileKV(2x2) -> 32x2,32x2=64x64: 
  // |  [64,64]  |    warp_KV 0    |    warp_KV 1    |    warp_KV 2    |    warp_KV 3    |
  // | warp_QP 0 |-- MMA 0,MMA 0 --|-- MMA 2,MMA 2 --|-- MMA 4,MMA 4 --|-- MMA 6,MMA 6 --|
  // | warp_QP 0 |-- MMA 0,MMA 0 --|-- MMA 2,MMA 2 --|-- MMA 4,MMA 4 --|-- MMA 6,MMA 6 --|
  // | warp_QP 1 |-- MMA 1,MMA 1 --|-- MMA 3,MMA 2 --|-- MMA 5,MMA 5 --|-- MMA 7,MMA 7 --|
  // | warp_QP 1 |-- MMA 1,MMA 1 --|-- MMA 3,MMA 2 --|-- MMA 5,MMA 5 --|-- MMA 7,MMA 7 --|
  // gridDim.y = head_num, gridDim.z = N/Br = Tr.
  const int KV_gmem_offset = ((QKV_batch_id * gridDim.y * N * d) + (QKV_head_id * N * d)); 
  const int QO_gmem_offset = ((QKV_batch_id * gridDim.y * N * d) + (QKV_head_id * N * d));
  
  // Shared memory for Q,K,V,O, d=64->24M, d=128=48M
  extern __shared__ half smem[];
  constexpr int QO_tile_size = Br * (d + kPad); // 64*64=4096, ~8192 bytes=8M
  constexpr int KV_tile_size = Bc * (d + kPad); // 64*64=4096, ~8192 bytes=8M, KV may shared 8M
  // Only apply multi stages for K across N(seq_len) dim, not for Q,V.
  half* Q_tile_smem = smem; // 8M/16M
  half* K_tile_smem = Q_tile_smem + QO_tile_size; // 8M/16M
  half* V_tile_smem = K_tile_smem + kStage * KV_tile_size; // no shared smem for KV
  // TODO: KV may shared same smem to reduce smem usage for headdim>=256
  // half* V_tile_smem = K_tile_smem; // KV may shared same smem 8M/16M
  // stage 2, no shared KV smem, Br=Bc=64,  d=64: 8M+(8M)*2+8M   =32M,  shared KV smem: 24M
  // stage 2, no shared KV smem, Br=Bc=64, d=128: 16M+(16M)*2+16M=64M,  shared KV smem: 48M
  // stage 2, no shared KV smem, Br=Bc=64, d=256: 32M+(32M)*2+32M=128M, shared KV smem: 96M
  // stage 1, no shared KV smem, Br=Bc=64, d=256: 32M+(32M)*1+32M=96M,  shared KV smem: 64M
 
  // Mapping gmem -> tid -> smem, Q[Br,d]=[64,64 or 128], 256 threads.
  int load_smem_Q_n = (tid / (Tn / Br)); // Br 64, tid / 4, row 0~64
  int load_smem_Q_d = (tid % (Tn / Br)) * (d / (Tn / Br)); // (tid % 4) * 16, 0,16,32,48
  int load_smem_K_n = (tid / (Tn / Bc)); // Bc 64, tid / 4, row 0~64
  int load_smem_K_d = (tid % (Tn / Bc)) * (d / (Tn / Bc)); // (tid % 4) * 16, 0,16,32,48
  int load_smem_V_n = load_smem_K_n;
  int load_smem_V_d = load_smem_K_d;
  // global Q row of current head with tile [Br,d] per block.
  int load_gmem_Q_n = QO_tile_id * Br + load_smem_Q_n; 
  if (load_gmem_Q_n >= N) return;
  // KV tile gmem load index starts from 0 and increments with 
  // each iteration as we loop over N.
  int load_gmem_K_n = 0; 
  int load_gmem_V_n = 0; 

  uint32_t smem_Q_base_ptr = __cvta_generic_to_shared(Q_tile_smem);
  uint32_t smem_K_base_ptr = __cvta_generic_to_shared(K_tile_smem);
  uint32_t smem_V_base_ptr = __cvta_generic_to_shared(V_tile_smem);
  uint32_t smem_O_base_ptr = __cvta_generic_to_shared(O_tile_smem);

  // load Q from gmem -> smem, only load once.
  {
    int load_gmem_Q_d = load_smem_Q_d;
    int load_gmem_Q_addr = (
      QO_gmem_offset + load_gmem_Q_n * d + load_gmem_Q_d);
    uint32_t load_smem_Q_ptr = (
      smem_Q_base_ptr + (load_smem_Q_n * (d + kPad) + 
                         load_smem_Q_d) * sizeof(half)
    );
    // load d / (Tn / Br) vals, 64 or 128 div 4, 16 or 32, 
    // need 2 or 4 128 bits memory issues.
    #pragma unroll
    for (int i = 0; i < (d / (Tn / Br)); i += 8) {
      CP_ASYNC_CG(load_smem_Q_ptr + i * sizeof(half), 
                  &Q[load_gmem_Q_addr + i], 16);
    }
    CP_ASYNC_COMMIT_GROUP();
  }

  // load K from gmem -> smem, (kStage - 1) K tiles, [Bc,d]
  #pragma unroll
  for (int stage = 0; stage < (kStage - 1); ++stage) {
    // update the offset of n according to stages
    load_gmem_K_n += stage * Bc; // s2, +offset 0
    int load_gmem_K_d = load_smem_K_d;
    int load_gmem_K_addr = (
      KV_gmem_offset + load_gmem_K_n * d + load_gmem_K_d);
     uint32_t load_smem_K_ptr = (
      smem_K_base_ptr + (stage * KV_tile_size + 
                         load_smem_K_n * (d + kPad) + 
                         load_smem_K_d) * sizeof(half)
    );
    // load d / (Tn / Bc) vals, 64 or 128 div 4, 16 or 32, 
    // need 2 or 4 128 bits memory issues.
    #pragma unroll
    for (int i = 0; i < (d / (Tn / Bc)); i += 8) {
      CP_ASYNC_CG(load_smem_K_ptr + i * sizeof(half), 
                  &K[load_gmem_K_addr + i], 16);
    }
    CP_ASYNC_COMMIT_GROUP();
  }

  // wait Q and at least (kStage - 1) for K ready.
  if constexpr (kStage - 2 >= 0) {
    CP_ASYNC_WAIT_GROUP(kStage - 2); // s2->0, s3->1, s4->2
  } else {
    CP_ASYNC_WAIT_GROUP(0);
  }
  __syncthreads(); 

  // NOTE: Init registers/smem for m_i[Br], l_i[Br] and O_i[Br,d] ?
  // or perform as each thread keep one part of m_i, because we will 
  // keep two 32 bits each thread for S/P.

  // m_old, l_old, may use float to keep precision ?
  half thread_max_old[2] = {-INFHALF, -INFHALF}; 
  half thread_sum_old[2] = {ZEROHALF, ZEROHALF};

  // <loop over N>: for K[N,d] with K_tile[Bc,d]
  // tile_n: compute S_tile[Br,Bc] = Q @ K^T = Q_tile[Br,d] * K[Bc,d]
  #pragma unroll
  for (int tile_n = 0; tile_n < Tc; ++tile_n) { 
    // TODO: process last tile_n ? pad to multiple of 8.

    // s2 tn 0->0, 1->1, 2->0; s3 tn 0->0, 1->1, 2->2, 3->0;
    int smem_sel      = (tile_n) % kStage;   
    // s2 tn 0->1, 1->0, 2->1; s3 tn 0->2, 1->0, 2->1, 3->2;  
    int smem_sel_next = (tile_n + (kStage - 1)) % kStage;
    // multi stages pipeling gmem -> smem
    // NOTE: kStage must be > 1 for pipeling. For s1, smem_sel 
    // and smem_sel_next will always equal 0, thus, we can not 
    // prefetch KV from gmem to smem before tile_n MMA done.

    // Prefetch curr V tile_n (no stages)
    {
      load_gmem_V_n += tile_n * Bc;
      int load_gmem_V_d = load_smem_V_d;
      int load_gmem_V_addr = (
        KV_gmem_offset + load_gmem_V_n * d + load_gmem_V_d);
      uint32_t load_smem_V_ptr = (
        smem_V_base_ptr + (load_smem_V_n * (d + kPad) + 
                           load_smem_V_d) * sizeof(half)
      );
      // load d / (Tn / Bc) vals, 64 or 128 div 4, 16 or 32, 
      // need 2 or 4 128 bits memory issues.
      #pragma unroll
      for (int i = 0; i < (d / (Tn / Bc)); i += 8) {
        CP_ASYNC_CG(load_smem_V_ptr + i * sizeof(half), 
                    &K[load_gmem_V_addr + i], 16);
      }
      CP_ASYNC_COMMIT_GROUP();
    }

    // Prefetch next stage K (tile_n + 1)
    if constexpr (kStage > 1) {
      if ((tile_n + 1) < Tc) {
        load_gmem_K_n += (tile_n + 1) * Bc;
        int load_gmem_K_d = load_smem_K_d;
        int load_gmem_K_addr = (
          KV_gmem_offset + load_gmem_K_n * d + load_gmem_K_d);
        uint32_t load_smem_K_ptr = (
          smem_K_base_ptr + (smem_sel_next * KV_tile_size + 
                             load_smem_K_n * (d + kPad) + 
                             load_smem_K_d) * sizeof(half)
        );
        // load d / (Tn / Bc) vals, 64 or 128 div 4, 16 or 32, 
        // need 2 or 4 128 bits memory issues.
        #pragma unroll
        for (int i = 0; i < (d / (Tn / Bc)); i += 8) {
          CP_ASYNC_CG(load_smem_K_ptr + i * sizeof(half), 
                      &K[load_gmem_K_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();
      } else {
        // wait all memory issues ready for last tile.
        CP_ASYNC_WAIT_GROUP(0);
        __syncthreads(); 
      }
    }
    
    // registers for current tile_n within <loop over N>, 
    // [64,64] = S_tile[Br,Bc] = Q_tile[Br,d] * K[Bc,d]
    // each thread hold 2x32 bits regs. S,P,O may shared 
    // the same registers.
    uint32_t R_SPO[kWarpTileQP][kWarpTileKV][2]; // [2][2][2]
    fill_SPO_regs<kWarpTileQP, kWarpTileKV>(R_SPO, 0);

    // registers for Q, K(V reuse)
    uint32_t R_QP[kWarpTileQP][4];
    uint32_t R_KV[kWarpTileKV][2];
    
    // <loop over d>: tile_d, Bd = 16, K_tile_d[Bc,Bd]
    #pragma unroll
    for (int tile_d = 0; tile_d < Td; ++tile_d) {
      // offset d according tile_d
      // smem -> reg, load smem Q
      // ldmatrix.x4 for Q_tile_smem, ldmatrix.x2 for K_tile_smem
      #pragma unroll
      for (int i = 0; i < kWarpTileQP; ++i) {
        int warp_smem_Q_n = warp_QP * (kMmaQP * kWarpTileQP) + i * kMmaQP;
        int lane_smem_Q_n = warp_smem_Q_n + lane_id % 16; // 0~15
        int lane_smem_Q_d = tile_d * Bd + (lane_id / 16) * 8; // 0,8
        uint32_t lane_smem_Q_ptr = (
            smem_Q_base_ptr + (lane_smem_Q_n * (d + kPad) + 
                               lane_smem_Q_d) * sizeof(half)
        );
        LDMATRIX_X4(R_QP[i][0], R_QP[i][1], R_QP[i][2], R_QP[i][3], 
                    lane_smem_Q_ptr); // R_Q
      }

      #pragma unroll
      for (int j = 0; j < kWarpTileKV; ++j) {
        int warp_smem_K_n = warp_KV * (kMmaKV * kWarpTileKV) + j * kMmaKV;
        int lane_smem_K_n = warp_smem_K_n + lane_id % 8; // 0~7, MMA_N=8
        int lane_smem_K_d = tile_d * Bd + ((lane_id / 8) % 2) * 8; // 0,8
        uint32_t lane_smem_K_ptr = (
            smem_K_base_ptr + (smem_sel * KV_tile_size + 
                               warp_smem_K_n * (d + kPad) + 
                               lane_smem_K_d) * sizeof(half)
        );
        LDMATRIX_X2(R_KV[j][0], R_KV[j][1], lane_smem_K_ptr); // R_K
      }

      // MMA compute
      #pragma unroll
      for (int i = 0; i < kWarpTileQP; ++i) {
        #pragma unroll
        for (int j = 0; j < kWarpTileKV; ++j) {
          HMMA16816(R_SPO[i][j][0], R_SPO[i][j][1], 
                    R_QP[i][0],     R_QP[i][1],    R_QP[i][2], R_QP[i][3], 
                    R_KV[j][0],     R_KV[j][1], 
                    R_SPO[i][j][0], R_SPO[i][j][1]);
        }
      }
    } // end loop over d

    // TODO: May reuse K smem for V, for example, stages 2, stage
    // 0 K smem can be reuse as V smem 0 because we do not need 
    // K values on stage 0 K smem anymore.

    // Now, we got a computed tile of S[Br,N], tile with shape [Br,Bc].
    // Assume [Br, Bc] = [64, 64] = 64x64 = 4096 values. Each thread holds
    // a portion of this [Br, Bc] block, specifically, R_S = R_SPO[2][2][2]. 
    // This means that each Warp (MMA) repeats 2 times in the N direction 
    // for both Q and K, resulting in 2x2 = 4 sets of MMA results. Each set 
    // of results is stored in 2 32-bit registers, with each register holding 
    // 2 half-precision values. In other words, each thread stores (4x2)x2 = 16 
    // half-precision values. With a total of 256 threads, the total number of 
    // half-precision values is 256x16 = 4096, which exactly matches the total 
    // [Br, Bc] = [64, 64] values.
    // reference: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
    // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
    // The layout of 8 MMA m16n8k16 (2x4)  [after] kWarpTileQPxkWarpTileKV(2x2) -> 32x2,32x2=64x64: 
    // |  [64,64]  |    warp_KV 0    |    warp_KV 1    |    warp_KV 2    |    warp_KV 3    |
    // | warp_QP 0 |-- MMA 0,MMA 0 --|-- MMA 2,MMA 2 --|-- MMA 4,MMA 4 --|-- MMA 6,MMA 6 --| row max
    // | warp_QP 0 |-- MMA 0,MMA 0 --|-- MMA 2,MMA 2 --|-- MMA 4,MMA 4 --|-- MMA 6,MMA 6 --| row max
    // | warp_QP 1 |-- MMA 1,MMA 1 --|-- MMA 3,MMA 2 --|-- MMA 5,MMA 5 --|-- MMA 7,MMA 7 --| row max
    // | warp_QP 1 |-- MMA 1,MMA 1 --|-- MMA 3,MMA 2 --|-- MMA 5,MMA 5 --|-- MMA 7,MMA 7 --| row max
    // TODO: online safe softmax, warp/block reduce max/sum, row wise
    // m, l, may use float to keep precision ? rowmax总共有Br=64个值
    // 首先，对于每一个MMA持有的结果计算warp row max
    // warp 0/2/4/6 包含了前[Br/2=32,Bc=64]的值，因此需要前32个rowmax值
    // warp 1/3/5/7 包含了后[Br/2=32,Bc=64]的值，因此需要后32个rowmax值
    // 一个warp=32线程，刚好每个线程保存一个max
    // half lane_row_max = -INFHALF; // m, 第i个lane保存第i行的max, 32行
    // half lane_row_sum = ZEROHALF; // l, 第i个lane保存第i行的sum, 32行
    // 每个warp(MMA)处理(16x2)*(8x2)=32x16的大小，row=32, col=16
    #pragma unroll
    for (int i = 0; i < kWarpTileQP; ++i) {
      float lane_max[2] = {-INFINITY, -INFINITY};
      #pragma unroll
      for (int j = 0; j < kWarpTileKV; ++j) {
        // 聚焦到一次MMA的结果上 m16n8
        half2 t_reg_0 = HALF2(R_SPO[i][j][0]); // 0~7  {c0, c1}
        half2 t_reg_1 = HALF2(R_SPO[i][j][1]); // 8~15 {c2, c3}
        float tmp_max_0 = max(__half2float(t_reg_0.x), __half2float(t_reg_0.y));
        float tmp_max_1 = max(__half2float(t_reg_1.x), __half2float(t_reg_1.y));
        lane_max[0] = max(lane_tile_max[0], tmp_max_0);
        lane_max[1] = max(lane_tile_max[1], tmp_max_1);
      }
    }

    // Here, we have to wait V ready before compute O = P @ V
    if constexpr (kStage == 2) {
      // NOTE: we have send V mem issues before K
      CP_ASYNC_WAIT_GROUP(1); // s1->-1, s2->0, s3->1, s4->2
    } else {
      CP_ASYNC_WAIT_GROUP(0);
    }
    __syncthreads(); 

    // NOTE: After compute P @ V, we have to wait next K tile ready in smem.
    // do not need to wait any things if kStage == 1.
    if constexpr (kStage == 2) {
      CP_ASYNC_WAIT_GROUP(0);
      __syncthreads(); 
    }

  } // end loop over N
   

}
