#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <mma.h>
using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2*>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
// gmem -> smem
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n) asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes) asm volatile("cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes) asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
// smem -> gmem: requires sm_90 or higher.
#define CP_ASYNC_BULK_COMMIT_GROUP() asm volatile("cp.async.bulk.commit_group;\n" ::)
#define CP_ASYNC_BULK_WAIT_ALL() asm volatile("cp.async.bulk.wait_all;\n" ::)
#define CP_ASYNC_BULK_WAIT_GROUP(n) asm volatile("cp.async.bulk.wait_group %0;\n" ::"n"(n))
#define CP_ASYNC_BULK(dst, src, bytes) asm volatile("cp.async.bulk.global.shared::cta.bulk_group.L2::128B [%0], [%1], %2;\n" ::"r"(dst), "l"(src), "n"(bytes))
// ldmatrix
#define LDMATRIX_X1(R, addr) asm volatile("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
#define LDMATRIX_X1_T(R, addr) asm volatile("ldmatrix.sync.aligned.x1.trans.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))
#define LDMATRIX_X2_T(R0, R1, addr) asm volatile("ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))
#define LDMATRIX_X4_T(R0, R1, R2, R3, addr) asm volatile("ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3) : "r"(addr))
// stmatrix: requires sm_90 or higher.
#define STMATRIX_X1(addr, R) asm volatile("stmatrix.sync.aligned.x1.m8n8.shared.b16 [%0], {%1};\n" :: "r"(addr), "r"(R))
#define STMATRIX_X2(addr, R0, R1) asm volatile("stmatrix.sync.aligned.x2.m8n8.shared.b16 [%0], {%1, %2};\n" :: "r"(addr), "r"(R0), "r"(R1))
#define STMATRIX_X4(addr, R0, R1, R2, R3) asm volatile("stmatrix.sync.aligned.x4.m8n8.shared.b16 [%0], {%1, %2, %3, %4};\n" :: "r"(addr), "r"(R0), "r"(R1), "r"(R2), "r"(R3))
#define STMATRIX_X1_T(addr, R) asm volatile("stmatrix.sync.aligned.x1.trans.m8n8.shared.b16 [%0], {%1};\n" :: "r"(addr), "r"(R))
#define STMATRIX_X2_T(addr, R0, R1) asm volatile("stmatrix.sync.aligned.x2.trans.m8n8.shared.b16 [%0], {%1, %2};\n" :: "r"(addr), "r"(R0), "r"(R1))
#define STMATRIX_X4_T(addr, R0, R1, R2, R3) asm volatile("stmatrix.sync.aligned.x4.trans.m8n8.shared.b16 [%0], {%1, %2, %3, %4};\n" :: "r"(addr), "r"(R0), "r"(R1), "r"(R2), "r"(R3))
// mma m16n8k16
#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1) asm volatile("mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, %4, %5}, {%6, %7}, {%8, %9};\n" : "=r"(RD0), "=r"(RD1) : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0), "r"(RC1))

__device__ inline int div_ceil(int a, int b) { 
  return (a % b != 0) ? (a / b + 1) : (a / b); 
}

template<typename T, const int kWarpSize = WARP_SIZE>
__device__ inline T warp_reduce_sum(T val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask, kWarpSize);
  }
  return val;
}

template<typename T, const int kWarpSize = WARP_SIZE>
__device__ inline T warp_reduce_max(T val) {
  #pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    T val_compare = __shfl_xor_sync(0xffffffff, val, mask, kWarpSize);
    val = val > val_compare ? val : val_compare;
  }
  return val;
}

template<typename T, const int kNumThreads = 256, const int kWarpSize = WARP_SIZE>
__device__ T block_reduce_sum(T val) {
  static_assert(kWarpSize == 32, "only support warp size = 32.");
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int kNumWarps = (kNumThreads + kWarpSize - 1) / kWarpSize;
  int warp = threadIdx.x / kWarpSize;
  int lane = threadIdx.x % kWarpSize;
  static __shared__ T shared[kNumWarps];
  
  T value = warp_reduce_sum<T, kWarpSize>(val);
  if (lane == 0) shared[warp] = value;
  __syncthreads();
  value = (lane < kNumWarps) ? shared[lane] : 0.0f;
  value = warp_reduce_sum<T, kNumWarps>(value);  
  // WRAN: need to broadcast value to all threads within warp
  value = __shfl_sync(0xffffffff, value, 0);
  return value;
}

template<typename T, const int kNumThreads = 256, const int kWarpSize = WARP_SIZE>
__device__ T block_reduce_max(T val) {
  static_assert(kWarpSize == 32, "only support warp size = 32.");
  // always <= 32 warps per block (limited by 1024 threads per block)
  constexpr int kNumWarps = (kNumThreads + kWarpSize - 1) / kWarpSize;
  int warp = threadIdx.x / kWarpSize;
  int lane = threadIdx.x % kWarpSize;
  static __shared__ T shared[kNumWarps];
  
  T value = warp_reduce_max<T, kWarpSize>(val);
  if (lane == 0) shared[warp] = value;
  __syncthreads();
  value = (lane < kNumWarps) ? shared[lane] : -FLT_MAX;
  value = warp_reduce_max<T, kNumWarps>(value);
  // WRAN: need to broadcast value to all threads within warp
  value = __shfl_sync(0xffffffff, value, 0);
  return value;
}

template<const int M, const int N>
__device__ inline void fill_SPO_regs(uint32_t (&R_SP)[M][N][2], uint32_t val) {
  #pragma unroll
  for (int i = 0; i < M; ++i) {
    #pragma unroll
    for (int j = 0; j < N; ++j) {
      R_SP[i][j][0] = val;
      R_SP[i][j][1] = val;
    }
  }
}

#define INFHALF  = __float2half(65536.0f)
#define ZEROHALF = __float2half(0.0f)

// Write FlashAttention-2 from scratch using Tensor Cores with MMA PTX instruction.
// The input is Q,K,V, 4D tensor with shape [batch_size, num_heads, seq_len, head_dim].
// The output is O, a 4D tensor with shape [batch_size, num_heads, seq_len, head_dim].

// The FlashAttention-2 algorithm is described in the following paper:
// https://arxiv.org/abs/2110.08210

// Q,K,V,O: [batch_size, num_heads, seq_len, head_dim], [B,H,N,d]
// each block processes Q_tile with shape [Br,d] and full K,V with shape [N,d]
// Br or Bc = 64,128,256, etc.

// [64,64], m16n8k16, mma2x4, warp2x2(32,16,16)
// (32x2,16x4,16)=(64,64,16), 256 threads, 8 warps.
// default: Br=128|64, Bc=128|64, d=64|128, kStage=2, kPad=0
// tiling: Q_tile[Br,d]=[128,64], K/V_tile[Bc,d]=[128,64]
// outputs: O_tile[Br,d], lse=logsumexp[Br] per thread block.
// iteration: loop over N for K/V with K/V_tile[Bc,d], Tc iters.
// launch: grid(batch, head_num, N/Br=Tr), block(256=8*mma)
// TODO: may return lse=logsumexp[Br].

// Tiling headdim with Bd=16, SRAM complexity is O((Br+Bc)*16) vs O((Br+Bc)*d)
