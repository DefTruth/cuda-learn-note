#include "hip/hip_runtime.h"
// Modified from: https://github.com/tspeterkim/flash-attention-minimal/blob/main/flash.cu  
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <torch/types.h>
#include <torch/extension.h>

#define ENABLE_NOTE_LOG 0
#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])


__global__ void flash_attn_1_fwd_f32_kernel(
  const float* Q, 
  const float* K, 
  const float* V, 
  const int N, 
  const int d,
  const int Tc,
  const int Tr, 
  const int Bc, 
  const int Br, 
  const float scale,
  float* l, 
  float *m, 
  float* O) {
  int tx = threadIdx.x;
  int bx = blockIdx.x; 
  int by = blockIdx.y;  // batch and head index

  // Offset into Q,K,V,O,l,m - different for each batch and head
  int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = nh
  int lm_offset  = (bx * gridDim.y * N) + (by * N);  // offset for l and m

  // Define SRAM for Q,K,V,S
  extern __shared__ float sram[];
  int tile_size = Bc * d;  // size of Qi, Kj, Vj
  float* Qi = sram;
  float* Kj = &sram[tile_size];
  float* Vj = &sram[tile_size * 2];
  float* S = &sram[tile_size * 3];

  for (int j = 0; j < Tc; j++) {

    // Load Kj, Vj to SRAM
    #pragma unroll
    for (int x = 0; x < d; x++) {
      Kj[(tx * d) + x] = K[qkv_offset + (tile_size * j) + (tx * d) + x];
      Vj[(tx * d) + x] = V[qkv_offset + (tile_size * j) + (tx * d) + x];
    }
    __syncthreads();  // such that the inner loop can use the correct Kj, Vj

    #pragma unroll
    for (int i = 0; i < Tr; i++)  {

      // Load Qi to SRAM, l and m to registers
      #pragma unroll
      for (int x = 0; x < d; x++) {
        Qi[(tx * d) + x] = Q[qkv_offset + (tile_size * i) + (tx * d) + x];
      }
      float row_m_prev = m[lm_offset + (Br * i) + tx];
      float row_l_prev = l[lm_offset + (Br * i) + tx];

      // S = QK^T, row_m = rowmax(S)
      float row_m = -INFINITY;
      #pragma unroll
      for (int y = 0; y < Bc; y++) {
        float sum = 0;
        #pragma unroll
        for (int x = 0; x < d; x++) {
          sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
        }
        sum *= scale;
        S[(Bc * tx) + y] = sum;

        if (sum > row_m)
          row_m = sum;
      }

      // P = exp(S - row_m), row_l = rowsum(P)
      float row_l = 0;
      #pragma unroll
      for (int y = 0; y < Bc; y++) {
        S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - row_m);
        row_l += S[(Bc * tx) + y];
      }

      // Compute new m and l
      float row_m_new = max(row_m_prev, row_m);
      float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) \
                      + (__expf(row_m - row_m_new) * row_l);

      // Write O, l, m to HBM
      #pragma unroll
      for (int x = 0; x < d; x++) {
        float pv = 0;  // Pij * Vj
        #pragma unroll
        for (int y = 0; y < Bc; y++) {
          pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
        }
        O[qkv_offset + (tile_size * i) + (tx * d) + x] = \
          (1 / row_l_new) * ((row_l_prev * __expf(row_m_prev - row_m_new) \
          * O[qkv_offset + (tile_size * i) + (tx * d) + x]) \
          + (__expf(row_m - row_m_new) * pv));
      }
      m[lm_offset + (Br * i) + tx] = row_m_new;
      l[lm_offset + (Br * i) + tx] = row_l_new;
    }
    __syncthreads();
  }
}

// --------------------- PyTorch bindings for custom kernel -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func) \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                 \
if(((T).options().dtype() != (th_type))) {                   \
  std::cout << "Tensor Info:" << (T).options() << std::endl; \
  throw std::runtime_error("values must be "#th_type);       \
}

#define CHECK_TORCH_TENSOR_SHAPE(T1, T2)             \
if (((T2).size(0) != (T1).size(0)) ||                \
    ((T2).size(1) != (T1).size(1)) ||                \
    ((T2).size(2) != (T1).size(2)) ||                \
    ((T2).size(3) != (T1).size(3))) {                \
  throw std::runtime_error("Tensor size mismatch!"); \
}

void flash_attn_1_fwd_f32(
  torch::Tensor Q, 
  torch::Tensor K, 
  torch::Tensor V,
  torch::Tensor O) {
  CHECK_TORCH_TENSOR_DTYPE(Q, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(K, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(V, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(O, torch::kFloat32)
  // TODO: determine Bc, Br dynamically
  const int Bc = 32; 
  const int Br = 32;
  // batch_size, n_head, seq_len, head_dim (B,nh,N,d)
  const int B = Q.size(0); 
  const int nh = Q.size(1);
  const int N = Q.size(2); 
  const int d = Q.size(3);
  CHECK_TORCH_TENSOR_SHAPE(K, Q)
  CHECK_TORCH_TENSOR_SHAPE(V, Q)
  CHECK_TORCH_TENSOR_SHAPE(O, Q)
  const int Tc = ceil((float) N / Bc); 
  const int Tr = ceil((float) N / Br);
  const float scale = 1.0 / sqrt(d);
  
  // Initialize O, l, m to HBM
  auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, 0);
  auto l = torch::zeros({B, nh, N}, options); 
  auto m = torch::full({B, nh, N}, -INFINITY, options);
  
  // Calculate SRAM size needed per block
  const int sram_size = (3 * Bc * d * sizeof(float)) + (Bc * Br * sizeof(float));
  int max_sram_size;
  hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
#if ENABLE_NOTE_LOG
  printf("Max shared memory: %d, requested shared memory: %d \\n", max_sram_size, sram_size);
#endif 
  dim3 grid(B, nh);  // batch_size x num_heads
  dim3 block(Bc);  // Bc threads per block
  
  flash_attn_1_fwd_f32_kernel<<<grid, block, sram_size>>>(
    reinterpret_cast<float*>(Q.data_ptr()), 
    reinterpret_cast<float*>(K.data_ptr()), 
    reinterpret_cast<float*>(V.data_ptr()), 
    N, 
    d, 
    Tc, 
    Tr, 
    Bc, 
    Br, 
    scale,
    reinterpret_cast<float*>(l.data_ptr()), 
    reinterpret_cast<float*>(m.data_ptr()), 
    reinterpret_cast<float*>(O.data_ptr())
  );
}
  

